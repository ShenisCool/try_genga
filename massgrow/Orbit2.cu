#include "hip/hip_runtime.h"
#include "Orbit2.h"

//Constructor
__host__ Data::Data(long long Restart): Host(Restart){


}

//Allocate orbit data
__host__ int Data::AllocateOrbit(){
	hipError_t error;

	//allocate memory on host//
	rcrit_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));
	x4_h = (double4*)malloc(NconstT * sizeof(double4));
	v4_h = (double4*)malloc(NconstT * sizeof(double4));
	v4Helio_h = (double4*)malloc(NconstT * sizeof(double4));
	index_h = (int*)malloc(NconstT * sizeof(int));
	spin_h = (double4*)malloc(NconstT * sizeof(double4));
	love_h = (double3*)malloc(NconstT * sizeof(double3));
	if(P.UseMigrationForce > 0){
		migration_h = (double3*)malloc(NconstT * sizeof(double3));
	}
	else{
		migration_h = NULL;
	}
	if(P.CreateParticles > 0){
		createFlag_h = (int*)malloc(NconstT * sizeof(int));
	}
	else{
		createFlag_h = NULL;
	}
	U_h = (double*)malloc(Nst * sizeof(double));
	LI_h = (double*)malloc(Nst * sizeof(double));
	LI0_h = (double*)malloc(Nst * sizeof(double));
	Energy_h = (double*)malloc(NEnergyT * sizeof(double));
	Energy0_h = (double*)malloc(Nst * sizeof(double));
	Coll_h = (double*)malloc(def_NColl * def_MaxColl * Nst * sizeof(double));
	writeEnc_h = (double*)malloc(def_NColl * def_MaxWriteEnc * Nst * sizeof(double));
	Fragments_h = (double*)malloc(25 * P.Nfragments * Nst * sizeof(double));
	aelimits_h = (float4*)malloc(NconstT * sizeof(float4));
	aecount_h = (unsigned int*)malloc(NconstT * sizeof(unsigned int));
	enccount_h = (unsigned int*)malloc(NconstT * sizeof(unsigned int));
	aecountT_h = (unsigned long long*)malloc(NconstT * sizeof(unsigned long long));
	enccountT_h = (unsigned long long*)malloc(NconstT * sizeof(unsigned long long));

	coordinateBuffer_h = (double*)malloc(P.Buffer * def_BufferSize * NconstT * sizeof(double));
	coordinateBufferIrr_h = (double*)malloc(P.Buffer * def_BufferSize * NconstT * sizeof(double));
	timestepBuffer = (long long int*)malloc(P.Buffer * sizeof(long long int));
	timestepBufferIrr = (long long int*)malloc(P.Buffer * sizeof(long long int));
	NBuffer = (int2*)malloc(Nst * P.Buffer * sizeof(int2));
	NBufferIrr = (int2*)malloc(Nst * P.Buffer * sizeof(int2));
	



#if def_TTV > 0
	{
		int n = def_NtransitTimeMax * NconstT;
		if(def_TTV == 2 && P.PrintTransits == 0){
			n = 0;
		}
		TransitTime_h = (double*)malloc(n * sizeof(double));
		TransitTimeObs_h = (double2*)malloc(def_NtransitTimeMax * N_h[0] * sizeof(double2));
		NtransitsT_h = (int2*)malloc(NconstT * sizeof(int2));
		NtransitsTObs_h = (int*)malloc(N_h[0] * sizeof(int));
	}
#else
	TransitTime_h = NULL;
	TransitTimeObs_h = NULL;
	NtransitsT_h = NULL;
	NtransitsTObs_h = NULL;
#endif

#if def_RV > 0
	RV_h = (double2*)malloc(def_NRVMax * Nst * sizeof(double2));
	RVObs_h = (double3*)malloc(def_NRVMax * Nst * sizeof(double3));
	NRVT_h = (int2*)malloc(Nst * sizeof(int2));
	NRVTObs_h = (int*)malloc(Nst * sizeof(int));
#else
	RV_h = NULL;
	RVObs_h = NULL;
	NRVT_h = NULL;
	NRVTObs_h = NULL;
#endif

#if def_TTV > 0
	elementsA_h = (double4*)malloc(NconstT * sizeof(double4));
	elementsB_h = (double4*)malloc(NconstT * sizeof(double4));
	elementsT_h = (double4*)malloc(NconstT * sizeof(double4));
	elementsSpin_h = (double4*)malloc(NconstT * sizeof(double4));
	elementsL_h = (elements10*)malloc(NconstT * sizeof(elements10));
	elementsC_h = (int2*)malloc((Nst + MCMC_NT) * sizeof(int2));
	elementsP_h = (double4*)malloc(Nst * sizeof(double4));
	elementsSA_h = (double*)malloc(Nst * sizeof(double));
	elementsI_h = (int4*)malloc(NconstT * sizeof(int4));
	elementsM_h = (double*)malloc(Nst * sizeof(double));

  #if MCMC_NCOV > 0
	elementsCOV_h = (double*)malloc(NconstT * N_h[0] * MCMC_NCOV * MCMC_NCOV * sizeof(double));
  #else
	elementsCOV_h = NULL;
  #endif
#else
	elementsA_h = NULL;
	elementsB_h = NULL;
	elementsT_h = NULL;
	elementsSpin_h = NULL;
	elementsL_h = NULL;
	elementsC_h = NULL;
	elementsP_h = NULL;
	elementsSA_h = NULL;
	elementsI_h = NULL;
	elementsM_h = NULL;
	elementsCOV_h = NULL;
#endif

	groupIterate_h = (int*)malloc(sizeof(int));

#if def_poincareFlag == 1
	PFlag_h = (int*)malloc(sizeof(int));
	PFlag_h[0] = 0;
#endif

	BSAstop_h = (int*)malloc(sizeof(int));

	error = hipGetLastError();
	fprintf(masterfile,"CPU malloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("CPU malloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	//allocate pinned memory on host//
#if def_CPU == 0
	hipHostAlloc((void **)&test_h, NconstT * sizeof(double), hipHostMallocDefault);
	hipHostAlloc((void **)&Nencpairs_h, P.ndev * (Nst + 1) * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&Nencpairs2_h, (Nst + 1) * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void **)&Nencpairs3_h, P.SLevels * sizeof(int), hipHostMallocDefault);
#else
	test_h = (double*)malloc(NconstT * sizeof(double));
	Nencpairs_h = (int*)malloc(P.ndev * (Nst + 1) * sizeof(int));
	Nencpairs2_h = (int*)malloc((Nst + 1) * sizeof(int));
	Nencpairs3_h = (int*)malloc(P.SLevels * sizeof(int));

#endif

	error = hipGetLastError();
	fprintf(masterfile,"CPU HostAlloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("CPU HostAlloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	//allocate memory on device//
	hipMalloc((void **) &x4_d, NconstT * sizeof(double4));
	hipMalloc((void **) &v4_d, NconstT * sizeof(double4));
	hipMalloc((void **) &xold_d, NconstT * sizeof(double4));
	hipMalloc((void **) &vold_d, NconstT * sizeof(double4));
	hipMalloc((void **) &rcrit_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &rcritv_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &rcritb_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &rcritvb_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &rcritbb_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &rcritvbb_d, NconstT * P.SLevels * sizeof(double));
	hipMalloc((void **) &test_d, NconstT * sizeof(double));
	hipMalloc((void **) &index_d, NconstT * sizeof(int));
	hipMalloc((void **) &spin_d, NconstT * sizeof(double4));
	hipMalloc((void **) &spinb_d, NconstT * sizeof(double4));
	hipMalloc((void **) &spinbb_d, NconstT * sizeof(double4));
	hipMalloc((void **) &love_d, NconstT * sizeof(double3));
	if(P.UseMigrationForce > 0){
		hipMalloc((void **) &migration_d, NconstT * sizeof(double3));
	}
	else{
		migration_d = NULL;
	}
	if(P.CreateParticles > 0){
		hipMalloc((void **) &createFlag_d, NconstT * sizeof(int));
	}
	else{
		createFlag_d = NULL;
	}
	hipMalloc((void **) &U_d, Nst * sizeof(double));
	hipMalloc((void **) &LI_d, Nst *sizeof(double));
	hipMalloc((void **) &LI0_d, Nst * sizeof(double));
	hipMalloc((void **) &a_d, NconstT * sizeof(double3));
	hipMalloc((void **) &Energy_d, NEnergyT * sizeof(double));
	hipMalloc((void **) &Energy0_d, Nst * sizeof(double));
	hipMalloc((void **) &EnergySum_d, NconstT * sizeof(double));
	hipMalloc((void **) &Nencpairs_d, (Nst + 1) * sizeof(int));
	hipMalloc((void **) &Nencpairs2_d, (Nst + 1) * sizeof(int));
	hipMalloc((void **) &Nencpairs3_d, P.SLevels * sizeof(int));
	hipMalloc((void **) &groupIterate_d, 1 * sizeof(int));
	hipMalloc((void **) &Encpairs_d, sizeof(int2) * NBNencT);
	hipMalloc((void **) &Encpairs2_d, sizeof(int2) * NBNencT);
	hipMalloc((void **) &Encpairs3_d, sizeof(int) * NBNencT * P.SLevels);
	hipMalloc((void **) &scan_d, sizeof(int2) * NconstT);
	hipMalloc((void **) &Coll_d, sizeof(double) * Nst * def_NColl * def_MaxColl);
	hipMalloc((void **) &writeEnc_d, sizeof(double) * Nst * def_NColl * def_MaxWriteEnc);
	hipMalloc((void **) &Fragments_d, sizeof(double) * Nst * 25 * P.Nfragments);
	hipMalloc((void **) &aelimits_d, NconstT * sizeof(float4));
	hipMalloc((void **) &aecount_d, NconstT * sizeof(unsigned int));
	hipMalloc((void **) &enccount_d, NconstT * sizeof(unsigned int));
	hipMalloc((void **) &aecountT_d, NconstT * sizeof(unsigned long long));
	hipMalloc((void **) &enccountT_d, NconstT * sizeof(unsigned long long));

	hipMalloc((void **) &coordinateBuffer_d, P.Buffer * def_BufferSize * NconstT * sizeof(double));
	hipMalloc((void **) &coordinateBufferIrr_d, P.Buffer * def_BufferSize * NconstT * sizeof(double));

	if(P.WriteEncounters == 2){
		hipMalloc((void **) &morton_d, NconstT * sizeof(unsigned int));
		hipMalloc((void **) &sortRank_d, NconstT * sizeof(unsigned int));
		hipMalloc((void **) &sortCount_d, ((NconstT + 255) / 256 + 1) * 16 * sizeof(unsigned int));
		hipMalloc((void **) &sortIndex_d, NconstT * sizeof(int2));
		hipMalloc((void **) &leafNodes_d, NconstT * sizeof(Node));
		hipMalloc((void **) &internalNodes_d, NconstT * sizeof(Node));
	}
	else{
		morton_d = nullptr;
		sortRank_d = nullptr;
		sortCount_d = nullptr;
		sortIndex_d = nullptr;
		leafNodes_d = nullptr;
		internalNodes_d = nullptr;
	}

	if(Nst > 1){
		hipMalloc((void **) &groupIndex_d, NconstT*sizeof(int2));
	}
	else{
		groupIndex_d = NULL;
	}


#if def_CPU == 1
	//arrays for backup step
	xold_h = (double4*)malloc(NconstT * sizeof(double4));
	vold_h = (double4*)malloc(NconstT * sizeof(double4));
	a_h = (double3*)malloc(NconstT * sizeof(double3));
	b_h = (double3*)malloc(Nomp * NconstT * sizeof(double3));


	x4b_h = (double4*)malloc(NconstT * sizeof(double4));
	v4b_h = (double4*)malloc(NconstT * sizeof(double4));
	x4bb_h = (double4*)malloc(NconstT * sizeof(double4));
	v4bb_h = (double4*)malloc(NconstT * sizeof(double4));
	ab_h = (double3*)malloc(NconstT * sizeof(double3));
	indexb_h = (int*)malloc(NconstT * sizeof(int));
	indexbb_h = (int*)malloc(NconstT * sizeof(int));

	rcritb_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));
	rcritbb_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));
	rcritv_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));
	rcritvb_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));
	rcritvbb_h = (double*)malloc(NconstT * P.SLevels * sizeof(double));

	spinb_h = (double4*)malloc(NconstT * sizeof(double4));
	spinbb_h = (double4*)malloc(NconstT * sizeof(double4));

	vcom_h = (double3*)malloc(Nst * sizeof(double3));
	EnergySum_h = (double*)malloc(NconstT * sizeof(double));
	Encpairs_h = (int2*)malloc(NBNencT * sizeof(int2));
	Encpairs2_h = (int2*)malloc(NBNencT * sizeof(int2));
	Encpairs3_h = (int*)malloc(NBNencT * P.SLevels * sizeof(int));
	scan_h = (int2*)malloc(NconstT * sizeof(int2));

	if(Nst > 1){
		groupIndex_h = (int2*)malloc(NconstT * sizeof(int2));
	}
	else{
		groupIndex_h = NULL;
	}

	xt_h = (double4*)malloc(NconstT * sizeof(double4));
	vt_h = (double4*)malloc(NconstT * sizeof(double4));
	xp_h = (double4*)malloc(NconstT * sizeof(double4));
	vp_h = (double4*)malloc(NconstT * sizeof(double4));
	dx_h = (double3*)malloc(NconstT * 8 * sizeof(double3));
	dv_h = (double3*)malloc(NconstT * 8 * sizeof(double3));
	dt1_h = (double*)malloc(NconstT * sizeof(double));
	t1_h = (double*)malloc(NconstT * sizeof(double));
	dtgr_h = (double*)malloc(NconstT * sizeof(double));
	Coltime_h = (double*)malloc(sizeof(double));
	BSstop_h = (int*)malloc(sizeof(int));
	BSAstop_h = (int*)malloc(sizeof(int));

  #if USE_RANDOM == 1
	srand48(time(NULL));
	random_h = (hiprandState*)malloc(NconstT * sizeof(hiprandState));
  #else
	random_h = NULL;
  #endif
	if(P.WriteEncounters == 2){
		morton_h = (unsigned int*)malloc(NconstT * sizeof(unsigned int));
		sortRank_h = (unsigned int*)malloc(NconstT * sizeof(unsigned int));
		sortCount_h = (unsigned int*)malloc(((NconstT + 255) / 256 + 1) * 16 * sizeof(unsigned int));
		sortIndex_h = (int2*)malloc(NconstT * sizeof(int2));
		leafNodes_h = (Node*)malloc(NconstT * sizeof(Node));
		internalNodes_h = (Node*)malloc(NconstT * sizeof(Node));
	}
	else{
		morton_h = nullptr;
		sortRank_h = nullptr;
		sortCount_h = nullptr;
		sortIndex_h = nullptr;
		leafNodes_h = nullptr;
		internalNodes_h = nullptr;
	}

#endif

	// ------------------------
	// MultiGPU allocation
#if def_CPU == 0
	if(P.ndev > 1){
		hipSetDevice(P.dev[1]);
		hipMalloc((void **) &rcritv_d1, NconstT * P.SLevels * sizeof(double));
		hipMalloc((void **) &x4_d1, NconstT * sizeof(double4));
		hipMalloc((void **) &Nencpairs_d1, (Nst + 1) * sizeof(int));
		hipMalloc((void **) &Encpairs_d1, sizeof(int2) * NBNencT);
		hipMalloc((void **) &Encpairs2_d1, sizeof(int2) * NBNencT);
	}
	if(P.ndev > 2){
		hipSetDevice(P.dev[2]);
		hipMalloc((void **) &rcritv_d2, NconstT * P.SLevels * sizeof(double));
		hipMalloc((void **) &x4_d2, NconstT * sizeof(double4));
		hipMalloc((void **) &Nencpairs_d2, (Nst + 1) * sizeof(int));
		hipMalloc((void **) &Encpairs_d2, sizeof(int2) * NBNencT);
		hipMalloc((void **) &Encpairs2_d2, sizeof(int2) * NBNencT);
	}
	if(P.ndev > 3){
		hipSetDevice(P.dev[3]);
		hipMalloc((void **) &rcritv_d3, NconstT * P.SLevels * sizeof(double));
		hipMalloc((void **) &x4_d3, NconstT * sizeof(double4));
		hipMalloc((void **) &Nencpairs_d3, (Nst + 1) * sizeof(int));
		hipMalloc((void **) &Encpairs_d3, sizeof(int2) * NBNencT);
		hipMalloc((void **) &Encpairs2_d3, sizeof(int2) * NBNencT);
	}
	if(P.ndev < 2){
		rcritv_d1 = nullptr;
		x4_d1 = nullptr;
		Nencpairs_d1 = nullptr;
		Encpairs_d1 = nullptr;
		Encpairs2_d1 = nullptr;
	}
	if(P.ndev < 3){
		rcritv_d2 = nullptr;
		x4_d2 = nullptr;
		Nencpairs_d2 = nullptr;
		Encpairs_d2 = nullptr;
		Encpairs2_d2 = nullptr;
	}
	if(P.ndev < 4){
		rcritv_d3 = nullptr;
		x4_d3 = nullptr;
		Nencpairs_d3 = nullptr;
		Encpairs_d3 = nullptr;
		Encpairs2_d3 = nullptr;
	}

	if(P.ndev > 1){
		hipSetDevice(P.dev[0]);
	}
#endif
	// ------------------------


#if def_TTV == 1
	hipMalloc((void **) &Transit_d, def_NtransitMax * sizeof(int));
#else
	Transit_d = NULL;

#endif

#if def_TTV > 0
	{
		int n = def_NtransitTimeMax * NconstT;
		if(def_TTV == 2 && P.PrintTransits == 0){
			n = 0;
		}
		hipMalloc((void **) &TransitTime_d, n * sizeof(double));
		hipMalloc((void **) &TransitTimeObs_d, def_NtransitTimeMax * N_h[0] * sizeof(double2));
		hipMalloc((void **) &NtransitsT_d, NconstT * sizeof(int2));
		hipMalloc((void **) &NtransitsTObs_d, N_h[0] * sizeof(int));
	}
#else
	TransitTime_d = NULL;
	TransitTimeObs_d = NULL;
	NtransitsT_d = NULL;
	NtransitsTObs_d = NULL;
#endif

#if def_RV > 0
	hipMalloc((void **) &RV_d, def_NRVMax * Nst * sizeof(double2));
	hipMalloc((void **) &RVObs_d, def_NRVMax * Nst * sizeof(double3));
	hipMalloc((void **) &NRVT_d, Nst * sizeof(int2));
	hipMalloc((void **) &NRVTObs_d, Nst * sizeof(int2));
	hipMalloc((void **) &RVP_d, Nst * sizeof(double));
#else
	RV_d = NULL;
	RVObs_d = NULL;
	NRVT_d = NULL;
	NRVTObs_d = NULL;
	RVP_d = NULL;
#endif
#if def_TTV > 0
	hipMalloc((void **) &elementsA_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsB_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsT_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsSpin_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsAOld_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsAOld2_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsBOld_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsBOld2_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsTOld_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsTOld2_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsSpinOld_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsSpinOld2_d, NconstT * sizeof(double4));
	hipMalloc((void **) &elementsL_d, NconstT * sizeof(elements10));
	hipMalloc((void **) &elementsC_d, (Nst + MCMC_NT) * sizeof(int2));
	hipMalloc((void **) &elementsP_d, Nst * sizeof(double4));
	hipMalloc((void **) &elementsSA_d, Nst * sizeof(double));
	hipMalloc((void **) &elementsI_d, NconstT * sizeof(int4));
	hipMalloc((void **) &elementsM_d, Nst * sizeof(double));
  #if MCMC_BLOCK == 5
	hipMalloc((void **) &elementsG_d, NconstT * sizeof(elements8));
	hipMalloc((void **) &elementsGh_d, NconstT * sizeof(elements8));
	hipMalloc((void **) &elementsD_d, NconstT * sizeof(elements8));
	hipMalloc((void **) &elementsMean_d, NconstT * sizeof(elements8));
	hipMalloc((void **) &elementsVar_d, NconstT * sizeof(elements8));
  #else
	elementsG_d = NULL;
	elementsGh_d = NULL;
	elementsD_d = NULL;
	elementsMean_d = NULL;
	elementsVar_d = NULL;
  #endif
  #if MCMC_BLOCK == 6
printf("size %lu %lu %lu\n", sizeof(double), sizeof(elements), Nst * (N_h[0] + 1) * sizeof(elements));
	hipMalloc((void **) &Symplex_d, Nst * (N_h[0] * P.mcmcNE + 2) * N_h[0] * sizeof(elements));
	hipMalloc((void **) &SymplexCount_d, Nst * sizeof(int));
  #else
	Symplex_d = NULL;
	SymplexCount_d = NULL;
  #endif
  #if MCMC_BLOCK == 7
	hipMalloc((void **) &elementsStep_d, NconstT * sizeof(elementsS));
	hipMalloc((void **) &elementsHist_d, (Nst + N_h[0] * P.mcmcNE) / (N_h[0] * P.mcmcNE + 1) * N_h[0] * MCMC_NH * sizeof(elementsH));
  #else
	elementsStep_d = NULL;
	elementsHist_d = NULL;
  #endif

  #if MCMC_NCOV > 0
	hipMalloc((void **) &elementsCOV_d, NconstT * N_h[0] * MCMC_NCOV * MCMC_NCOV * sizeof(double));
  #else
	elementsCOV_d = NULL;
  #endif
#else
	elementsA_d = NULL;
	elementsB_d = NULL;
	elementsT_d = NULL;
	elementsSpin_d = NULL;
	elementsAOld_d = NULL;
	elementsAOld2_d = NULL;
	elementsBOld_d = NULL;
	elementsBOld2_d = NULL;
	elementsTOld_d = NULL;
	elementsTOld2_d = NULL;
	elementsSpinOld_d = NULL;
	elementsSpinOld2_d = NULL;
	elementsL_d = NULL;
	elementsC_d = NULL;
	elementsP_d = NULL;
	elementsSA_d = NULL;
	elementsI_d = NULL;
	elementsM_d = NULL;
	elementsG_d = NULL;
	elementsGh_d = NULL;
	elementsD_d = NULL;
	elementsMean_d = NULL;
	elementsVar_d = NULL;
	elementsCOV_d = NULL;
	Symplex_d = NULL;
	SymplexCount_d = NULL;
	elementsStep_d = NULL;
	elementsHist_d = NULL;
#endif

#if def_TTV == 2
	hipMalloc((void **) &timeold_d, Nst * sizeof(double));
	hipMalloc((void **) &lastTransitTime_d, NconstT * sizeof(double));
	hipMalloc((void **) &transitIndex_d, Nst * sizeof(int));
	hipMalloc((void **) &EpochCount_d, NconstT * sizeof(int2));
	hipMalloc((void **) &TTV_d, NconstT * sizeof(int));
#else
	timeold_d = NULL;
	lastTransitTime_d = NULL;
	transitIndex_d = NULL;
	EpochCount_d = NULL;
	TTV_d = NULL;

#endif
	//arrays for backup step
	hipMalloc((void **) &x4b_d, NconstT * sizeof(double4));
	hipMalloc((void **) &v4b_d, NconstT * sizeof(double4));
	hipMalloc((void **) &x4bb_d, NconstT * sizeof(double4));
	hipMalloc((void **) &v4bb_d, NconstT * sizeof(double4));
	hipMalloc((void **) &ab_d, NconstT * sizeof(double3));
	hipMalloc((void **) &indexb_d, NconstT * sizeof(int));
	hipMalloc((void **) &indexbb_d, NconstT * sizeof(int));


	//arrays for BSA
	hipMalloc((void **) &xt_d, NconstT * sizeof(double4));
	hipMalloc((void **) &vt_d, NconstT * sizeof(double4));
	hipMalloc((void **) &xp_d, NconstT * sizeof(double4));
	hipMalloc((void **) &vp_d, NconstT * sizeof(double4));
	hipMalloc((void **) &dx_d, NconstT * 8 * sizeof(double3));
	hipMalloc((void **) &dv_d, NconstT * 8 * sizeof(double3));
	hipMalloc((void **) &dt1_d, NconstT * sizeof(double));
	hipMalloc((void **) &t1_d, NconstT * sizeof(double));
	hipMalloc((void **) &dtgr_d, NconstT * sizeof(double));
	hipMalloc((void **) &BSAstop_d, sizeof(int));
	hipMalloc((void **) &BSstop_d, sizeof(int));
	hipMalloc((void **) &Coltime_d, sizeof(double));
#if def_G3 > 0
	hipMalloc((void **) &K_d, NconstT * NconstT * sizeof(double));
	hipMalloc((void **) &Kold_d, NconstT * NconstT * sizeof(double));
	hipMalloc((void **) &x4G3_d, NconstT * sizeof(double4));
	hipMalloc((void **) &v4G3_d, NconstT * sizeof(double4));
#else
	K_d = NULL;
	Kold_d = NULL;
	x4G3_d = NULL;
	v4G3_d = NULL;
	
#endif
	hipMalloc((void **) &vcom_d, Nst * sizeof(double3));
	hipMalloc((void **) &StopFlag_d, sizeof(int));
	hipMalloc((void **) &ErrorFlag_d, sizeof(int));

#if def_poincareFlag == 1
	hipMalloc((void **) &PFlag_d, sizeof(int));
	hipMemcpy(PFlag_d, PFlag_h, sizeof(int), hipMemcpyHostToDevice);
#endif

#if USE_RANDOM == 1
	hipMalloc((void **) &random_d, NconstT * sizeof(hiprandState));
#else
	random_d = NULL;
#endif

	CollisionFlag = 0;

	error = hipGetLastError();
	fprintf(masterfile,"hipMalloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("hipMalloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	return 1;
};


//This function allocates mapped memory
__host__ int Data::CMallocateOrbit(){
#if def_CPU == 0
	hipError_t error;

	hipHostAlloc((void **)&Nenc_m, def_GMax * sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&Nenc_d, (void *)Nenc_m, 0);

	hipHostAlloc((void **)&Ncoll_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&Ncoll_d, (void *)Ncoll_m, 0);

	hipHostAlloc((void **)&Ntransit_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&Ntransit_d, (void *)Ntransit_m, 0);

	hipHostAlloc((void **)&NWriteEnc_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&NWriteEnc_d, (void *)NWriteEnc_m, 0);

	hipHostAlloc((void **)&EjectionFlag_m, (Nst + 1)*sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&EjectionFlag_d, (void *)EjectionFlag_m, 0);

	hipHostAlloc((void **)&nFragments_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&nFragments_d, (void *)nFragments_m, 0);

	hipHostAlloc((void **)&EncFlag_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&EncFlag_d, (void *)EncFlag_m, 0);

	hipHostAlloc((void **)&StopFlag_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&StopFlag_d, (void *)StopFlag_m, 0);

	hipHostAlloc((void **)&ErrorFlag_m, sizeof(int), hipHostMallocMapped);
	hipHostGetDevicePointer((void **)&ErrorFlag_d, (void *)ErrorFlag_m, 0);
#else
	Nenc_m = (int*)malloc(def_GMax * sizeof(int));
	Ncoll_m = (int*)malloc(sizeof(int));
	Ntransit_m = (int*)malloc(sizeof(int));
	NWriteEnc_m = (int*)malloc(sizeof(int));
	EjectionFlag_m = (int*)malloc((Nst + 1) * sizeof(int));
	nFragments_m = (int*)malloc(sizeof(int));
	EncFlag_m = (int*)malloc(sizeof(int));
	StopFlag_m = (int*)malloc(sizeof(int));
	ErrorFlag_m = (int*)malloc(sizeof(int));
#endif
	EncFlag_m[0] = 0;
	StopFlag_m[0] = 0;
	ErrorFlag_m[0] = 0;

	error = hipGetLastError();
	fprintf(masterfile,"mapping error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("mapping error = %d = %s\n",error, hipGetErrorString(error));
		 return 0;
	}

	return 1;

}


//This function allocates the Gridae and set values to zero
__host__ int Data::GridaeAlloc(){
	hipError_t error;
	GridNae = Gridae.Na * Gridae.Ne;
	hipMalloc((void **) &Gridaecount_d, GridNae * sizeof(unsigned int));
	Gridaecount_h = (unsigned int*)malloc(GridNae * sizeof(unsigned int));
	GridaecountT_h = (unsigned long long*)malloc(GridNae * sizeof(unsigned long long));
	GridaecountS_h = (unsigned long long*)malloc(GridNae * sizeof(unsigned long long));

	for(int i = 0; i < GridNae; ++i){
		Gridaecount_h[i] = 0u;
		GridaecountT_h[i] = 0ull;
		GridaecountS_h[i] = 0ull;
	}
	hipMemcpy(Gridaecount_d, Gridaecount_h, sizeof(unsigned int)*GridNae, hipMemcpyHostToDevice);
	GridNai = Gridae.Na * Gridae.Ni;
	hipMalloc((void **) &Gridaicount_d, GridNai * sizeof(unsigned int));
	Gridaicount_h = (unsigned int*)malloc(GridNai * sizeof(unsigned int));
	GridaicountT_h = (unsigned long long*)malloc(GridNai * sizeof(unsigned long long));
	GridaicountS_h = (unsigned long long*)malloc(GridNai * sizeof(unsigned long long));

	for(int i = 0; i < GridNai; ++i){
		Gridaicount_h[i] = 0u;
		GridaicountT_h[i] = 0ull;
		GridaicountS_h[i] = 0ull;
	}
	hipMemcpy(Gridaicount_d, Gridaicount_h, sizeof(unsigned int)*GridNai, hipMemcpyHostToDevice);

	constantCopy();

	error = hipGetLastError();
	fprintf(masterfile,"GrideaeAlloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("GrideaeAlloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	return 1;
}


__host__ int Data::FGAlloc(){
	hipError_t error;
	double S_h[def_FGN + 1];
	double C_h[def_FGN + 1];

	//Table for fastfg//
	for (int j = 0; j<= def_FGN; ++j) {
		double dEj = j*PI_N;
		S_h[j] = sin(dEj);
		C_h[j] = cos(dEj);
	}
	constantCopySC(S_h, C_h);
	error = hipGetLastError();
	fprintf(masterfile,"FGAlloc error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("FGAlloc error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	return 1;
}


//This function reads at a restart the correspondent Gridae file
__host__ int Data::readGridae(){
	if(P.tRestart > 0){
		sprintf(Gridae.filename, "aeCount%s_%.*lld.dat", Gridae.X, def_NFileNameDigits, P.tRestart);
		Gridae.file = fopen(Gridae.filename, "r");
		if(Gridae.file == NULL){
			fprintf(masterfile, "Error: aeGrid file not found: aeCount%s_%.*lld.dat\n", Gridae.X, def_NFileNameDigits, P.tRestart);
			printf("Error: aeGrid file not found: aeCount%s_%.*lld.dat\n", Gridae.X, def_NFileNameDigits, P.tRestart);
			return 0;
		}
		//Read Total aeGrid
		int er = 0;
		for(int i = 0; i < Gridae.Ne; ++i){
			for(int j = 0; j < Gridae.Na; ++j){
				er = fscanf(Gridae.file, "%lld",&GridaecountT_h[i * Gridae.Na + j]);
				if(er <= 0){
					return 0;
				}
			}
		}
		//Skip Temporal aeGrid
		int skip;
		for(int i = 0; i < Gridae.Ne; ++i){
			for(int j = 0; j < Gridae.Na; ++j){
				er = fscanf(Gridae.file, "%d",&skip);
				if(er <= 0){
					return 0;
				}
			}
		}
		//Read Total aiGrid
		for(int i = 0; i < Gridae.Ni; ++i){
			for(int j = 0; j < Gridae.Na; ++j){
				er = fscanf(Gridae.file, "%lld",&GridaicountT_h[i * Gridae.Na + j]);
				if(er <= 0){
					return 0;
				}
			}
		}
		fclose(Gridae.file);
	}
	return 1;
}

//This function copies values from the current Gridae to the total and summing host Grid
__host__ int Data::copyGridae(){
	hipError_t error;
	//ae grid
	hipMemcpy(Gridaecount_h, Gridaecount_d, sizeof(unsigned int) * GridNae, hipMemcpyDeviceToHost);
	for(int i = 0; i < Gridae.Ne; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			if(timeStep > Gridae.Start){
				GridaecountS_h[i * Gridae.Na + j] += Gridaecount_h[i * Gridae.Na + j];
				GridaecountT_h[i * Gridae.Na + j] += Gridaecount_h[i * Gridae.Na + j];
			}
		}
	}
	hipMemset(Gridaecount_d, 0, sizeof(int)*GridNae);
	//ae grid
	hipMemcpy(Gridaicount_h, Gridaicount_d, sizeof(unsigned int) * GridNai, hipMemcpyDeviceToHost);
	for(int i = 0; i < Gridae.Ni; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			if(timeStep > Gridae.Start){
				GridaicountS_h[i * Gridae.Na + j] += Gridaicount_h[i * Gridae.Na + j];
				GridaicountT_h[i * Gridae.Na + j] += Gridaicount_h[i * Gridae.Na + j];
			}
		}
	}
	hipMemset(Gridaicount_d, 0, sizeof(int)*GridNai);
	error = hipGetLastError();
	fprintf(masterfile,"Grideae copy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("Grideae copy error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	return 1;
}


//This function reads the covariance matrix for MCMC sampling
//The file must contain the Cholesky decompoistion part L from COV = L L^T
__host__ int Data::readMCMC_COV(){
	FILE *COVfile;

	COVfile = fopen("MCMCL.dat", "r");
	if(COVfile == NULL){
		fprintf(masterfile, "Error: MCMCL.dat file not found\n");
		printf("Error: MCMCL.dat file not found\n");
		return 0;
	}
	int ii, jj;
	int er = 0;
	for(int i = 0; i < NconstT * MCMC_NCOV; ++i){
		for(int j = 0; j < N_h[0] * MCMC_NCOV; ++j){
			er = fscanf(COVfile, "%d",&ii);
			er = fscanf(COVfile, "%d",&jj);
			er = fscanf(COVfile, "%lf",&elementsCOV_h[i * N_h[0] * MCMC_NCOV + j]);
//printf("MCMCL %d %d %d %g\n", i, i % N_h[0], j, elementsCOV_h[i * N_h[0] * MCMC_NCOV + j]);
			if(er <= 0){
				return 0;
			}
			int iii = 0;
			int NM = N_h[0] * MCMC_NCOV;
			if(NM != 0) iii = i % NM;
			if(ii != iii || jj != j){
				fprintf(masterfile, "Error: MCMCL.dat file not the correct size %d %d %d %d\n", ii, iii, jj, j);
				printf("Error: MCMCL.dat file not the correct size %d %d %d %d\n", ii, iii, jj, j);
				return 0;
			}
		}
	}
	return 1;
}

__global__ void BufferInit_kernel(double *coordinateBuffer_d, const int N){

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < N){
		coordinateBuffer_d[id] = 0.0;
	}
}

#if USE_RANDOM == 1
__global__ void randomInit_kernel(hiprandState *random_d, const int N){
  #if def_CPU == 0
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < N){
		//hiprand_init(0, id, 0, &random_d[id]);
		hiprand_init(clock64(), id, 0, &random_d[id]);
	}
  #endif
}

#endif



//This function initializes the data
__host__ int Data::init(){

	doTransits = 0;
#if def_TTV == 1
	doTransits = 1;
#endif
	Ncoll_m[0] = 0;
	Ntransit_m[0] = 0;
	NWriteEnc_m[0] = 0;
	nFragments_m[0] = 0;
	for(int i = 0; i < def_GMax; ++i){
		Nenc_m[i] = 0;
	}
	EjectionFlag_m[0] = 0;
	for(int i = 0; i < NconstT * P.SLevels; ++i){
		rcrit_h[i] = 0.0;
	}
	for(int i = 0; i < NconstT; ++i){
		index_h[i] = -1;
		x4_h[i].x = 1.0;
		x4_h[i].y = 0.0;
		x4_h[i].z = 0.0; 
		x4_h[i].w = -1.0e-12;
		v4_h[i].x = 0.0;
		v4_h[i].y = 0.0;
		v4_h[i].z = 0.0;
		v4_h[i].w = 0.0;
		test_h[i] = -1.0;
		spin_h[i].x = 0.0;
		spin_h[i].y = 0.0;
		spin_h[i].z = 0.0;
		spin_h[i].w = 0.4;	//2.0/5.0
		love_h[i].x = 0.0;
		love_h[i].y = 0.0;
		love_h[i].z = 0.0;
		if(P.UseMigrationForce > 0){ 
			migration_h[i].x = 0.0;
			migration_h[i].y = 0.0;
			migration_h[i].z = 0.0;
		}
		if(P.CreateParticles > 0){
			createFlag_h[i] = -1;
		}
		aelimits_h[i].x = 0.0f;
		aelimits_h[i].y = 1.0f;
		aelimits_h[i].z = 0.0f;
		aelimits_h[i].w = 1.0f;
		aecount_h[i] = 0u;
		enccount_h[i] = 0u;
		aecountT_h[i] = 0ull;
		enccountT_h[i] = 0ull;
#if def_TTV > 0
		elementsA_h[i].x = 0.0;
		elementsA_h[i].y = 0.0;
		elementsA_h[i].z = 0.0;
		elementsA_h[i].w = -1.0e-12;
		elementsB_h[i].x = 0.0;
		elementsB_h[i].y = 0.0;
		elementsB_h[i].z = 0.0;
		elementsB_h[i].w = 0.0;
		elementsT_h[i].x = 0.0;
		elementsT_h[i].y = 0.0;
		elementsT_h[i].z = 0.0;
		elementsT_h[i].w = 0.0;
		elementsSpin_h[i].x = 0.0;
		elementsSpin_h[i].y = 0.0;
		elementsSpin_h[i].z = 0.0;
		elementsSpin_h[i].w = 0.0;
		elementsL_h[i].P = 0.0;
		elementsL_h[i].T = 0.0;
		elementsL_h[i].m = 0.0;
		elementsL_h[i].e = 0.0;
		elementsL_h[i].w = 0.0;
		elementsL_h[i].inc = 0.0;
		elementsL_h[i].O = 0.0;
		elementsL_h[i].r = 0.0;
		elementsL_h[i].a = 0.0;
		elementsL_h[i].M = 0.0;
		elementsI_h[i].x = 0;
		elementsI_h[i].y = 0;
		elementsI_h[i].z = 0;
		elementsI_h[i].w = P.mcmcNE * N_h[0];


		if(i < Nst){
			elementsP_h[i].x = 1.0e300;		//initial value for sum
			elementsP_h[i].y = 0.0;		//contains later a random number
			elementsP_h[i].z = 1.0e300;	//new p
			elementsP_h[i].w = 1.0;		//tunig factor according to acceptance rate
			elementsSA_h[i] = 1.0;
			elementsM_h[i] = Msun_h[i].x;
		}
		if(i < Nst + MCMC_NT){
			elementsC_h[i].x = 0;
			elementsC_h[i].y = 0;
		}
#endif
	}
#if def_TTV > 0
  #if MCMC_NCOV > 0
	for(int j = 0; j < NconstT * N_h[0] * MCMC_NCOV * MCMC_NCOV; ++j){
		elementsCOV_h[j] = 0.0;
	}
  #endif
#endif
	for(int st = 0; st < Nst; ++st){
		EjectionFlag_m[st + 1] = 0;
		for(int i = 0; i < N_h[st] + Nsmall_h[st]; ++i){
			index_h[NBS_h[st] + i] = i + st * def_MaxIndex;
		}
	}
	for(int i = 0; i < P.Buffer * def_BufferSize * NconstT; ++i){
		coordinateBuffer_h[i] = 0.0;
		coordinateBufferIrr_h[i] = 0.0;
	}
	for(int i = 0; i < P.Buffer; ++i){
		timestepBuffer[i] = 0ll;
		timestepBufferIrr[i] = 0ll;
		for(int st = 0; st < Nst; ++st){
			NBuffer[i * Nst + st].x = N_h[st];
			NBuffer[i * Nst + st].y = Nsmall_h[st];
			NBufferIrr[i * Nst + st].x = N_h[st];
			NBufferIrr[i * Nst + st].y = Nsmall_h[st];
		}
	}
	BufferInit_kernel <<< (P.Buffer * def_BufferSize * NconstT + 511) / 512, 512 >>> (coordinateBuffer_d, P.Buffer * def_BufferSize * NconstT);
	BufferInit_kernel <<< (P.Buffer * def_BufferSize * NconstT + 511) / 512, 512 >>> (coordinateBufferIrr_d, P.Buffer * def_BufferSize * NconstT);
	for(int i = 0; i < NEnergyT; ++i){
		Energy_h[i] = 0.0;
	}

	for(int i = 0; i < Nst * def_NColl * def_MaxColl; ++i){
		Coll_h[i] = 0.0;
	}

	for(int i = 0; i < Nst * def_NColl * def_MaxWriteEnc; ++i){
		writeEnc_h[i] = 0.0;
	}

	for(int i = 0; i < Nst * 25 * P.Nfragments; ++i){
		Fragments_h[i] = 0.0;
	}

	for(int st = 0; st < P.ndev * (Nst + 1); ++st){
		Nencpairs_h[st] = 0;
	}
	for(int st = 0; st < Nst + 1; ++st){
		Nencpairs2_h[st] = 0;
	}
	for(int i = 0; i < P.SLevels; ++i){
		Nencpairs3_h[i] = 0;
	}
	for(int st = 0; st < Nst; ++st){
		U_h[st] = 0.0;
		LI_h[st] = 0.0;
		LI0_h[st] = 1.0;
		Energy0_h[st] = 1.0;
	}

#if USE_RANDOM == 1
	randomInit_kernel <<< (NconstT + 255) / 256, 256>>> (random_d, NconstT);
#endif

	return 1;
}


//This function calls the readic function and copies the data to the GPU.
__host__ int Data::ic(){
	for(int st = 0; st < Nst; ++st){
		if(N_h[st] + Nsmall_h[st] > 0){
			GSF[st].logfile = fopen(GSF[st].logfilename, "a");
			int NBS = NBS_h[st];
			fprintf(GSF[st].logfile, "\n************* Read initial conditions ****************\n \n");
			int icerr = 0;
			icerr = readic(st);
			if(icerr == 0){
				printf("Error: Could not read initial conditions\n");
				fprintf(GSF[st].logfile, "Error: Could not read initial conditions\n");
				fprintf(masterfile, "Error in Simulation %s\n", GSF[st].path);
				return 0;
			}
			if(Nsmall_h[st] < Nmin[st].y && P.UseTestParticles > 0){
				printf("Error: No Test Particles found\n");
				fprintf(GSF[st].logfile, "Error: No Test Particles found\n");
				fprintf(masterfile, "Error: No Test Particles found %s\n", GSF[st].path);
				return 0;
			}
			fclose(GSF[st].logfile);
			HelioToDemo(x4_h + NBS, v4_h + NBS, Msun_h[st].x, N_h[st] + Nsmall_h[st]);
			//HelioToBary(x4_h + NBS, v4_h + NBS, Msun_h[st].x, N_h[st] + Nsmall_h[st]);
		}
	}
	//Copy memory to device//
	hipMemcpy(x4_d, x4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(v4_d, v4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(x4b_d, x4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(v4b_d, v4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(x4bb_d, x4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(v4bb_d, v4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(xold_d, x4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(vold_d, v4_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(rcrit_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(rcritv_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(rcritb_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(rcritvb_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(rcritbb_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(rcritvbb_d, rcrit_h, sizeof(double) * NconstT * P.SLevels, hipMemcpyHostToDevice);
	hipMemcpy(U_d, U_h, Nst * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(LI_d, LI_h, Nst * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(LI0_d, LI0_h, Nst * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Energy_d, Energy_h, sizeof(double) * NEnergyT, hipMemcpyHostToDevice);
	hipMemcpy(Energy0_d, Energy0_h, Nst * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(test_d, test_h, sizeof(double) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(index_d, index_h, sizeof(int) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(indexb_d, index_h, sizeof(int) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(indexbb_d, index_h, sizeof(int) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(spin_d, spin_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(love_d, love_h, sizeof(double3) * NconstT, hipMemcpyHostToDevice);
	if(P.UseMigrationForce > 0){
		hipMemcpy(migration_d, migration_h, sizeof(double3) * NconstT, hipMemcpyHostToDevice);
	}
	hipMemcpy(N_d, N_h, Nst * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Nencpairs_d, Nencpairs_h, (Nst + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Nencpairs2_d, Nencpairs2_h, (Nst + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Nencpairs3_d, Nencpairs3_h, P.SLevels * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Coll_d, Coll_h, sizeof(double) * Nst * def_NColl * def_MaxColl, hipMemcpyHostToDevice);
	hipMemcpy(writeEnc_d, writeEnc_h, sizeof(double) * Nst * def_NColl * def_MaxWriteEnc, hipMemcpyHostToDevice);
	hipMemcpy(Fragments_d, Fragments_h, sizeof(double) * Nst * 25 * P.Nfragments, hipMemcpyHostToDevice);
	hipMemcpy(aelimits_d, aelimits_h, sizeof(float4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(aecount_d, aecount_h, sizeof(unsigned int) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(enccount_d, enccount_h, sizeof(unsigned int) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(aecountT_d, aecountT_h, sizeof(unsigned long long) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(enccountT_d, enccountT_h, sizeof(unsigned long long) * NconstT, hipMemcpyHostToDevice);

	hipMemcpy(Nsmall_d, Nsmall_h, Nst * sizeof(int), hipMemcpyHostToDevice);

#if def_CPU == 1
	memcpy(x4b_h, x4_h, sizeof(double4) * NconstT);
	memcpy(v4b_h, v4_h, sizeof(double4) * NconstT);
	memcpy(x4bb_h, x4_h, sizeof(double4) * NconstT);
	memcpy(v4bb_h, v4_h, sizeof(double4) * NconstT);
	memcpy(xold_h, x4_h, sizeof(double4) * NconstT);
	memcpy(vold_h, v4_h, sizeof(double4) * NconstT);
	memcpy(rcritv_h, rcrit_h, sizeof(double) * NconstT * P.SLevels);
	memcpy(rcritb_h, rcrit_h, sizeof(double) * NconstT * P.SLevels);
	memcpy(rcritvb_h, rcrit_h, sizeof(double) * NconstT * P.SLevels);
	memcpy(rcritbb_h, rcrit_h, sizeof(double) * NconstT * P.SLevels);
	memcpy(rcritvbb_h, rcrit_h, sizeof(double) * NconstT * P.SLevels);
	memcpy(indexb_h, index_h, sizeof(int) * NconstT);
	memcpy(indexbb_h, index_h, sizeof(int) * NconstT);

#endif


#if def_TTV > 0
	hipMemcpy(elementsA_d, elementsA_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsB_d, elementsB_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsT_d, elementsT_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsSpin_d, elementsSpin_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsAOld_d, elementsA_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsAOld2_d, elementsA_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsBOld_d, elementsB_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsBOld2_d, elementsB_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsTOld_d, elementsT_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsTOld2_d, elementsT_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsSpinOld_d, elementsSpin_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsSpinOld2_d, elementsSpin_h, sizeof(double4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsL_d, elementsL_h, sizeof(elements10) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsC_d, elementsC_h, sizeof(int2) * (Nst + MCMC_NT), hipMemcpyHostToDevice);
	hipMemcpy(elementsSA_d, elementsSA_h, sizeof(double) * Nst, hipMemcpyHostToDevice);
	hipMemcpy(elementsP_d, elementsP_h, sizeof(double4) * Nst, hipMemcpyHostToDevice);
	hipMemcpy(elementsI_d, elementsI_h, sizeof(int4) * NconstT, hipMemcpyHostToDevice);
	hipMemcpy(elementsM_d, elementsM_h, sizeof(double) * Nst, hipMemcpyHostToDevice);

  #if MCMC_BLOCK == 5
	hipMemset(elementsG_d, 0, NconstT * sizeof(elements8));
	hipMemset(elementsGh_d, 0, NconstT * sizeof(elements8));
	hipMemset(elementsD_d, 0, NconstT * sizeof(elements8));
	hipMemset(elementsMean_d, 0, NconstT * sizeof(elements8));
	hipMemset(elementsVar_d, 0, NconstT * sizeof(elements8));
  #endif
  #if MCMC_BLOCK == 7
	hipMemset(elementsStep_d, 0, NconstT * sizeof(elementsS));
	hipMemset(elementsHist_d, 0, (Nst + N_h[0] * P.mcmcNE) / (N_h[0] * P.mcmcNE + 1) * N_h[0] * MCMC_NH * sizeof(elementsH));
  #endif
  #if MCMC_NCOV > 0
	hipMemcpy(elementsCOV_d, elementsCOV_h, sizeof(double) * NconstT * N_h[0] * MCMC_NCOV * MCMC_NCOV, hipMemcpyHostToDevice);
  #endif
#endif

	hipError_t error;

	hipMemcpy(NBS_d, NBS_h, Nst * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ict_d, ict_h, Nst * sizeof(double), hipMemcpyHostToDevice);
	error = hipGetLastError();
	fprintf(masterfile,"cudaMemcopy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("cudaMemcopy error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}

	return 1;
}

// ************************************** //
//This function reads the initial conditions from the IC file.
//Authors: Simon Grimm, Joachim Stadel
//March 2014
// *****************************************
__host__ int Data::readic(int st){

	int N = N_h[st];
	int Nsmall = Nsmall_h[st];
	int NBS = NBS_h[st];

	FILE *infile;	

	double AU = def_AU * 100.0; // in cm
	double Solarmass = def_Solarmass * 1000.0; //in g
	if(P.mcmcRestart == 0){
		if(P.FormatP == 1 || P.tRestart == 0){
			if(P.OutBinary > 0 && P.tRestart > 0){
				infile = fopen(GSF[st].inputfilename, "rb");
			}
			else{
				infile = fopen(GSF[st].inputfilename, "r");
			}
		}
		else{
			infile = NULL;
		}
//		printf("Read file %s %d %d\n", GSF[st].inputfilename, N, Nsmall);
	}
	else{
		if(st == 0){
			MCMCRestartFile = fopen("MCMCR.dat", "r");
			printf("Use MCMCR.dat file\n");
			if(MCMCRestartFile == NULL){
				printf("Error, file MCMCR.dat does not exist, needed for mcmc restart options");
				return 0;
			}
		}
		infile = MCMCRestartFile;
	}


	int ii = 0;
	int iismall = 0;
	MaxIndex = 0;
	
	double skip, test;
	double4 x, v;
	double rcrit;
	double4 spin;
	double3 love;
	double3 migration;
	int index;
	float4 aelimits;
	unsigned long long enccountT;
	double mJ = 0.0;	//Jacoby mass
	if(P.tRestart == 0 || def_TTV > 0){
		int er = 0;
		for(int i = 0; i < N + Nsmall; ++i){
			x = x4_h[i + NBS];
			v = v4_h[i + NBS];
			rcrit = rcrit_h[i + NBS];
			spin = spin_h[i + NBS];
			love = love_h[i + NBS];
			if(P.UseMigrationForce > 0){
				migration = migration_h[i + NBS];
			}
			test = test_h[i + NBS];
			enccountT = enccountT_h[i + NBS];
			//index = index_h[i + NBS];
			index = i + st * def_MaxIndex;
			aelimits = aelimits_h[i + NBS];
			int keplerian = 0;
			int convertPToA = 0;
			double p = 0.0;
			int convertTToM = 0;
			double T = 0.0;
			int kepCheck = 0;
			int cartCheck = 0;
#if def_TTV > 0
			double4 elementsA = elementsA_h[i + NBS];
			double4 elementsB = elementsB_h[i + NBS];
			double4 elementsT = elementsT_h[i + NBS];
			double4 elementsSpin = elementsSpin_h[i + NBS];
			elements10  elementsL = elementsL_h[i + NBS];
			double elementsSA = elementsSA_h[st];
			double4 elementsP = elementsP_h[st];
#endif

			for(int f = 0; f < def_Ninformat; ++f){
				if(GSF[st].informat[f] == 1){
					//x
					er = fscanf (infile, "%lf",&x.x);
					cartCheck += 1;
				}
				if (GSF[st].informat[f] == 2){
					//y
					er = fscanf (infile, "%lf",&x.y);
					cartCheck += 2;
				}
				if (GSF[st].informat[f] == 3){
					//z
					er = fscanf (infile, "%lf",&x.z);
					cartCheck += 4;
				}
				if (GSF[st].informat[f] == 4){
					//m
					er = fscanf (infile, "%lf",&x.w);
				}
				if (GSF[st].informat[f] == 5){
					//vx
					er = fscanf (infile, "%lf",&v.x);
					cartCheck += 8;
				}
				if (GSF[st].informat[f] == 6){
					//vy
					er = fscanf (infile, "%lf",&v.y);
					cartCheck += 16;
				}
				if (GSF[st].informat[f] == 7){
					//vz
					er = fscanf (infile, "%lf",&v.z);
					cartCheck += 32;
				}
				if (GSF[st].informat[f] == 8){
					//r
					er = fscanf (infile, "%lf",&v.w);
				}
				if (GSF[st].informat[f] == 9){
					//default rho
					er = fscanf (infile, "%lf",&rho[st]);
				}
				if (GSF[st].informat[f] == 10){
					//Sx
					er = fscanf (infile, "%lf",&spin.x);
#if def_TTV > 0
					elementsSpin.y = spin.x;
#endif
				}
				if (GSF[st].informat[f] == 11){
					//Sy
					er = fscanf (infile, "%lf",&spin.y);
#if def_TTV > 0
					elementsSpin.y = spin.y;
#endif
				}
				if (GSF[st].informat[f] == 12){
					//Sz
					er = fscanf (infile, "%lf",&spin.z);
#if def_TTV > 0
					elementsSpin.y = spin.z;
#endif
				}
				if (GSF[st].informat[f] == 13){
					//index
					er = fscanf (infile, "%d",&index);
				}
				if (GSF[st].informat[f] == 14){
					er = fscanf (infile, "%lf",&skip);
				}
				if (GSF[st].informat[f] == 15) er = fscanf (infile, "%f",&aelimits.x);	//amin
				if (GSF[st].informat[f] == 16) er = fscanf (infile, "%f",&aelimits.y);
				if (GSF[st].informat[f] == 17) er = fscanf (infile, "%f",&aelimits.z);
				if (GSF[st].informat[f] == 18) er = fscanf (infile, "%f",&aelimits.w);
				if (GSF[st].informat[f] == 19){
					if(ict_h[st] == 0){
						er = fscanf (infile, "%lf",&ict_h[st]);
					}
					else{
						er = fscanf (infile, "%lf",&skip);
					}
				}
				if (GSF[st].informat[f] == 20) er = fscanf (infile, "%lf",&love.x);
				if (GSF[st].informat[f] == 21) er = fscanf (infile, "%lf",&love.y);
				if (GSF[st].informat[f] == 22) er = fscanf (infile, "%lf",&love.z);
				if (GSF[st].informat[f] == 23){
					//a
					er = fscanf (infile, "%lf",&x.x);
#if def_TTV > 0
					elementsA.x = x.x;
#endif
					keplerian = 1;
					kepCheck += 1;
				}
				if (GSF[st].informat[f] == 24){
					//e
					er = fscanf (infile, "%lf",&x.y);
#if def_TTV > 0
					elementsA.y = x.y;
#endif
					keplerian = 1;
					kepCheck += 2;
				}
				if (GSF[st].informat[f] == 25){
					//inc
					er = fscanf (infile, "%lf",&x.z);
					if(P.AngleUnits == 1) x.z = x.z / 180.0 * M_PI;
#if def_TTV > 0
					elementsA.z = x.z;
#endif
					keplerian = 1;
					kepCheck += 4;
				}
				if (GSF[st].informat[f] == 26){
					//Omega
					er = fscanf (infile, "%lf",&v.x);
					if(P.AngleUnits == 1) v.x = v.x / 180.0 * M_PI;
#if def_TTV > 0
					elementsB.x = v.x;
#endif
					keplerian = 1;
					kepCheck += 8;
				}
				if (GSF[st].informat[f] == 27){
					//w
					er = fscanf (infile, "%lf",&v.y);
					if(P.AngleUnits == 1) v.y = v.y / 180.0 * M_PI;
#if def_TTV > 0
					elementsB.y = v.y;
#endif
					keplerian = 1;
					kepCheck += 16;
				}
				if (GSF[st].informat[f] == 28){
					//M
					er = fscanf (infile, "%lf",&v.z);
					if(P.AngleUnits == 1) v.z = v.z / 180.0 * M_PI;
#if def_TTV > 0
					elementsB.z = v.z;
#endif
					keplerian = 1;
					kepCheck += 32;
				}
				if (GSF[st].informat[f] == 38){
					//P
					er = fscanf (infile, "%lf",&p);
#if def_TTV > 0
					elementsT.z = p;
					elementsT.w = 0.0;
#endif
					keplerian = 1;
					convertPToA = 1;
					kepCheck += 1;
				}
				if (GSF[st].informat[f] == 40){
					//T
					er = fscanf (infile, "%lf",&T);
#if def_TTV > 0
					elementsT.x = T;
					elementsT.y = 0.0;
#endif
					keplerian = 1;
					convertTToM = 1;
					kepCheck += 32;
				}
				if (GSF[st].informat[f] == 42){
					//Rcrit
					er = fscanf (infile, "%lf",&rcrit);
				}
				if (GSF[st].informat[f] == 44){
					//Ic
					er = fscanf (infile, "%lf",&spin.w);
				}
				if (GSF[st].informat[f] == 45){
					//test
					er = fscanf (infile, "%lf",&test);
				}
				if (GSF[st].informat[f] == 46){
					//encc
					er = fscanf (infile, "%llu",&enccountT);
				}
#if def_TTV > 0
				if (GSF[st].informat[f] == 29){
					er = fscanf (infile, "%lf",&elementsL.a);	//aL
				}
				if (GSF[st].informat[f] == 30){
					er = fscanf (infile, "%lf",&elementsL.e);	//eL
				}
				if (GSF[st].informat[f] == 31){
					er = fscanf (infile, "%lf",&elementsL.inc);	//incL
					if(P.AngleUnits == 1) elementsL.inc = elementsL.inc / 180.0 * M_PI;
				}
				if (GSF[st].informat[f] == 32){
					 er = fscanf (infile, "%lf",&elementsL.m);	//mL
				}
				if (GSF[st].informat[f] == 33){
					er = fscanf (infile, "%lf",&elementsL.O);	//OmegaL
					if(P.AngleUnits == 1) elementsL.O = elementsL.O / 180.0 * M_PI;
				}
				if (GSF[st].informat[f] == 34){
					er = fscanf (infile, "%lf",&elementsL.w);	//wL
					if(P.AngleUnits == 1) elementsL.w = elementsL.w / 180.0 * M_PI;
				}
				if (GSF[st].informat[f] == 35){
					er = fscanf (infile, "%lf",&elementsL.M);	//ML
					if(P.AngleUnits == 1) elementsL.M = elementsL.M / 180.0 * M_PI;
				}
				if (GSF[st].informat[f] == 36){
					er = fscanf (infile, "%lf",&elementsL.r);	//rL
				}
				if (GSF[st].informat[f] == 37){
					er = fscanf (infile, "%lf",&elementsSA);	//SAT
				}
				if (GSF[st].informat[f] == 39){
					er = fscanf (infile, "%lf",&elementsL.P);	//PL
				}
				if (GSF[st].informat[f] == 41){
					er = fscanf (infile, "%lf",&elementsL.T);	//TL
				}
				if (GSF[st].informat[f] == 43){
					er = fscanf (infile, "%lf",&elementsP.w);	//gw
				}

#else
				if (GSF[st].informat[f] == 29) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 30) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 31) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 32) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 33) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 34) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 35) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 36) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 37) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 39) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 41) er = fscanf (infile, "%lf",&skip);
				if (GSF[st].informat[f] == 43) er = fscanf (infile, "%lf",&skip);
#endif
				if(P.UseMigrationForce > 0){	
					if (GSF[st].informat[f] == 49) er = fscanf (infile, "%lf",&migration.x);
					if (GSF[st].informat[f] == 50) er = fscanf (infile, "%lf",&migration.y);
					if (GSF[st].informat[f] == 51) er = fscanf (infile, "%lf",&migration.z);
				}
				else{
					if (GSF[st].informat[f] == 49) er = fscanf (infile, "%lf",&skip);
					if (GSF[st].informat[f] == 50) er = fscanf (infile, "%lf",&skip);
					if (GSF[st].informat[f] == 51) er = fscanf (infile, "%lf",&skip);

				}
				if (GSF[st].informat[f] == 0){
				}
				if (GSF[st].informat[f] < 0 || GSF[st].informat[f] > 51){
					printf("Error, initial condition file format is not valid, %d\n", GSF[st].informat[f]);
					return 0;
				}
			}

			if(dayUnit == 1) x.w *= def_Kg;
			if(convertPToA == 1){
				mJ += x.w;
				double mu = def_ksq * (Msun_h[st].x + mJ);

				volatile double a3 = p * p * dayUnit * dayUnit * mu / (4.0 * M_PI * M_PI);
				double a = cbrt(a3);
				x.x = a;
#if def_TTV > 0
				elementsA.x = a;
//printf("read a %d %.30g %.30g %.30g %.30g %.30g\n", i, p, mu, a, p * p * dayUnit * dayUnit * mu, a3);
#endif
			}
			else{
				// a to p
				mJ += x.w;
				double mu = def_ksq * (Msun_h[st].x + mJ);
				double p2 = x.x * x.x * x.x * 4.0 * M_PI * M_PI / (dayUnit * dayUnit * mu);
				p = sqrt(p2);
				//double a = x.x;
#if def_TTV > 0
				elementsT.z = p;
				elementsT.w = 0.0;
#endif
//printf("read p %d %.30g %.30g %.30g %.30g\n", i, p, mu, a, p2);

			}

			if(convertTToM == 1){

				double w = v.y;
				double e = x.y;

				double nu = M_PI * 0.5 - w;	//true anomaly at first transit
				double ee2 = e * e;
				double ee4 = ee2 * ee2;
				//double time = time_h[0] - dt_h[0] / dayUnit;
				double time = ict_h[0] * 365.25;
				//compute Mean Anomaly of the first transit
				double Mt = nu - 2.0 * e * sin(nu) + (3.0 * 0.25 * ee2 + 0.125 * ee4) * sin(2.0 * nu) - 1.0 / 3.0 * e * ee2 * sin(3.0 * nu) + 5.0/32.0 * ee4 * sin(4.0 * nu);
				double M = -(T - time) / p * 2.0 * M_PI + Mt;
//printf("T to M %.20g %.20g %.20g %.20g %.20g %.20g\n", time, nu, Mt, T, p, M);
				M = fmod(M, 2.0 * M_PI);
				if(M < 0.0) M += 2.0 * M_PI;

				v.z = M;
#if def_TTV > 0
				elementsB.z = M;
#endif
			}
			else{
#if def_TTV > 0

				double w = v.y;
				double e = x.y;
				double M = v.z;

				double nu = M_PI * 0.5 - w;	//true anomaly at first transit
				double ee2 = e * e;
				double ee4 = ee2 * ee2;
				//double time = time_h[0] - dt_h[0] / dayUnit;
				double time = ict_h[0] * 365.25;
				//compute Mean Anomaly of the first transit
				double Mt = nu - 2.0 * e * sin(nu) + (3.0 * 0.25 * ee2 + 0.125 * ee4) * sin(2.0 * nu) - 1.0 / 3.0 * e * ee2 * sin(3.0 * nu) + 5.0/32.0 * ee4 * sin(4.0 * nu);

//printf("M to T %g %g %g %g %g %g\n", M, time, nu, Mt, T, p);

				double T = -(M - Mt) / (2.0 * M_PI) * p + time;
				elementsT.x = T;
				elementsT.y = 0.0;
#endif
			}
			if(keplerian == 0){
				if(cartCheck != 63 || kepCheck != 0){
					printf("Error, initial conditions are not complete. Must include x, y, z, vx, vy, vz \n");
					return 0;
				}
			}
			if(keplerian == 1){
				if(kepCheck != 63 || cartCheck != 0){
					printf("Error, initial conditions are not complete. Must include a (or P), e, inc, O, w, M (or T)\n");
					return 0;
				}

#if def_TTV > 0
				elementsA.w = x.w;		//m
				elementsB.w = v.w;		//r
//printf("read elements %d %.20g %.20g %.20g\n",ii, elementsA.w, elementsA.x, elementsA.y); 
#endif	
				KepToCart(x, v, Msun_h[st].x);
			}
			if(index < 0) index *= -1;
			if(v.w == 0){
				v.w = cbrt((x.w * 0.75 ) / (M_PI * rho[st] * AU * AU * AU / Solarmass));
			}

			//avoid max for long long int
			if(index > MaxIndex){
				MaxIndex = index;
			}
			
			int NBSN = NBS;
			if(x.w >= 0.0 && x.w <= P.MinMass && P.UseTestParticles > 0){
				NBSN += N - ii + iismall; //shift test particles to the end of the arrays
			}
			else{
				NBSN -= iismall;
			}

			x4_h[ii + NBSN] = x;
			v4_h[ii + NBSN] = v;
			rcrit_h[ii + NBSN] = rcrit;
			spin_h[ii + NBSN] = spin;
			love_h[ii + NBSN] = love;
			if(P.UseMigrationForce > 0){
				migration_h[ii + NBSN] = migration;
			}
			if(Nst == 1) index_h[ii + NBSN] = index;
			else index_h[ii + NBSN] = index % def_MaxIndex + def_MaxIndex * st;
			aelimits_h[ii + NBSN] = aelimits;
			enccountT_h[ii + NBSN] = enccountT;
			test_h[ii + NBSN] = test;
#if def_TTV > 0
			elementsA_h[ii + NBSN] = elementsA;
			elementsB_h[ii + NBSN] = elementsB;
			elementsT_h[ii + NBSN] = elementsT;
			elementsSpin_h[ii + NBSN] = elementsSpin;
			elementsL_h[ii + NBSN] = elementsL;
			int iT = st / (Nst / MCMC_NT);			//index of temperature in parallel tempering
			 
			elementsSA_h[st] = elementsSA * pow(sqrt(2.0), iT);
			elementsP_h[st] = elementsP;
#endif
			++ii;
			if(x.w >= 0 && x.w <= P.MinMass && P.UseTestParticles > 0) ++iismall;
		}// end of particle loop
		//check now if the file is finished
		if(def_TTV == 0){
			er = fscanf (infile, "%lf",&skip);
			if(er != -1 && st == Nst -1){
				printf("Error, initial condition file format is not correct\n");
				return 0;
			}
		}

	}
	else{
#if def_TTV > 0
		printf("Restart for TTV not possible\n");
		return 0;

#endif
		//read from restart time step
		double Et;
		if(P.OutBinary == 0){
			char Ets[160]; //exact time at restart time step, must be the same format as the coordinate output
			sprintf(Ets, "%.16g", (P.tRestart * idt_h[st] + ict_h[st] * 365.25) / 365.25);
			Et = atof(Ets);
		}
		else{
			Et = (P.tRestart * idt_h[st] + ict_h[st] * 365.25) / 365.25;
		}

		double time = 0.0;
		double aecountf = 0.0;
		double aecountTf = 0.0;
		unsigned int aecount;
		unsigned long long aecountT;
		unsigned long long enccountT;

		spin.x = 0.0;
		spin.y = 0.0;
		spin.z = 0.0;
		spin.w = 0.4;
		love.x = 0.0;
		love.y = 0.0;
		love.z = 0.0;
		migration.x = 0.0;
		migration.y = 0.0;
		migration.z = 0.0;
		rcrit = 0.0;

		if(P.FormatP == 1){

			//skip previous time steps
			if(P.FormatT == 0){
				readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("T0 %d %d %g %g | %d %g %g\n", st, 0, time, Et, index, x.w, x.x);
			}
			if(P.FormatT == 1){
				readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
				while((time < Et && idt_h[st] > 0) || (time > Et && idt_h[st] < 0)){
					if(time == Et) break;
					int er = readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("T1 %d %d %g %g | %d %g %g\n", st, 0, time, Et, index, x.w, x.x);
					if(er <= 0){
						break;
					}
				}
			}


			//skip previous simulation data
			if(P.FormatS == 1){
				for(int i = 0; i < NBS; ++i){
					readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("S %d %d %g %g | %d %g %g\n", st, i, time, Et, index, x.w, x.x);
				}
			}

			int iismall = 0;
			for(int i = 0; i < N + Nsmall; ++i){
				if(i > 0) readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("r %d %d %g %g | %d %g %g\n", st, i, time, Et, index, x.w, x.x);

				if(P.FormatS == 0) index += def_MaxIndex * st;
				aecount = (unsigned int)(aecountf * P.ci);
				unsigned long long tt = P.tRestart - P.tRestart % P.ci;
				aecountT = (unsigned long long)(aecountTf * tt);

				//avoid max for long long int
				if(index > MaxIndex){
					MaxIndex = index;
				}

				int NBSN = NBS;
				if(x.w >= 0.0 && x.w <= P.MinMass && P.UseTestParticles > 0){
					NBSN += N - i + iismall; //shift test particles to the end of the arrays
				}
				else{
					NBSN -= iismall;
				}
				index_h[ii + NBSN] = index;
				x4_h[ii + NBSN] = x;
				v4_h[ii + NBSN] = v;
				rcrit_h[ii + NBSN] = rcrit;
				spin_h[ii + NBSN] = spin;
				love_h[ii + NBSN] = love;
				if(P.UseMigrationForce > 0){
					migration_h[ii + NBSN] = migration;
				}
				aelimits_h[ii + NBSN] = aelimits;
				enccountT_h[ii + NBSN] = enccountT;
				aecount_h[ii + NBSN] = aecount;
				aecountT_h[ii + NBSN] = aecountT;
				test_h[ii + NBSN] = test;
				++ii;
				if(x.w >= 0 && x.w <= P.MinMass && P.UseTestParticles > 0) ++iismall;
			}
		}
		if(P.FormatP == 0){
			ii = 0;
			FILE *OrigInfile;	
			char Origfilename[512];
			sprintf(Origfilename, "%s%s", GSF[st].path, GSF[st].Originputfilename);
			OrigInfile = fopen(Origfilename, "r");

			FILE *fragmentsfile;
			if(P.UseSmallCollisions > 0 || P.CreateParticles > 0){
				fragmentsfile = fopen(GSF[st].fragmentfilename, "r");
			}
			else{
				fragmentsfile = NULL;
			}

			int iismall = 0;
			int index;

			for(int k = 0; k < 1000000000; ++k){
				int i = k;
				double skip = 0.0;
				int eri = 1;
				for(int f = 0; f < def_Ninformat; ++f){
					if(GSF[st].informat[f] == 13){
						eri = fscanf (OrigInfile, "%d",&i);
					}
					else if(GSF[st].informat[f] > 0){
						eri = fscanf (OrigInfile, "%lf",&skip);
					}
				}
				if(eri < 0){
					if(P.UseSmallCollisions > 0 || P.CreateParticles > 0){
//printf("Search for fragments %s\n", GSF[st].fragmentfilename);
						double ttime, mm;
						double skip;
						eri = fscanf(fragmentsfile, "%lf", &ttime);
						eri = fscanf(fragmentsfile, "%d", &i);
						eri = fscanf(fragmentsfile, "%lf", &mm);

						for(int jj = 0; jj < 11; ++jj){
							eri = fscanf(fragmentsfile, "%lf", &skip);
						}
						if(eri <= 0){
							break;
						}

						if(ttime > Et) continue;
					}
					else{
						break;
					}
				}
		
				int er = 0;
				char infilename[384];
				if(P.OutBinary == 0){
					sprintf(infilename, "%sOut%s_p%.6d.dat", GSF[st].path, GSF[st].X, i);
					infile = fopen(infilename, "r");
				}
				else{
					sprintf(infilename, "%sOut%s_p%.6d.bin", GSF[st].path, GSF[st].X, i);
					infile = fopen(infilename, "rb");
				}
//printf("Read file %s %d %d %d\n", infilename, ii, N, Nsmall);
				if(infile == NULL) continue;
	
				//skip previous time steps
				er = readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("T0 %d %d %g %g | %d %g %g\n", st, 0, time, Et, index, x.w, x.x);
				while((time < Et && idt_h[st] > 0) || (time > Et && idt_h[st] < 0)){
					if(time == Et){
						break;
					}
					er = readOutLine(time, index, x, v, spin, love, migration, aelimits, aecountf, aecountTf, enccountT, rcrit, test, infile, st);
//printf("T1 %d %d %g %g | %d %g %g\n", st, 0, time, Et, index, x.w, x.x);
					if(er <= 0){
						break;
					}
				}
				if(er <= 0){
					continue;
				}

				if(P.FormatS == 0) index += def_MaxIndex * st;
				aecount = (unsigned int)(aecountf * P.ci);
				unsigned long long tt = P.tRestart - P.tRestart % P.ci;
				aecountT = (unsigned long long)(aecountTf * tt);

				//avoid max for long long int
				if(index > MaxIndex){
					MaxIndex = index;
				}

				int NBSN = NBS;
				if(x.w >= 0.0 && x.w <= P.MinMass && P.UseTestParticles > 0){
					NBSN += N - ii + iismall; //shift test particles to the end of the arrays
				}
				else{
					NBSN -= iismall;
				}
				index_h[ii + NBSN] = index;
				x4_h[ii + NBSN] = x;
				v4_h[ii + NBSN] = v;
				rcrit_h[ii + NBSN] = rcrit;
				spin_h[ii + NBSN] = spin;
				love_h[ii + NBSN] = love;
				if(P.UseMigrationForce > 0){
					migration_h[ii + NBSN] = migration;
				}
				aelimits_h[ii + NBSN] = aelimits;
				enccountT_h[ii + NBSN] = enccountT;
				aecount_h[ii + NBSN] = aecount;
				aecountT_h[ii + NBSN] = aecountT;
				test_h[ii + NBSN] = test;

				++ii;
				if(x.w >= 0 && x.w <= P.MinMass && P.UseTestParticles > 0) ++iismall;

				fclose(infile);
//printf("%d %d %d\n", ii, iismall, N + Nsmall);
				if(ii == N + Nsmall) break;
			}
			fclose(OrigInfile);
			if(P.UseSmallCollisions > 0 || P.CreateParticles > 0){
				fclose(fragmentsfile);
			}

		}
	}
	if(P.mcmcRestart == 0){
		if(P.FormatP == 1 || P.tRestart == 0) fclose(infile);
	}
	else{
		if(st == Nst - 1){
			fclose(infile);
		}
	}
	return ii;
} 


// *************************************
//This function converts Keplerian Elements into Cartesian Coordinates
__host__ void Data::KepToCart(double4 &x, double4 &v, double Msun){

	double a = x.x;
	double e = x.y;
	double inc = x.z;
	double Omega = v.x;
	double w = v.y;
	double M = v.z;
//printf("A KtoC m:%g r:%g a:%g e:%g i:%g O:%g w:%g M:%g\n", x.w, v.w, x.x, x.y, x.z, v.x ,v.y, v.z);

	double mu = def_ksq * (Msun + x.w);

	double E;
	if(e < 1.0 - 1.0e-10){	
		//Eccentric Anomaly
		E = M + e * 0.5;
		double Eold = E;
		for(int j = 0; j < 32; ++j){
			E = E - (E - e * sin(E) - M) / (1.0 - e * cos(E));
			if(fabs(E - Eold) < 1.0e-15) break;
			Eold = E;
		}
	}
	else if(e > 1.0 + 1.0e-10){
		//hyperbolic
		//E is assumed to be the hyperbolic eccentricity 
		E = M;
		double Eold = E;
		for(int j = 0; j < 32; ++j){
			E = E + (E - e * sinh(E) + M) / (e * cosh(E) - 1.0);
			if(fabs(E - Eold) < 1.0e-15) break;
			Eold = E;
		}

	}
	else{
		//parabolic, solve Barkers equation 
		// M = D + D^3 / 3, 
		// use cot(s) = 1.5 * M  -> s = pi / 2 - atan(1.5 * M)

		//double s = M_PI * 0.5 - atan(1.5 * M);
		E = M;
		double Eold = E;
		for(int j = 0; j < 32; ++j){
			E = E - (E + E * E * E / 3.0 - M) / (1.0 + E * E);
			if(fabs(E - Eold) < 1.0e-15) break;
			Eold = E;
		}

	}


	double cw = cos(w);
	double sw = sin(w);
	double cOmega = cos(Omega);
	double sOmega = sin(Omega);
	double ci = cos(inc);
	double si = sin(inc);

	double Px = cw * cOmega - sw * ci * sOmega;
	double Py = cw * sOmega + sw * ci * cOmega;
	double Pz = sw * si;

	double Qx = -sw * cOmega - cw * ci * sOmega;
	double Qy = -sw * sOmega + cw * ci * cOmega;
	double Qz = cw * si;

	double cE = cos(E);
	double sE = sin(E);

	double t0, t1, t2;

	if(e < 1.0 - 1.0e-10){
		//elliptic

		//double r = a * ( 1.0 - e * cE);
		//double r = a * (1.0 - e*e)/(1.0 + e *cos(Theta));
		//double t1 = r * cos(Theta); 
		//double t2 = r * sin(Theta); 
		t1 = a * (cE - e);
		t2 = a * sqrt(1.0 - e * e) * sE;
	}
	else if(e > 1.0 + 1.0e-10){
		//hyperbolic
		//double r = a * (1.0 - e*e)/(1.0 + e *cos(Theta));
		//or
		//double r = a * ( 1.0 - e * cosh(E));
		//t1 = r * cos(Theta); 
		//t2 = r * sin(Theta); 
		t1 = a * (cosh(E) - e);
		t2 = -a * sqrt(e * e - 1.0) * sinh(E);
	}
	else{
		//parabolic
		// a is assumed to be q, p = 2q, p = h^2/mu
		double Theta = 2.0 * atan(E);
		double r = 2 * a /(1.0 + cos(Theta));
		t1 = r * cos(Theta);
		t2 = r * sin(Theta);
	}


	x.x = t1 * Px + t2 * Qx;
	x.y = t1 * Py + t2 * Qy;
	x.z = t1 * Pz + t2 * Qz;

	if(e < 1.0 - 1.0e-10){
		//elliptic
		t0 = 1.0 / (1.0 - e * cE) * sqrt(mu / a);
		t1 = -sE;
		t2 = sqrt(1.0 - e * e) * cE;
	}
	else if(e > 1.0 + 1.0e-10){
		//hyperbolic
		//double r = a * (1.0 - e*e)/(1.0 + e *cos(Theta));
		double r = a * ( 1.0 - e * cosh(E));
		t0 = sqrt(-mu * a) / r;
		t1 = -sinh(E);
		t2 = sqrt(e * e - 1.0) * cosh(E);
	}
	else{
		//parabolic
		double Theta = 2.0 * atan(E);
		t0 = mu / sqrt(2.0 * a * mu);
		t1 = -sin(Theta);
		t2 = 1.0 + cos(Theta);
	}



	v.x = t0 * (t1 * Px + t2 * Qx);
	v.y = t0 * (t1 * Py + t2 * Qy);
	v.z = t0 * (t1 * Pz + t2 * Qz);
//printf("B KtoC m:%g r:%g x:%g y:%g z:%g vx:%g vy:%g vz:%g\n", x.w, v.w, x.x, x.y, x.z, v.x ,v.y, v.z);
}

// **************************************
//This function converts heliocentric coordinates to democratic coordinates.
__host__ void Data::HelioToDemo(double4 *x4_h, double4 *v4_h, double Msun, int N){

	double mtot = 0.0;
	double3 vcom;
	vcom.x = 0.0;
	vcom.y = 0.0;
	vcom.z = 0.0;
	
	for(int i = 0; i < N; ++i){
		if(x4_h[i].w > 0.0){
			double m = x4_h[i].w;
			mtot += m;
			vcom.x += m * v4_h[i].x;
			vcom.y += m * v4_h[i].y;
			vcom.z += m * v4_h[i].z;
		}
	}
	mtot += Msun;
	vcom.x /= mtot;
	vcom.y /= mtot;
	vcom.z /= mtot;

	for(int i = 0; i < N; ++i){
		v4_h[i].x -= vcom.x;
		v4_h[i].y -= vcom.y;
		v4_h[i].z -= vcom.z;
	}
}
// This function converts heliocentric coordinates to barycentric coordinates.
// The zeroth body must be the central star here
__host__ void Data::HelioToBary(double4 *x4_h, double4 *v4_h, double Msun, int N){

	double mtot = 0.0;
	double3 vcom;
	double3 xcom;
	xcom.x = 0.0;
	xcom.y = 0.0;
	xcom.z = 0.0;
	vcom.x = 0.0;
	vcom.y = 0.0;
	vcom.z = 0.0;
	
	for(int i = 0; i < N; ++i){
//printf("A HtB %g %g %g %g %g %g %g %g\n", x4_h[i].w, v4_h[i].w, x4_h[i].x, x4_h[i].y, x4_h[i].z, v4_h[i].x ,v4_h[i].y, v4_h[i].z);
		if(x4_h[i].w > 0.0){
			double m = x4_h[i].w;
			mtot += m;
			xcom.x += m * x4_h[i].x;
			xcom.y += m * x4_h[i].y;
			xcom.z += m * x4_h[i].z;
			vcom.x += m * v4_h[i].x;
			vcom.y += m * v4_h[i].y;
			vcom.z += m * v4_h[i].z;
		}
	}
	xcom.x /= mtot;
	xcom.y /= mtot;
	xcom.z /= mtot;
	vcom.x /= mtot;
	vcom.y /= mtot;
	vcom.z /= mtot;

	for(int i = 0; i < N; ++i){
		x4_h[i].x -= xcom.x;
		x4_h[i].y -= xcom.y;
		x4_h[i].z -= xcom.z;
		v4_h[i].x -= vcom.x;
		v4_h[i].y -= vcom.y;
		v4_h[i].z -= vcom.z;
//printf("B HtB %g %g %g %g %g %g %g %g\n", x4_h[i].w, v4_h[i].w, x4_h[i].x, x4_h[i].y, x4_h[i].z, v4_h[i].x ,v4_h[i].y, v4_h[i].z);
	}
}
// **************************************
//This function converts democratic coordinates to heliocentric coordinates.
__host__ void Data::DemoToHelio(double4 *x4_h, double4 *v4_h, double4 *v4Helio_h, double Msun, int N){

	double3 vcom;
	vcom.x = 0.0;
	vcom.y = 0.0;
	vcom.z = 0.0;

	for(int i = 0; i < N; ++i){
		if(x4_h[i].w > 0.0){
			vcom.x += x4_h[i].w * v4_h[i].x;
			vcom.y += x4_h[i].w * v4_h[i].y;
			vcom.z += x4_h[i].w * v4_h[i].z;
		}
	}
	vcom.x /= Msun;
	vcom.y /= Msun;
	vcom.z /= Msun;

	for(int i = 0; i < N; ++i){
		v4Helio_h[i].x = v4_h[i].x + vcom.x;
		v4Helio_h[i].y = v4_h[i].y + vcom.y;
		v4Helio_h[i].z = v4_h[i].z + vcom.z;
		v4Helio_h[i].w = v4_h[i].w;
	}

}
// **************************************
//This function converts barycentric coordinates to heliocentric coordinates.
// The zeroth body must be the cetnral star here
__host__ void Data::BaryToHelio(double4 *x4_h, double4 *v4_h, double Msun, int N){

	double3 xcom;
	double3 vcom;
	xcom.x = 0.0;
	xcom.y = 0.0;
	xcom.z = 0.0;
	vcom.x = 0.0;
	vcom.y = 0.0;
	vcom.z = 0.0;

	for(int i = 0; i < N; ++i){
		if(x4_h[i].w > 0.0){
			xcom.x += x4_h[i].w * x4_h[i].x;
			xcom.y += x4_h[i].w * x4_h[i].y;
			xcom.z += x4_h[i].w * x4_h[i].z;
			vcom.x += x4_h[i].w * v4_h[i].x;
			vcom.y += x4_h[i].w * v4_h[i].y;
			vcom.z += x4_h[i].w * v4_h[i].z;
		}
	}
	xcom.x /= x4_h[0].w;
	xcom.y /= x4_h[0].w;
	xcom.z /= x4_h[0].w;
	vcom.x /= x4_h[0].w;
	vcom.y /= x4_h[0].w;
	vcom.z /= x4_h[0].w;

	for(int i = 0; i < N; ++i){
		x4_h[i].x += xcom.x;
		x4_h[i].y += xcom.y;
		x4_h[i].z += xcom.z;
		v4_h[i].x += vcom.x;
		v4_h[i].y += vcom.y;
		v4_h[i].z += vcom.z;
	}
}


// **************************************
//This kernel removes ghost-masses and decreases the number of bodies.
//It also removes bodies wich a semi major axis bigger than Rcut.
//It runs with only one thread ond the GPU, to avoid unnecesary data copies
//Authors: Simon Grimm, Joachim Stadel
//March 2014
// ***************************************
__global__ void remove_kernel(double4 *x4_d, double4 *v4_d, double3 *a_d, int *N_d, int *Nsmall_d, int *index_d, double4 *spin_d, double3 *love_d, double3 *migration_d, int *createFlag_d, double *test_d, double *EnergySum_d, double *rcrit_d, double *rcritv_d, int NBS, int st, float4 *aelimits_d, unsigned int *aecount_d, unsigned int *enccount_d, unsigned long long *aecountT_d, unsigned long long *enccountT_d, double *K_d, double *Kold_d, int NB, const int NconstT, const int SLevels, const int UseMigrationForce, const int CreateParticles, double *nafx_d, double *nafy_d, int nafn){
	int NOld;
	int NsmallOld;
	int N = N_d[st];
	int Nsmall = Nsmall_d[st];
	int f = 1;
	int fc = 0;

	while(f == 1 && fc < 100){
		NOld = N;
		NsmallOld = Nsmall;
		f = 0;
		++fc;
		for(int j = 0; j < N; ++j){
			//remove ghost bodies and rearrange arrays
			if(x4_d[j + NBS].w < 0){
				int Na = j + NBS;
				int Nb = N-1 + NBS;
				
				x4_d[Na] = x4_d[Nb];
				v4_d[Na] = v4_d[Nb];

				x4_d[Nb].x = 0.0;
				x4_d[Nb].y = 1.0;
				x4_d[Nb].z = 0.0;
				x4_d[Nb].w = -1.0e-12;
	
				v4_d[Nb].x = 0.0;
				v4_d[Nb].y = 0.0;
				v4_d[Nb].z = 0.0;
				v4_d[Nb].w = 0.0;

				a_d[Na] = a_d[Nb];
				a_d[Nb].x = 0.0;
				a_d[Nb].y = 0.0;
				a_d[Nb].z = 0.0;

				index_d[Na] = index_d[Nb];
				index_d[Nb] = -1;

				spin_d[Na] = spin_d[Nb];
				spin_d[Nb].x = 0.0;
				spin_d[Nb].y = 0.0;
				spin_d[Nb].z = 0.0;
				spin_d[Nb].w = 0.0;
	
				love_d[Na] = love_d[Nb];
				love_d[Nb].x = 0.0;
				love_d[Nb].y = 0.0;
				love_d[Nb].z = 0.0;

				if(UseMigrationForce > 0){
					migration_d[Na] = migration_d[Nb];
					migration_d[Nb].x = 0.0;
					migration_d[Nb].y = 0.0;
					migration_d[Nb].z = 0.0;
				}

				if(CreateParticles > 0){
					createFlag_d[Na] = createFlag_d[Nb];
					createFlag_d[Nb] = -1;
				}

				for(int l = 0; l < SLevels; ++l){
					rcrit_d[Na + l * NconstT] = rcrit_d[Nb + l * NconstT];
					rcritv_d[Na + l * NconstT] = rcritv_d[Nb + l * NconstT];
					rcrit_d[Nb + l * NconstT] = 0.0;
					rcritv_d[Nb + l * NconstT] = 0.0;
				}

				aelimits_d[Na] = aelimits_d[Nb];
				aelimits_d[Nb].x = 0.0f;
				aelimits_d[Nb].y = 0.0f;
				aelimits_d[Nb].z = 0.0f;	
				aelimits_d[Nb].w = 0.0f;

				aecount_d[Na] = aecount_d[Nb];
				aecount_d[Nb] = 0u;
				enccount_d[Na] = enccount_d[Nb];
				enccount_d[Nb] = 0u;
				aecountT_d[Na] = aecountT_d[Nb];
				aecountT_d[Nb] = 0ull;
				enccountT_d[Na] = enccountT_d[Nb];
				enccountT_d[Nb] = 0ull;

				test_d[Na] = test_d[Nb];
				test_d[Nb] = -1.0;

				EnergySum_d[Na] += EnergySum_d[Nb];
				EnergySum_d[Nb] = 0.0;

				for(int i = 0; i < nafn; ++i){
					nafx_d[(Na) * nafn + i] = nafx_d[(Nb) * nafn + i];
					nafy_d[(Na) * nafn + i] = nafy_d[(Nb) * nafn + i];
					nafx_d[(Nb) * nafn + i] = 0.0;
					nafy_d[(Nb) * nafn + i] = 0.0;
				}
#if def_G3 > 0
				for(int i = 0; i < N; ++i){
					K_d[(Na) * NB + i] = K_d[(Nb) * NB + i];
					K_d[i * NB + Na] = K_d[i * NB + (Nb)];
					K_d[(Nb) * NB + i] = 1.0;
					K_d[i * NB + (Nb)] = 1.0;
					Kold_d[(Na) * NB + i] = Kold_d[(Nb) * NB + i];
					Kold_d[i * NB + Na] = Kold_d[i * NB + (Nb)];
					Kold_d[(Nb) * NB + i] = 1.0;
					Kold_d[i * NB + (Nb)] = 1.0;
				}
#endif
				//move Test Particles
				if(Nsmall > 0){
					int Na = N-1 + NBS;
					int Nb = N-1 + NBS + Nsmall;
					
					x4_d[Na] = x4_d[Nb];
					v4_d[Na] = v4_d[Nb];

					x4_d[Nb].x = 0.0;
					x4_d[Nb].y = 1.0;
					x4_d[Nb].z = 0.0;
					x4_d[Nb].w = -1.0e-12;
		
					v4_d[Nb].x = 0.0;
					v4_d[Nb].y = 0.0;
					v4_d[Nb].z = 0.0;
					v4_d[Nb].w = 0.0;

					a_d[Na] = a_d[Nb];
					a_d[Nb].x = 0.0;
					a_d[Nb].y = 0.0;
					a_d[Nb].z = 0.0;

					index_d[Na] = index_d[Nb];
					index_d[Nb] = -1;

					spin_d[Na] = spin_d[Nb];
					spin_d[Nb].x = 0.0;
					spin_d[Nb].y = 0.0;
					spin_d[Nb].z = 0.0;
					spin_d[Nb].w = 0.0;

					love_d[Na] = love_d[Nb];
					love_d[Nb].x = 0.0;
					love_d[Nb].y = 0.0;
					love_d[Nb].z = 0.0;
	
					if(UseMigrationForce > 0){
						migration_d[Na] = migration_d[Nb];
						migration_d[Nb].x = 0.0;
						migration_d[Nb].y = 0.0;
						migration_d[Nb].z = 0.0;
					}

					if(CreateParticles > 0){
						createFlag_d[Na] = createFlag_d[Nb];
						createFlag_d[Nb] = 0;
					}
			
					for(int l = 0; l < SLevels; ++l){
						rcrit_d[Na + l * NconstT] = rcrit_d[Nb + l * NconstT];
						rcritv_d[Na + l * NconstT] = rcritv_d[Nb + l * NconstT];
						rcrit_d[Nb + l * NconstT] = 0.0;
						rcritv_d[Nb + l * NconstT] = 0.0;
					}

					aelimits_d[Na] = aelimits_d[Nb];
					aelimits_d[Nb].x = 0.0f;
					aelimits_d[Nb].y = 0.0f;
					aelimits_d[Nb].z = 0.0f;	
					aelimits_d[Nb].w = 0.0f;

					aecount_d[Na] = aecount_d[Nb];
					aecount_d[Nb] = 0u;
					enccount_d[Na] = enccount_d[Nb];
					enccount_d[Nb] = 0u;
					aecountT_d[Na] = aecountT_d[Nb];
					aecountT_d[Nb] = 0ull;
					enccountT_d[Na] = enccountT_d[Nb];
					enccountT_d[Nb] = 0ull;

					test_d[Na] = test_d[Nb];
					test_d[Nb] = -1.0;

					EnergySum_d[Na] += EnergySum_d[Nb];
					EnergySum_d[Nb] = 0.0;

					for(int i = 0; i < nafn; ++i){
						nafx_d[(Na) * nafn + i] = nafx_d[(Nb) * nafn + i];
						nafy_d[(Na) * nafn + i] = nafy_d[(Nb) * nafn + i];
						nafx_d[(Nb) * nafn + i] = 0.0;
						nafy_d[(Nb) * nafn + i] = 0.0;
					}
				}

				N -= 1;
			}
		}
		for(int j = N; j < N + Nsmall; ++j){
			//remove ghost test particles and rearrange arrays
			if(x4_d[j + NBS].w < 0){

				int Na = j + NBS;
				int Nb = N-1 + NBS + Nsmall;
				x4_d[Na] = x4_d[Nb];
				v4_d[Na] = v4_d[Nb];

				x4_d[Nb].x = 0.0;
				x4_d[Nb].y = 1.0;
				x4_d[Nb].z = 0.0;
				x4_d[Nb].w = -1.0e-12;
	
				v4_d[Nb].x = 0.0;
				v4_d[Nb].y = 0.0;
				v4_d[Nb].z = 0.0;
				v4_d[Nb].w = 0.0;

				a_d[Na] = a_d[Nb];
				a_d[Nb].x = 0.0;
				a_d[Nb].y = 0.0;
				a_d[Nb].z = 0.0;

				index_d[Na] = index_d[Nb];
				index_d[Nb] = -1;

				spin_d[Na] = spin_d[Nb];
				spin_d[Nb].x = 0.0;
				spin_d[Nb].y = 0.0;
				spin_d[Nb].z = 0.0;
				spin_d[Nb].w = 0.0;

				love_d[Na] = love_d[Nb];
				love_d[Nb].x = 0.0;
				love_d[Nb].y = 0.0;
				love_d[Nb].z = 0.0;

				if(UseMigrationForce > 0){
					migration_d[Na] = migration_d[Nb];
					migration_d[Nb].x = 0.0;
					migration_d[Nb].y = 0.0;
					migration_d[Nb].z = 0.0;
				}

				if(CreateParticles > 0){
					createFlag_d[Na] = createFlag_d[Nb];
					createFlag_d[Nb] = -1;
				}

				for(int l = 0; l < SLevels; ++l){
					rcrit_d[Na + l * NconstT] = rcrit_d[Nb + l * NconstT];
					rcritv_d[Na + l * NconstT] = rcritv_d[Nb + l * NconstT];
					rcrit_d[Nb + l * NconstT] = 0.0;
					rcritv_d[Nb + l * NconstT] = 0.0;
				}

				aelimits_d[Na] = aelimits_d[Nb];
				aelimits_d[Nb].x = 0.0f;
				aelimits_d[Nb].y = 0.0f;
				aelimits_d[Nb].z = 0.0f;	
				aelimits_d[Nb].w = 0.0f;

				aecount_d[Na] = aecount_d[Nb];
				aecount_d[Nb] = 0u;
				enccount_d[Na] = enccount_d[Nb];
				enccount_d[Nb] = 0u;
				aecountT_d[Na] = aecountT_d[Nb];
				aecountT_d[Nb] = 0ull;
				enccountT_d[Na] = enccountT_d[Nb];
				enccountT_d[Nb] = 0ull;

				test_d[Na] = test_d[Nb];
				test_d[Nb] = -1.0;

				EnergySum_d[Na] += EnergySum_d[Nb];
				EnergySum_d[Nb] = 0.0;

				for(int i = 0; i < nafn; ++i){
					nafx_d[(Na) * nafn + i] = nafx_d[(Nb) * nafn + i];
					nafy_d[(Na) * nafn + i] = nafy_d[(Nb) * nafn + i];
					nafx_d[(Nb) * nafn + i] = 0.0;
					nafy_d[(Nb) * nafn + i] = 0.0;
				}
				Nsmall -= 1;
			}
		}
		if(NOld != N) f = 1;
		if(NsmallOld != Nsmall) f = 1;
	}
	N_d[st] = N;
	Nsmall_d[st] = Nsmall;
}


// **************************************
//This function prints out data of ejected bodies
//It sets the masses of ejected bodies to zero, this are then later removed
//It Updates the lost Energy term U
//
//Authors: Simon Grimm, Joachim Stadel
//Mai 2015
//****************************************
__host__ void Data::Ejection(){

	FILE *ejectfile;
	FILE *logfile;

	if(Nst == 1) EjectionFlag_m[1] = 1;
	if(Nst > 1) hipMemcpy(time_h, time_d, Nst*sizeof(double), hipMemcpyDeviceToHost);

	for(int st = 0; st < Nst; ++st){
		if(EjectionFlag_m[st + 1] > 0){
			EjectionFlag_m[st + 1] = 0;

			int NBS = NBS_h[st];

			ejectfile = fopen(GSF[st].ejectfilename, "a");
			logfile = fopen(GSF[st].logfilename, "a");

			hipMemcpy(x4_h + NBS, x4_d + NBS, sizeof(double4) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			hipMemcpy(v4_h + NBS, v4_d + NBS, sizeof(double4) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			hipMemcpy(index_h + NBS, index_d + NBS, sizeof(int) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			hipMemcpy(spin_h + NBS, spin_d + NBS, sizeof(double4) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			hipMemcpy(love_h + NBS, love_d + NBS, sizeof(double3) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			if(P.UseMigrationForce > 0){
				hipMemcpy(migration_h + NBS, migration_d + NBS, sizeof(double3) * (N_h[st] + Nsmall_h[st]), hipMemcpyDeviceToHost);
			}

			hipMemset(Nencpairs_d, 0, sizeof(int));

			int c = 0;
			for(int i = 0; i < N_h[st] + Nsmall_h[st]; ++i){
				c = 0;
				double rsq = x4_h[i + NBS].x*x4_h[i + NBS].x + x4_h[i + NBS].y*x4_h[i + NBS].y + x4_h[i + NBS].z*x4_h[i + NBS].z;
				if(rsq > Rcut_h[st] * Rcut_h[st] && x4_h[i + NBS].w >= 0){
					c = -3;
					if(Nst == 1){
						if(x4_h[i + NBS].w > 0.0){
							printf("Body %d ejected\n", index_h[i + NBS]);
							fprintf(logfile, "Body %d ejected\n", index_h[i + NBS]);
						}
						else{
							printf("Test Particle %d ejected\n", index_h[i + NBS]);
							fprintf(logfile, "Test Particle %d ejected\n", index_h[i + NBS]);
						}
					}
					else{
						if(x4_h[i + NBS].w > 0.0){
							printf("In Simulation %s: Body %d ejected \n", GSF[st].path, index_h[i + NBS] % def_MaxIndex);
							fprintf(logfile, "Body %d ejected\n", index_h[i + NBS] % def_MaxIndex);
						}
						else{
							printf("In Simulation %s: Test Particle %d ejected \n", GSF[st].path, index_h[i + NBS] % def_MaxIndex);
							fprintf(logfile, "Test Particle %d ejected\n", index_h[i + NBS] % def_MaxIndex);
						}
					}
				}
//if(i == 619) printf("ejection %d %g %g %g\n", i, rsq, RcutSun_h[st] * RcutSun_h[st], x4_h[i + NBS].w);
				if( rsq < RcutSun_h[st] * RcutSun_h[st] && x4_h[i + NBS].w >= 0){
					c = -2;
					if(Nst == 1){
						if(x4_h[i + NBS].w > 0.0){
							printf("Body %d too close to central mass -> removed\n", index_h[i + NBS]);
							fprintf(logfile, "Body %d too close to central mass -> removed\n", index_h[i + NBS]);
						}
						else{
							printf("Test Particle %d too close to central mass -> removed\n", index_h[i + NBS]);
							fprintf(logfile, "Test Particle %d too close to central mass -> removed\n", index_h[i + NBS]);
						}
					}
					else{
						if(x4_h[i + NBS].w > 0.0){
							printf("In Simulation %s: Body %d too close to central mass -> removed\n", GSF[st].path, index_h[i + NBS] % def_MaxIndex);
							fprintf(logfile, "Body %d too close to central mass -> removed\n", index_h[i + NBS] % def_MaxIndex);
						}
						else{
							printf("In Simulation %s: Test Particle %d too close to central mass -> removed\n", GSF[st].path, index_h[i + NBS] % def_MaxIndex);
							fprintf(logfile, "Test Particle %d too close to central mass -> removed\n", index_h[i + NBS] % def_MaxIndex);
						}
					}
				}
				if(c < 0){
					if(Nst == 1) fprintf(ejectfile, "%.20g %d %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %d\n", time_h[0]/365.25, index_h[i + NBS], x4_h[i + NBS].w, v4_h[i + NBS].w, x4_h[i + NBS].x, x4_h[i + NBS].y, x4_h[i + NBS].z, v4_h[i + NBS].x, v4_h[i + NBS].y, v4_h[i + NBS].z, spin_h[i + NBS].x, spin_h[i + NBS].y, spin_h[i + NBS].z, c);
					else fprintf(ejectfile, "%.20g %d %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %d\n", time_h[st]/365.25, index_h[i + NBS] % def_MaxIndex, x4_h[i + NBS].w, v4_h[i + NBS].w, x4_h[i + NBS].x, x4_h[i + NBS].y, x4_h[i + NBS].z, v4_h[i + NBS].x, v4_h[i + NBS].y, v4_h[i + NBS].z, spin_h[i + NBS].x, spin_h[i + NBS].y, spin_h[i + NBS].z, c);
					
					EjectionEnergyCall(st, i);
				}
			}
			fclose(ejectfile);
			fclose(logfile);
		}
	}
}


//This function removes ghost particles and reorders the arrays
//It returns 1 if a simulation has less than the minimal number of bodies, otherwise zero
__host__ int Data::remove(){
	int NminFlag = 0;
	NBmax = 0;
	for(int st = 0; st < Nst; ++st){
#if USE_NAF == 1
		remove_kernel <<<1, 1>>> (x4_d, v4_d, a_d, N_d, Nsmall_d, index_d, spin_d, love_d, migration_d, createFlag_d, test_d, EnergySum_d, rcrit_d, rcritv_d, NBS_h[st], st, aelimits_d, aecount_d, enccount_d, aecountT_d, enccountT_d, K_d, Kold_d, NB[st], NconstT, P.SLevels, P.UseMigrationForce, P.CreateParticles, naf.x_d, naf.y_d, naf.n);
#else
		remove_kernel <<<1, 1>>> (x4_d, v4_d, a_d, N_d, Nsmall_d, index_d, spin_d, love_d, migration_d, createFlag_d, test_d, EnergySum_d, rcrit_d, rcritv_d, NBS_h[st], st, aelimits_d, aecount_d, enccount_d, aecountT_d, enccountT_d, K_d, Kold_d, NB[st], NconstT, P.SLevels, P.UseMigrationForce, P.CreateParticles, NULL, NULL, 0);
#endif
		hipMemcpy(N_h + st, N_d + st, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Nsmall_h + st, Nsmall_d + st, sizeof(int), hipMemcpyDeviceToHost);
		resize(N_h[st], NB[st], 1);
		resize(N_h[st] + Nsmall_h[st], NBT[st], 0);

		if(N_h[st] < Nmin[st].x){
			NminFlag = 1;
		}
		if(Nsmall_h[st] < Nmin[st].y){
			NminFlag = 2;
		}

	}
	return NminFlag;
}



// **************************************
//This function recomputes the value of NB, which is the next bigger 
//number to N which is a power of two.
//also called for NBT with test particles
__host__ void Data::resize(int N, int &NB, int f){

	NB = 16;
	if( N > 16) NB = 32;
	if( N > 32) NB = 64;
	if( N > 64) NB = 128;
	if( N > 128) NB = 256;
	if( N > 256) NB = 512;
	if( N > 512) NB = 1024;
	if( N > 1024) NB = 2048;
	if( N > 2048) NB = 4096;
	if( N > 4096) NB = 8192;
	if( N > 8192) NB = 16384;
	if( N > 16384) NB = 32768;
	if( N > 32768) NB = 65536;
	if( N > 65536) NB = 131072;
	if( N > 131072) NB = 262144;

	if(f == 1){
		//avoid max
		if(NB > NBmax){
			NBmax = NB;
		}
	}
}


//This function rearranges the memory if a simulations is stopped
//It runs with only one thread on the GPU, to avoid unnecesary data copies
__global__ void removeM_kernel(double4 *x4_d, double4 *v4_d, double4 *xold_d, double4 *vold_d, double4 *spin_d, double3 *love_d, double3 *migration_d, int *createFlag_d, double3 *a_d, double *test_d, int *index_d, double *rcrit_d,
double *rcritv_d, int st, int NBS, int NsmallS, int *N_d, int *Nsmall_d, int NT, int NsmallT, const int NconstT, float4 *aelimits_d, unsigned int *aecount_d, unsigned int *enccount_d, unsigned long long *aecountT_d, unsigned long long *enccountT_d, const int SLevels, const int UseMigrationForce, const int CreateParticles, double *nafx_d, double *nafy_d, int nafn, int2 *Encpairs2_d, int Nh){

	for(int j = 0; j < N_d[st]; ++j){
//printf("removeM %d %d %d %d %d\n", st, N_d[st], j, j + NBS, j + NT);
		Encpairs2_d[j + NBS].x = j + NT;
		Encpairs2_d[j + NBS].y = Nh;
		x4_d[j + NT] = x4_d[j + NBS];
		v4_d[j + NT] = v4_d[j + NBS];
		xold_d[j + NT] = xold_d[j + NBS];
		vold_d[j + NT] = vold_d[j + NBS];
		spin_d[j + NT] = spin_d[j + NBS];
		love_d[j + NT] = love_d[j + NBS];
		if(UseMigrationForce > 0){
			migration_d[j + NT] = migration_d[j + NBS];
		}
		if(CreateParticles > 0){
			createFlag_d[j + NT] = createFlag_d[j + NBS];
		}
		a_d[j + NT] = a_d[j + NBS];
		test_d[j + NT] = test_d[j + NBS];
		index_d[j + NT] = index_d[j + NBS];
		for(int l = 0; l < SLevels; ++l){
			rcrit_d[j + NT + l * NconstT] = rcrit_d[j + NBS + l * NconstT];
			rcritv_d[j + NT + l * NconstT] = rcritv_d[j + NBS + l * NconstT];
		}
		aelimits_d[j + NT] = aelimits_d[j + NBS];
		enccount_d[j + NT] = enccount_d[j + NBS];
		aecount_d[j + NT] = aecount_d[j + NBS];
		aecountT_d[j + NT] = aecountT_d[j + NBS];
		enccountT_d[j + NT] = enccountT_d[j + NBS];
		for(int i = 0; i < nafn; ++i){
			nafx_d[(j + NT) * nafn + i] = nafx_d[(j + NBS) * nafn + i];
			nafy_d[(j + NT) * nafn + i] = nafy_d[(j + NBS) * nafn + i];
		}
	}
}


//this kernel rearranges the simulations index
__global__ void remove3M_kernel(int *index_d, int *N_d, int *NBS_d, const int Nst){

	int idy = threadIdx.x;
	int st = blockIdx.x;

	if(st < Nst){

		int N = N_d[st];
		int NBS = NBS_d[st];

		if(idy < N){

			int index = index_d[idy + NBS] % def_MaxIndex;
			index_d[idy + NBS] = index + st * def_MaxIndex;
	//printf("index %d %d %d\n", st, index + st * def_MaxIndex, N);
		}
	}
}


//this kernel rearranges the indices of the prechecker list
__global__ void remove4M_kernel(int2 *Encpairs_d, int2 *Encpairs2_d, const int Nencpairs){

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < Nencpairs){

		int i = Encpairs_d[id].x;
		int j = Encpairs_d[id].y;
		
		int ii = Encpairs2_d[i].x;
		int jj = Encpairs2_d[j].x;

		Encpairs_d[id].x = ii;
		Encpairs_d[id].y = jj;

		if(Encpairs2_d[i].y == 0) Encpairs_d[id].x = -1;
		if(Encpairs2_d[j].y == 0) Encpairs_d[id].y = -1;

	}
}



// This function stops simulations with less than the minimal number of bodies 
// or if the simulation has ended.
// it rearanges the memory
__host__ void Data::stopSimulations(){
	NT = 0;
	NsmallT = 0;
	NEnergyT = 0;

	for(int st = 0; st < Nst; ++st){

		//In the following, set N_h to zero for all simulations which should be stopped
		if(N_h[st] < Nmin[st].x){
			N_h[st] = 0;
		}
		if(Nsmall_h[st] < Nmin[st].y){
			N_h[st] = 0;
		}

		if(StopFlag_m[0] > 0 && timeStep >= delta_h[st]){
			N_h[st] = 0;
		}
		if(P.StopAtEncounter > 0 && n1_h[st] < 0){
			N_h[st] = 0;

		}
		//rearange arrays//
#if USE_NAF == 1
		removeM_kernel <<< 1, 1>>> (x4_d, v4_d, xold_d, vold_d, spin_d, love_d, migration_d, createFlag_d, a_d, test_d, index_d, rcrit_d, rcritv_d,  
					    st, NBS_h[st], NsmallS_h[st], N_d, Nsmall_d, NT, NsmallT, NconstT, aelimits_d,
					    aecount_d, enccount_d, aecountT_d, enccountT_d, P.SLevels, P.UseMigrationForce, P.CreateParticles, naf.x_d, naf.y_d, naf.n, Encpairs2_d, N_h[st]);
#else
		removeM_kernel <<< 1, 1>>> (x4_d, v4_d, xold_d, vold_d, spin_d, love_d, migration_d, createFlag_d, a_d, test_d, index_d, rcrit_d, rcritv_d, 
					    st, NBS_h[st], NsmallS_h[st], N_d, Nsmall_d, NT, NsmallT, NconstT, aelimits_d,
					    aecount_d, enccount_d, aecountT_d, enccountT_d, P.SLevels, P.UseMigrationForce, P.CreateParticles, NULL, NULL, 0, Encpairs2_d, N_h[st]);
#endif

		NBS_h[st] = NT;
		NsmallS_h[st] = NsmallT;
		NEnergy[st] = NEnergyT;
		NT += N_h[st];
		NsmallT += Nsmall_h[st];
		NEnergyT += 8;
	}

	hipMemcpy(U_h, U_d, Nst*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(LI_h, LI_d, Nst*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Energy_h, Energy_d, NEnergyT*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Energy0_h, Energy0_d, Nst*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(LI0_h, LI0_d, Nst*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Msun_h, Msun_d, Nst*sizeof(double2), hipMemcpyDeviceToHost);
	hipMemcpy(Spinsun_h, Spinsun_d, Nst*sizeof(double4), hipMemcpyDeviceToHost);
	hipMemcpy(Lovesun_h, Lovesun_d, Nst*sizeof(double3), hipMemcpyDeviceToHost);
	hipMemcpy(J2_h, J2_d, Nst*sizeof(double2), hipMemcpyDeviceToHost);

	for(int st = 0; st < Nst; ++st){

//printf("stop simulations  %d %d %d %d Nst %d Ntot %d\n", st, N_h[st], Nmin[st].x, Nmin[st].y, Nst, NT);
		int s = 0;
		if(timeStep >= delta_h[st]){
			printf("In Simulation %s: Reached the end, simulation stopped\n", GSF[st].path);
			fprintf(masterfile,"In Simulation %s: Reached the end, simulation stopped\n", GSF[st].path);
			GSF[st].logfile = fopen(GSF[st].logfilename, "a");
			fprintf(GSF[st].logfile,"Reached the end, simulation stopped\n");
			fclose(GSF[st].logfile);
			s = 1;
		}
		else if(N_h[st] < Nmin[st].x){
			if(P.StopAtEncounter > 0 && n1_h[st] < 0){
				if(Nst > 1){
					printf("In Simulation %s: Close Encounter occurred, simulation stopped\n", GSF[st].path);
					fprintf(masterfile,"In Simulation %s: Close Encounter occurred, simulation stopped\n", GSF[st].path);
					GSF[st].logfile = fopen(GSF[st].logfilename, "a");
					fprintf(GSF[st].logfile,"Close Encounter occurred, simulation stopped\n");
					fclose(GSF[st].logfile);
					s = 1;
				}
				else{
					printf("Close Encounter occurred, simulation stopped\n");
					fprintf(masterfile,"Close Encounter occurred, simulation stopped\n");
					GSF[st].logfile = fopen(GSF[st].logfilename, "a");
					fprintf(GSF[st].logfile,"Close Encounter occurred, simulation stopped\n");
					fclose(GSF[st].logfile);
					s = 1;
				}
			}
			else{
				if(Nst > 1){
					if(Nsmall_h[st] < Nmin[st].y){
						printf("In Simulation %s: Number of test particles smaller than NminTP, simulation stopped\n", GSF[st].path);
						fprintf(masterfile,"In Simulation %s: Number of test particles smaller than NminTP, simulation stopped\n", GSF[st].path);
						GSF[st].logfile = fopen(GSF[st].logfilename, "a");
						fprintf(GSF[st].logfile,"Number of test particles smaller than NminTP, simulation stopped\n");
						fclose(GSF[st].logfile);
					}
					else{
						printf("In Simulation %s: Number of bodies smaller than Nmin, simulation stopped\n", GSF[st].path);
						fprintf(masterfile,"In Simulation %s: Number of bodies smaller than Nmin, simulation stopped\n", GSF[st].path);
						GSF[st].logfile = fopen(GSF[st].logfilename, "a");
						fprintf(GSF[st].logfile,"Number of bodies smaller than Nmin, simulation stopped\n");
						fclose(GSF[st].logfile);
					}
					s = 1;
				}
				else{
					if(Nsmall_h[st] < Nmin[st].y){
						printf("Number of test particles smaller than NminTP, simulation stopped\n");
						fprintf(masterfile,"Number of test particles smaller than NminTP, simulation stopped\n");
						GSF[0].logfile = fopen(GSF[0].logfilename, "a");
						fprintf(GSF[0].logfile,"Number of test particles smaller than NminTP, simulation stopped\n");
						fclose(GSF[0].logfile);
					}
					else{
						printf("Number of bodies smaller than Nmin, simulation stopped\n");
						fprintf(masterfile,"Number of bodies smaller than Nmin, simulation stopped\n");
						GSF[0].logfile = fopen(GSF[0].logfilename, "a");
						fprintf(GSF[0].logfile,"Number of bodies smaller than Nmin, simulation stopped\n");
						fclose(GSF[0].logfile);
					}
					s = 1;
				}
			}
		}
		if(s == 1){
			for(int sst = st; sst < Nst - 1; ++sst){
				GSF[sst] = GSF[sst + 1];

				NB[sst] = NB[sst + 1];
				Nmin[sst].x = Nmin[sst + 1].x;
				Nmin[sst].y = Nmin[sst + 1].y;
				rho[sst] = rho[sst + 1];
				n1_h[sst] = n1_h[sst + 1];
				n2_h[sst] = n2_h[sst + 1];
				N_h[sst] = N_h[sst + 1];
				Nsmall_h[sst] = Nsmall_h[sst + 1];
				Msun_h[sst] = Msun_h[sst + 1];
				Spinsun_h[sst] = Spinsun_h[sst + 1];
				Lovesun_h[sst] = Lovesun_h[sst + 1];
				J2_h[sst] = J2_h[sst + 1];
				idt_h[sst] = idt_h[sst + 1];
				ict_h[sst] = ict_h[sst + 1];
				Rcut_h[sst] = Rcut_h[sst + 1];
				RcutSun_h[sst] = RcutSun_h[sst + 1];
				time_h[sst] = time_h[sst + 1];
				dt_h[sst] = dt_h[sst + 1];
				delta_h[sst] = delta_h[sst + 1];

				U_h[sst] = U_h[sst + 1];
				LI_h[sst] = LI_h[sst + 1];
				Energy0_h[sst] = Energy0_h[sst + 1];
				LI0_h[sst] = LI0_h[sst + 1];

				NBS_h[sst] = NBS_h[sst + 1];
				NsmallS_h[sst] = NsmallS_h[sst + 1];
				NEnergy[sst] = NEnergy[sst + 1];

				for(int j = 0; j < 8; ++j){
					int NE0 = NEnergy[sst];
					int NE1 = NEnergy[sst + 1];
					Energy_h[NE0 + j] = Energy_h[NE1 + j];
				}
			}
			st -= 1;
			Nst -= 1;

		}
	}

	hipMemcpy(n1_d, n1_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(n2_d, n2_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(N_d, N_h, Nst*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Nsmall_d, Nsmall_h, Nst*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Msun_d, Msun_h, Nst*sizeof(double2), hipMemcpyHostToDevice);
	hipMemcpy(Spinsun_d, Spinsun_h, Nst*sizeof(double4), hipMemcpyHostToDevice);
	hipMemcpy(Lovesun_d, Lovesun_h, Nst*sizeof(double3), hipMemcpyHostToDevice);
	hipMemcpy(J2_d, J2_h, Nst*sizeof(double2), hipMemcpyHostToDevice);
	hipMemcpy(idt_d, idt_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(ict_d, ict_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Rcut_d, Rcut_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(RcutSun_d, RcutSun_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(time_d, time_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dt_d, dt_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(delta_d, delta_h, Nst*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(U_d, U_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(LI_d, LI_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(LI0_d, LI0_h, Nst*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Energy_d, Energy_h, NEnergyT*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(Energy0_d, Energy0_h, Nst*sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(NBS_d, NBS_h, Nst*sizeof(int), hipMemcpyHostToDevice);

	if(Nst > 0){
		remove3M_kernel <<< Nst, NBmax >>> (index_d, N_d, NBS_d, Nst);
		if(Nencpairs_h[0] > 0) remove4M_kernel <<< (Nencpairs_h[0] + 255) / 256, 256 >>> (Encpairs_d, Encpairs2_d, Nencpairs_h[0]);
	}
}

#if def_CPU == 1
__host__ void Data::ElapsedTime(float *times, timeval tt1, timeval tt2){
	
	times[0] = (1000 * tt2.tv_sec + 0.001 * tt2.tv_usec - 1000 * tt1.tv_sec - 0.001 * tt1.tv_usec); //time in milliseconds
}
#endif

__host__ int Data::freeOrbit(){
	
	hipError_t error;
	
	free(x4_h);
	free(v4_h);
	free(index_h);
	free(spin_h);
	free(love_h);
	if(P.UseMigrationForce > 0){
		free(migration_h);
	}
	if(P.CreateParticles > 0){
		free(createFlag_h);
	}
	free(rcrit_h);
	free(aelimits_h);
	free(aecount_h);
	free(enccount_h);
	free(aecountT_h);
	free(enccountT_h);

	free(coordinateBuffer_h);
	free(coordinateBufferIrr_h);
	free(timestepBuffer);
	free(timestepBufferIrr);
	free(NBuffer);
	free(NBufferIrr);

	free(RV_h);
	free(RVObs_h);
	free(TransitTime_h);
	free(TransitTimeObs_h);
	free(NtransitsT_h);
	free(NRVT_h);
	free(NtransitsTObs_h);
	free(NRVTObs_h);
	free(elementsA_h);
	free(elementsB_h);
	free(elementsT_h);
	free(elementsSpin_h);
	free(elementsL_h);
	free(elementsC_h);
	free(elementsP_h);
	free(elementsSA_h);
	free(elementsI_h);
	free(elementsM_h);
	free(elementsCOV_h);

	free(groupIterate_h);

	free(U_h);
	free(LI_h);
	free(Energy_h);
	free(Energy0_h);
	free(LI0_h);

#if def_CPU == 0

	hipHostFree(Nenc_m);
	hipHostFree(Ncoll_m);
	hipHostFree(Ntransit_m);
	hipHostFree(NWriteEnc_m);
	hipHostFree(EjectionFlag_m);
	hipHostFree(nFragments_m);
	hipHostFree(EncFlag_m);
	hipHostFree(StopFlag_m);
	hipHostFree(ErrorFlag_m);
	hipHostFree(test_h);
	hipHostFree(Nencpairs_h);
	hipHostFree(Nencpairs2_h);
	hipHostFree(Nencpairs3_h);
#else
#if def_CPU == 1
	free(xold_h);
	free(vold_h);
	free(x4b_h);
	free(v4b_h);
	free(x4bb_h);
	free(v4bb_h);
	free(a_h);
	free(b_h);
	free(ab_h);
	free(indexb_h);
	free(indexbb_h);
	free(rcritb_h);
	free(rcritbb_h);
	free(rcritv_h);
	free(rcritvb_h);
	free(rcritvbb_h);
	free(spinb_h);
	free(spinbb_h);
	free(vcom_h);
	free(EnergySum_h);
	free(Encpairs_h);
	free(Encpairs2_h);
	free(Encpairs3_h);
	free(scan_h);

	free(groupIndex_h);

	//BSA
	free(xt_h);
	free(vt_h);
	free(xp_h);
	free(vp_h);
	free(dx_h);
	free(dv_h);
	free(dt1_h);
	free(t1_h);
	free(dtgr_h);
	free(Coltime_h);
	free(BSstop_h);

	if(P.WriteEncounters == 2){
		free(morton_h);
		free(sortRank_h);
		free(sortCount_h);
		free(sortIndex_h);
		free(leafNodes_h);
		free(internalNodes_h);
	}
#endif


	free(Nenc_m);
	free(Ncoll_m);
	free(Ntransit_m);
	free(NWriteEnc_m);
	free(EjectionFlag_m);
	free(nFragments_m);
	free(EncFlag_m);
	free(StopFlag_m);
	free(ErrorFlag_m);
	free(test_h);
	free(Nencpairs_h);
	free(Nencpairs2_h);
	free(Nencpairs3_h);
#endif

	free(Coll_h);
	free(writeEnc_h);
	free(Fragments_h);

#if def_poincareFlag == 1
	free(PFlag_h);
#endif	
	free(BSAstop_h);

	hipFree(x4_d);
	hipFree(v4_d);
	hipFree(xold_d);
	hipFree(vold_d);
	hipFree(rcrit_d);
	hipFree(rcritv_d);
	hipFree(rcritb_d);
	hipFree(rcritvb_d);
	hipFree(rcritbb_d);
	hipFree(rcritvbb_d);
	hipFree(test_d);
	hipFree(index_d);
	hipFree(spin_d);
	hipFree(spinb_d);
	hipFree(spinbb_d);
	hipFree(love_d);
	if(P.UseMigrationForce > 0){
		hipFree(migration_d);
	}
	if(P.CreateParticles > 0){
		hipFree(createFlag_d);
	}
	hipFree(U_d);
	hipFree(LI_d);
	hipFree(LI0_d);
	hipFree(a_d);
	hipFree(Energy_d);
	hipFree(Energy0_d);
	hipFree(EnergySum_d);
	hipFree(Nencpairs_d);
	hipFree(Nencpairs2_d);
	hipFree(Nencpairs3_d);
	hipFree(groupIterate_d);
	hipFree(Encpairs_d);
	hipFree(Encpairs2_d);
	hipFree(Encpairs3_d);
	hipFree(scan_d);
	hipFree(Coll_d);
	hipFree(writeEnc_d);
	hipFree(Fragments_d);
	hipFree(aelimits_d);
	hipFree(aecount_d);
	hipFree(enccount_d);
	hipFree(aecountT_d);
	hipFree(enccountT_d);

	hipFree(coordinateBuffer_d);
	hipFree(coordinateBufferIrr_d);

	if(P.WriteEncounters == 2){
		hipFree(morton_d);
		hipFree(sortRank_d);
		hipFree(sortCount_d);
		hipFree(sortIndex_d);
		hipFree(leafNodes_d);
		hipFree(internalNodes_d);
	}

	if(Nst > 1){
		hipFree(groupIndex_d);
	}

	if(P.ndev > 1){
		hipFree(rcritv_d1);
		hipFree(x4_d1);
		hipFree(Nencpairs_d1);
		hipFree(Encpairs_d1);
		hipFree(Encpairs2_d1);
	}
	if(P.ndev > 2){
		hipFree(rcritv_d2);
		hipFree(x4_d2);
		hipFree(Nencpairs_d2);
		hipFree(Encpairs_d2);
		hipFree(Encpairs2_d2);
	}
	if(P.ndev > 3){
		hipFree(rcritv_d3);
		hipFree(x4_d3);
		hipFree(Nencpairs_d3);
		hipFree(Encpairs_d3);
		hipFree(Encpairs2_d3);
	}


	hipFree(xt_d);
	hipFree(vt_d);
	hipFree(xp_d);
	hipFree(vp_d);
	hipFree(dx_d);
	hipFree(dv_d);
	hipFree(dt1_d);
	hipFree(t1_d);
	hipFree(dtgr_d);
	hipFree(BSAstop_d);
	hipFree(BSstop_d);
	hipFree(Coltime_d);


	hipFree(x4b_d);
	hipFree(v4b_d);
	hipFree(x4bb_d);
	hipFree(v4bb_d);
	hipFree(ab_d);
	hipFree(indexb_d);
	hipFree(indexbb_d);

	hipFree(vcom_d);
	
#if def_poincareFlag == 1
	hipFree(PFlag_d);
#endif
#if def_G3 > 0
	hipFree(K_d);
	hipFree(Kold_d);
#endif

#if USE_RANDOM == 1
	hipFree(random_d);
#endif


	hipFree(Transit_d);
	hipFree(RV_d);
	hipFree(RVObs_d);
	hipFree(TransitTime_d);
	hipFree(TransitTimeObs_d);
	hipFree(NtransitsT_d);
	hipFree(NRVT_d);
	hipFree(NtransitsTObs_d);
	hipFree(NRVTObs_d);
	hipFree(RVP_d);
	hipFree(elementsA_d);
	hipFree(elementsB_d);
	hipFree(elementsT_d);
	hipFree(elementsSpin_d);
	hipFree(elementsAOld_d);
	hipFree(elementsAOld2_d);
	hipFree(elementsBOld_d);
	hipFree(elementsBOld2_d);
	hipFree(elementsTOld_d);
	hipFree(elementsTOld2_d);
	hipFree(elementsSpinOld_d);
	hipFree(elementsSpinOld2_d);
	hipFree(elementsL_d);
	hipFree(elementsC_d);
	hipFree(elementsP_d);
	hipFree(elementsSA_d);
	hipFree(elementsI_d);
	hipFree(elementsM_d);
	hipFree(elementsCOV_d);
	hipFree(elementsG_d);
	hipFree(elementsGh_d);
	hipFree(elementsD_d);
	hipFree(elementsMean_d);
	hipFree(elementsVar_d);
	hipFree(elementsStep_d);
	hipFree(elementsHist_d);
	hipFree(Symplex_d);
	hipFree(SymplexCount_d);

	
	error = hipGetLastError();
	if(error != 0){
		printf("Cuda Orbit free error = %d = %s\n",error, hipGetErrorString(error));
		fprintf(masterfile, "Cuda Orbit free error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	return 1;
}

