#include "hip/hip_runtime.h"
#include "Orbit2.h"
hipEvent_t tt1;			//start time
hipEvent_t tt2;			//start time of a output time intervall
hipEvent_t tt3;			//end time of a output time intervall
hipEvent_t tt4;			//end time//

float times;				//elapsed time in milliseconds

// ********************************************3
//This function prints the initial Energy and Coordinate output
//If Restart is set, then it reads the corespondent initial conditions from the files and writes no output
//
//Author: Simon Grimm
//June 2015
// *************************************

__host__ void aei(double3 x4i, double3 v4i, double mu, double &a, double &e, double &inc, double &Omega, double &w, double &Theta, double &E, double &M){
	double rsq = x4i.x * x4i.x + x4i.y * x4i.y + x4i.z * x4i.z;
	double vsq = v4i.x * v4i.x + v4i.y * v4i.y + v4i.z * v4i.z;
	double u =  x4i.x * v4i.x + x4i.y * v4i.y + x4i.z * v4i.z;
	double ir = 1.0 / sqrt(rsq);
	double ia = 2.0 * ir - vsq / mu;

	a = 1.0 / ia;

	//inclination
	double3 h3;
	double h2, h, t;
	h3.x = ( x4i.y * v4i.z) - (x4i.z * v4i.y);
	h3.y = (-x4i.x * v4i.z) + (x4i.z * v4i.x);
	h3.z = ( x4i.x * v4i.y) - (x4i.y * v4i.x);

	h2 = h3.x * h3.x + h3.y * h3.y + h3.z * h3.z;
	h = sqrt(h2);

	t = h3.z / h;
	if(t < -1.0) t = -1.0;
	if(t > 1.0) t = 1.0;

	inc = acos(t);

	//longitude of ascending node
	double n = sqrt(h3.x * h3.x + h3.y * h3.y);
	Omega = acos(-h3.y / n);
	if(h3.x < 0.0){
		Omega = 2.0 * M_PI - Omega;
	}

	if(inc < 1.0e-10 || n == 0) Omega = 0.0;

	//argument of periapsis
	double3 e3;
	e3.x = ( v4i.y * h3.z - v4i.z * h3.y) / mu - x4i.x * ir;
	e3.y = (-v4i.x * h3.z + v4i.z * h3.x) / mu - x4i.y * ir;
	e3.z = ( v4i.x * h3.y - v4i.y * h3.x) / mu - x4i.z * ir;


	e = sqrt(e3.x * e3.x + e3.y * e3.y + e3.z * e3.z);

	t = (-h3.y * e3.x + h3.x * e3.y) / (n * e);
	if(t < -1.0) t = -1.0;
	if(t > 1.0) t = 1.0;
	w = acos(t);
	if(e3.z < 0.0) w = 2.0 * M_PI - w;
	if(n == 0) w = 0.0;

	//True Anomaly
	t = (e3.x * x4i.x + e3.y * x4i.y + e3.z * x4i.z) / e * ir;
	if(t < -1.0) t = -1.0;
	if(t > 1.0) t = 1.0;
	Theta = acos(t);
	if(u < 0.0){
		if(e < 1.0 - 1.0e-10){
			//elliptic
			Theta = 2.0 * M_PI - Theta;
		}
		else if(e > 1.0 + 1.0e-10){
			//hyperbolic
			Theta = -Theta;
		}
		else{
			//parabolic
			Theta = - Theta;
		}
	}

	//Non circular, equatorial orbit
	if(e > 1.0e-10 && inc < 1.0e-10){
		Omega = 0.0;
		w = acos(e3.x / e);
		if(e3.y < 0.0) w = 2.0 * M_PI - w;
	}

	//circular, inclinded orbit
		if(e < 1.0e-10 && inc > 1.0e-11){
		w = 0.0;
	}

	//circular, equatorial orbit
	if(e < 1.0e-10 && inc < 1.0e-11){
		w = 0.0;
		Omega = 0.0;
	}

	if(w == 0 && Omega != 0.0){
		t = (-h3.y * x4i.x + h3.x * x4i.y) / n * ir;
		if(t < -1.0) t = -1.0;
		if(t > 1.0) t = 1.0;
		Theta = acos(t);
		if(x4i.z < 0.0){
			if(e < 1.0 - 1.0e-10){
				//elliptic
				Theta = 2.0 * M_PI - Theta;
			}
			else if(e > 1.0 + 1.0e-10){
				//hyperbolic
				Theta = -Theta;
			}
			else{
				//parabolic
				Theta = -Theta;
			}
		}
	}
	if(w == 0 && Omega == 0.0){
		Theta = acos(x4i.x * ir);
		if(x4i.y < 0.0){
			if(e < 1.0 - 1.0e-10){
				//elliptic
				Theta = 2.0 * M_PI - Theta;
			}
			else if(e > 1.0 + 1.0e-10){
				//hyperbolic
				Theta = -Theta;
			}
			else{
				//parabolic
				Theta = -Theta;
			}
		}
	}

	if(e < 1.0 - 1.0e-10){
		//Eccentric Anomaly
		E = acos((e + cos(Theta)) / (1.0 + e * cos(Theta)));
		if(M_PI < Theta && Theta < 2.0 * M_PI) E = 2.0 * M_PI - E;

		//Mean Anomaly
		M = E - e * sin(E);
	}
	else if(e > 1.0 + 1.0e-10){
		//Hyperbolic Anomaly
		//named still E instead of H or F
		E = acosh((e + t) / (1.0 + e * t));
		if(Theta < 0.0) E = - E;

		M = e * sinh(E) - E;
	}
	else{
		//Parabolic Anomaly
		E = tan(Theta * 0.5);

		if(E > M_PI) E = E - 2.0 * M_PI;

		M = E + E * E * E / 3.0;

		//use a to store q
		a = h * h / mu * 0.5;
	}
}


__host__ int Data::firstoutput(int irregular){

	char dat_bin[16];
	if(P.OutBinary == 0){
		sprintf(dat_bin, "%s", "dat");
	}
	else{
		sprintf(dat_bin, "%s", "bin");
	}

	for(int st = 0; st < Nst; ++st){
		FILE *Energyfile;

		//check if EnergyIrrfile already exists
		//This is needed for Gasolenga runs
		int readIrrEnergyFile = 0;
		if(irregular == 1){
			FILE *Efile;
			Efile = fopen(GSF[st].EnergyIrrfilename, "r");
			if(Efile != NULL){
				readIrrEnergyFile = 1;
				printf("read initial energy from %s file\n", GSF[st].EnergyIrrfilename);
			}

		}

		if(P.tRestart == 0 && readIrrEnergyFile == 0){
			int NBS = NBS_h[st];
			if(P.ei > 0 || irregular == 1){
				if(irregular == 0){
					Energyfile = fopen(GSF[st].Energyfilename, "a");
				}
				else{
					Energyfile = fopen(GSF[st].EnergyIrrfilename, "a");

				}
				if(Energyfile == NULL){
					printf("Error, Energyfile not valid %d %s\n", st, GSF[st].timefilename);
					return 0;
				}
				hipMemcpy(Energy_h + NEnergy[st], Energy_d + NEnergy[st], sizeof(double)*8, hipMemcpyDeviceToHost);
				fprintf(Energyfile,"%.16g %d %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g\n", ict_h[st], N_h[st] + Nsmall_h[st], Energy_h[0 + NEnergy[st]], Energy_h[1 + NEnergy[st]], Energy_h[2 + NEnergy[st]], Energy_h[3 + NEnergy[st]], Energy_h[4 + NEnergy[st]], Energy_h[5 + NEnergy[st]], Energy_h[6 + NEnergy[st]], Energy_h[7 + NEnergy[st]]);
				fclose(Energyfile);
			}
			if(P.ci > 0){
				if(P.FormatP == 1){
					if(Nst == 1 || P.FormatS == 0){
						//clear Irregular output files
						if(P.FormatT == 0) sprintf(GSF[st].outputfilename, "%sOutIrr%s_%.*d.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, 0, dat_bin);
						if(P.FormatT == 1) sprintf(GSF[st].outputfilename, "%sOutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						FILE *file;
						if(P.OutBinary == 0){
							file = fopen(GSF[st].outputfilename, "r");
						}
						else{
							file = fopen(GSF[st].outputfilename, "rb");
						}
						if(file != NULL){
							fclose(file);
							if(P.OutBinary == 0){
								file = fopen(GSF[st].outputfilename, "w");
							}
							else{
								file = fopen(GSF[st].outputfilename, "wb");
							}
							fclose(file);
						}
			
		
						if(P.FormatT == 0) sprintf(GSF[st].outputfilename, "%sOut%s_%.*d.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, 0, dat_bin);
						if(P.FormatT == 1) sprintf(GSF[st].outputfilename, "%sOut%s.%s", GSF[st].path, GSF[st].X, dat_bin);
#if def_TTV == 0
						if(P.OutBinary == 0){
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
						}
						else{
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
						}
#else
						if(P.OutBinary == 0){
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
						}
						else{
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
						}
#endif
					}
					else{
						//clear Irregular output files
						if(P.FormatT == 0)sprintf(GSF[st].outputfilename, "%s../OutIrr%s_%.*d.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, 0, dat_bin);
						if(P.FormatT == 1)sprintf(GSF[st].outputfilename, "%s../OutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						FILE *file;
						if(P.OutBinary == 0){
							file = fopen(GSF[st].outputfilename, "r");
						}
						else{
							file = fopen(GSF[st].outputfilename, "rb");
						}
						if(file != NULL){
							fclose(file);
							if(P.OutBinary == 0){
								file = fopen(GSF[st].outputfilename, "w");
							}
							else{
								file = fopen(GSF[st].outputfilename, "wb");
							}
							fclose(file);
						}
				

						if(P.FormatT == 0)sprintf(GSF[st].outputfilename, "%s../Out%s_%.*d.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, 0, dat_bin);
						if(P.FormatT == 1)sprintf(GSF[st].outputfilename, "%s../Out%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						if(P.OutBinary == 0){
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
						}
						else{
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
						}
					}
				}
				else{
					//clear Irregular output files
					if(Nst == 1 || P.FormatS == 0){
						for(int i = 0; i < N_h[st] + Nsmall_h[st]; ++i){
							char name[300];
							sprintf(name, "%sOutIrr%s_p%.6d.%s", GSF[st].path, GSF[st].X, i, dat_bin);
							FILE *file;
							if(P.OutBinary == 0){
								file = fopen(name, "r");
							}
							else{
								file = fopen(name, "rb");
							}
							if(file != NULL){
								fclose(file);
								if(P.OutBinary == 0){
									file = fopen(name, "w");
								}
								else{
									file = fopen(name, "wb");
								}
								fclose(file);
							}
						}
					}
					else{
						for(int i = 0; i < N_h[st] + Nsmall_h[st]; ++i){
							char name[300];
							sprintf(name, "%s../OutIrr%s_p%.6d.%s", GSF[st].path, GSF[st].X, i, dat_bin);
							FILE *file;
							if(P.OutBinary == 0){
								file = fopen(name, "r");
							}
							else{
								file = fopen(name, "rb");
							}
							if(file != NULL){
								fclose(file);
								if(P.OutBinary == 0){
									file = fopen(name, "w");
								}
								else{
									file = fopen(name, "wb");
								}
								fclose(file);
							}
						}

					}
				}

				printOutput(x4_h + NBS, v4_h + NBS, v4Helio_h + NBS, index_h + NBS, test_h + NBS, ict_h[st], 1, N_h[st], GSF[st].outputfile, Msun_h[st].x, spin_h + NBS, love_h + NBS, migration_h + NBS, rcrit_h + NBS, Nsmall_h[st], Nst, aelimits_h + NBS, aecount_h + NBS, enccount_h + NBS, aecountT_h + NBS, enccountT_h + NBS, P.ci, 0);
				if(P.FormatP == 1) fclose(GSF[st].outputfile);
			}
		}
		else if(N_h[st] + Nsmall_h[st] > 0){
			int tsign = 1;
			if(idt_h[st] < 0) tsign = -1;
			double skip;
			double Et;
			char Ets[160];
			int er = 0;
			if(readIrrEnergyFile == 0){
				Energyfile = fopen(GSF[st].Energyfilename, "r");
				sprintf(Ets, "%.16g", (P.tRestart * idt_h[st] + ict_h[st] * 365.25) / 365.25);

				er = fscanf (Energyfile, "%lf",&Et);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&LI_h[st]);
				er = fscanf (Energyfile, "%lf",&U_h[st]);
				er = fscanf (Energyfile, "%lf",&Energy0_h[st]);
				er = fscanf (Energyfile, "%lf",&LI0_h[st]);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
	
				U_h[st] /= def_Kg;
				LI_h[st] /= dayUnit;
			}
			else{
				//read only initial energy and angular momentum
				Energyfile = fopen(GSF[st].EnergyIrrfilename, "r");
				sprintf(Ets, "%.16g", (ict_h[st] * 365.25) / 365.25);

				er = fscanf (Energyfile, "%lf",&Et);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&Energy0_h[st]);
				er = fscanf (Energyfile, "%lf",&LI0_h[st]);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
			}

//printf("%.20g %.20g %d %d\n", Et, atof(Ets), tsign, er);
			while(Et * tsign < atof(Ets) * tsign){
				er = fscanf (Energyfile, "%lf",&Et);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&LI_h[st]);
				er = fscanf (Energyfile, "%lf",&U_h[st]);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
				er = fscanf (Energyfile, "%lf",&skip);
//printf("%.20g %.20g %d %d\n", Et, atof(Ets), tsign, er);
	
				U_h[st] /= def_Kg;
				LI_h[st] /= dayUnit;

				if(Et * tsign >= atof(Ets) * tsign) break;

				if(er <= 0){
					break;
				}				
			}		
			if(er <= 0){
				fprintf(masterfile, "Error: In Simulation %s: Restart time step not valid %g %g\n", GSF[st].path, atof(Ets), Et);
				printf("Error: In Simulation %s: Restart time step not valid %g %g\n", GSF[st].path, atof(Ets), Et);
				return 0;
			}
//printf("Energy %g %g %g %g\n", Energy0_h[0], U_h[0] * def_Kg, LI0_h[0], LI_h[0] * dayUnit);


			fclose(Energyfile);
			hipMemcpy(Energy0_d + st, Energy0_h + st, sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(U_d + st, U_h + st, sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(LI_d + st, LI_h + st, sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(LI0_d + st, LI0_h + st, sizeof(double), hipMemcpyHostToDevice);

			if(irregular == 0 && (P.UseTides > 0 || P.UseRotationalDeformation > 0)){
				//print star file
				FILE *starfile;
				int er = 0;
				starfile = fopen(GSF[st].starfilename, "r");

				er = fscanf (Energyfile, "%lf",&Et);
				er = fscanf (Energyfile, "%lf",&Msun_h[st].x);
				er = fscanf (Energyfile, "%lf",&Msun_h[st].y);
				er = fscanf (Energyfile, "%lf",&Spinsun_h[st].x);
				er = fscanf (Energyfile, "%lf",&Spinsun_h[st].y);
				er = fscanf (Energyfile, "%lf",&Spinsun_h[st].z);
				er = fscanf (Energyfile, "%lf",&Spinsun_h[st].w);
				er = fscanf (Energyfile, "%lf",&Lovesun_h[st].x);
				er = fscanf (Energyfile, "%lf",&Lovesun_h[st].y);
				er = fscanf (Energyfile, "%lf",&Lovesun_h[st].z);

//printf("%.20g %.20g %d %d\n", Et, atof(Ets), tsign, er);
				while(Et * tsign < atof(Ets) * tsign){
					er = fscanf (Energyfile, "%lf",&Et);
					er = fscanf (Energyfile, "%lf",&Msun_h[st].x);
					er = fscanf (Energyfile, "%lf",&Msun_h[st].y);
					er = fscanf (Energyfile, "%lf",&Spinsun_h[st].x);
					er = fscanf (Energyfile, "%lf",&Spinsun_h[st].y);
					er = fscanf (Energyfile, "%lf",&Spinsun_h[st].z);
					er = fscanf (Energyfile, "%lf",&Spinsun_h[st].w);
					er = fscanf (Energyfile, "%lf",&Lovesun_h[st].x);
					er = fscanf (Energyfile, "%lf",&Lovesun_h[st].y);
					er = fscanf (Energyfile, "%lf",&Lovesun_h[st].z);
//printf("%.20g %.20g %d %d\n", Et, atof(Ets), tsign, er);
					if(Et * tsign >= atof(Ets) * tsign) break;

					if(er <= 0){
						break;
					}				
				}
				if(er <= 0){
					fprintf(masterfile, "Error: In Simulation %s: Restart time step not valid for star file %g %g\n", GSF[st].path, atof(Ets), Et);
					printf("Error: In Simulation %s: Restart time step not valid for star file %g %g\n", GSF[st].path, atof(Ets), Et);
					return 0;
				}
	
				hipMemcpy(Msun_d + st, Msun_h + st, sizeof(double2), hipMemcpyHostToDevice);
				hipMemcpy(Spinsun_d + st, Spinsun_h + st, sizeof(double4), hipMemcpyHostToDevice);
				hipMemcpy(Lovesun_d + st, Lovesun_h + st, sizeof(double3), hipMemcpyHostToDevice);
//printf("Spin %g %g\n", Et, Spinsun_h[st].z);
				fclose(starfile);
			}
		}
	}
	return 1;
}


//**************************************
//This function prints the coordinate output
//Authors: Simon Grimm, Joachim Stadel
//March 2014
// ***************************************
//aei(double3 x4i, double3 v4i, double mu, double &a, double &e, double &inc, double &Omega, double &w, double &Theta, double &E, double &M)
__host__ void Data::printOutput(double4 *x4_h, double4 *v4_h, double4 *v4Helio_h, int *index_h, double *test_h, double time, long long timeStep, int N, FILE *outputfile, double Msun, double4 *spin_h, double3 *love_h, double3 *migration_h, double *rcrit_h, int Nsmall, int Nst, float4 *aelimits_h, unsigned int *aecount_h, unsigned int *enccount_h, unsigned long long *aecountT_h, unsigned long long *enccountT_h, int ci, int irregular){
	
	
	DemoToHelio(x4_h, v4_h, v4Helio_h, Msun, N + Nsmall);
	//BaryToHelio(x4_h, v4_h, Msun, N + Nsmall);

	int index;
	int st = 0;

	char dat_bin[16];
	if(P.OutBinary == 0){
		sprintf(dat_bin, "%s", "dat");
	}
	else{
		sprintf(dat_bin, "%s", "bin");
	}

	for(int j = 0; j < N + Nsmall; j+=1){
		if(Nst > 1) st = index_h[j] / def_MaxIndex;
		if(P.FormatP == 0){
			char outputfilename[300];
			if(Nst == 1){
				if(irregular == 0 || irregular == 3){
					sprintf(outputfilename, "%sOut%s_p%.6d.%s", GSF[st].path, GSF[st].X, index_h[j], dat_bin);
				}
				else{
					sprintf(outputfilename, "%sOutIrr%s_p%.6d.%s", GSF[st].path, GSF[st].X, index_h[j], dat_bin);
				}
			}
			else{
				if(irregular == 0 || irregular == 3){
					sprintf(outputfilename, "%sOut%s_p%.6d.%s", GSF[st].path, GSF[st].X, index_h[j] % def_MaxIndex, dat_bin);
				}
				else{
					sprintf(outputfilename, "%sOutIrr%s_p%.6d.%s", GSF[st].path, GSF[st].X, index_h[j] % def_MaxIndex, dat_bin);

				}
			}
			if((time > ict_h[st] && idt_h[st] > 0.0) || (time < ict_h[st] && idt_h[st] < 0.0)){
				if(P.OutBinary == 0){
					outputfile = fopen(outputfilename, "a");
				}
				else{
					outputfile = fopen(outputfilename, "ab");
				}
			}
			else{
				if(P.OutBinary == 0){
					outputfile = fopen(outputfilename, "w");
				}
				else{
					outputfile = fopen(outputfilename, "wb");
				}
			}
		}
#if def_TTV == 0
		if(Nst == 1 || P.FormatS == 1) index = index_h[j];
		else index = index_h[j] % def_MaxIndex;
#else
		index = index_h[j];
#endif

		aecountT_h[j] += aecount_h[j];
		enccountT_h[j] += enccount_h[j];

		if(x4_h[j].w >= 0.0){//在output中输出制定数据
			//aei
			//aei(double3 x4i, double3 v4i, double mu, double &a, double &e, double &inc, double &Omega, double &w, double &Theta, double &E, double &M)
			double mu = Msun + x4_h[j].w;
			double a,e,inc,Omega,w,Theta,E,M;
			double3 x3_h;
			x3_h.x=x4_h[j].x;
			x3_h.y=x4_h[j].y;
			x3_h.z=x4_h[j].z;
			double3 v3_h;
			v3_h.x=v4_h[j].x;
			v3_h.y=v4_h[j].y;
			v3_h.z=v4_h[j].z;
			aei(x3_h,v3_h,mu,a,e,inc,Omega,w,Theta,E,M);
			//printf("hahahahah:%g,%g,%g,%g,%g,%g,%g\n\n\n\n",a,e,inc,Omega,w,Theta,E,M);
			
			///end aei
			if(P.OutBinary == 0){
				//fprintf(outputfile,"%.16g %d %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.40g %.8g %.8g %.8g %.8g %.8g %.8g %lld %.40g \n", time, index, x4_h[j].w, v4Helio_h[j].w, x4_h[j].x, x4_h[j].y, x4_h[j].z, v4Helio_h[j].x, v4Helio_h[j].y, v4Helio_h[j].z, spin_h[j].x, spin_h[j].y, spin_h[j].z, aelimits_h[j].x, aelimits_h[j].y, aelimits_h[j].z, aelimits_h[j].w, (double)(aecount_h[j])/ci, (double)(aecountT_h[j])/timeStep, enccountT_h[j], test_h[j]);
				for(int f = 0; f < def_Ninformat; ++f){
					if(GSF[st].outformat[f] == 19){
						fprintf(outputfile,"%.16g ", time);
					}
					if(GSF[st].outformat[f] == 13){
						fprintf(outputfile,"%d ", index);
					}
					if(GSF[st].outformat[f] == 4){
						fprintf(outputfile,"%.40g ", x4_h[j].w);
					}
					if(GSF[st].outformat[f] == 8){
						fprintf(outputfile,"%.40g ", v4Helio_h[j].w);
					}
					if(GSF[st].outformat[f] == 1){
						fprintf(outputfile,"%.40g ", x4_h[j].x);
					}
					if(GSF[st].outformat[f] == 2){
						fprintf(outputfile,"%.40g ", x4_h[j].y);
					}
					if(GSF[st].outformat[f] == 3){
						fprintf(outputfile,"%.40g ", x4_h[j].z);
					}
					if(GSF[st].outformat[f] == 5){
						fprintf(outputfile,"%.40g ", v4Helio_h[j].x);
					}
					if(GSF[st].outformat[f] == 6){
						fprintf(outputfile,"%.40g ", v4Helio_h[j].y);
					}
					if(GSF[st].outformat[f] == 7){
						fprintf(outputfile,"%.40g ", v4Helio_h[j].z);
					}
					if(GSF[st].outformat[f] == 10){
						fprintf(outputfile,"%.40g ", spin_h[j].x);
					}
					if(GSF[st].outformat[f] == 11){
						fprintf(outputfile,"%.40g ", spin_h[j].y);
					}
					if(GSF[st].outformat[f] == 12){
						fprintf(outputfile,"%.40g ", spin_h[j].z);
					}
					if(GSF[st].outformat[f] == 15){
						fprintf(outputfile,"%.8g ", aelimits_h[j].x);
					}
					if(GSF[st].outformat[f] == 16){
						fprintf(outputfile,"%.8g ", aelimits_h[j].y);
					}
					if(GSF[st].outformat[f] == 17){
						fprintf(outputfile,"%.8g ", aelimits_h[j].z);
					}
					if(GSF[st].outformat[f] == 18){
						fprintf(outputfile,"%.8g ", aelimits_h[j].w);
					}
					if(GSF[st].outformat[f] == 20){
						fprintf(outputfile,"%.40g ", love_h[j].x);
					}
					if(GSF[st].outformat[f] == 21){
						fprintf(outputfile,"%.40g ", love_h[j].y);
					}
					if(GSF[st].outformat[f] == 22){
						fprintf(outputfile,"%.40g ", love_h[j].z);
					}
					if(GSF[st].outformat[f] == 47){
						fprintf(outputfile,"%.8g ", (double)(aecount_h[j])/ci);
					}
					if(GSF[st].outformat[f] == 48){
						fprintf(outputfile,"%.8g ", (double)(aecountT_h[j])/timeStep);
					}
					if(GSF[st].outformat[f] == 46){
						fprintf(outputfile,"%llu ", enccountT_h[j]);
					}
					if(GSF[st].outformat[f] == 42){
						fprintf(outputfile,"%.40g ", rcrit_h[j]);
					}
					if(GSF[st].outformat[f] == 44){
						fprintf(outputfile,"%.40g ", spin_h[j].w);
					}
					if(GSF[st].outformat[f] == 45){
						fprintf(outputfile,"%.40g ", test_h[j]);
					}	
					if(P.UseMigrationForce > 0){
						if(GSF[st].outformat[f] == 49){
							fprintf(outputfile,"%.40g ", migration_h[j].x);
						}
						if(GSF[st].outformat[f] == 50){
							fprintf(outputfile,"%.40g ", migration_h[j].y);
						}
						if(GSF[st].outformat[f] == 51){
							fprintf(outputfile,"%.40g ", migration_h[j].z);
						}
					}
					else{
						if(GSF[st].outformat[f] == 49){
							fprintf(outputfile,"%.40g ", 0.0);
						}
						if(GSF[st].outformat[f] == 50){
							fprintf(outputfile,"%.40g ", 0.0);
						}
						if(GSF[st].outformat[f] == 51){
							fprintf(outputfile,"%.40g ", 0.0);
						}

					}
					if(GSF[st].outformat[f] == 23){
						fprintf(outputfile,"%.40g ", a);
					}
					if(GSF[st].outformat[f] == 24){
						fprintf(outputfile,"%.40g ", e);
					}
					if(GSF[st].outformat[f] == 25){
						fprintf(outputfile,"%.40g ", inc);
					}
					if(GSF[st].outformat[f] == 26){
						fprintf(outputfile,"%.40g ", Omega);
					}
					if(GSF[st].outformat[f] == 27){
						fprintf(outputfile,"%.40g ", w);
					}
					if(GSF[st].outformat[f] == 28){
						fprintf(outputfile,"%.40g ", M);
					}
				}
				fprintf(outputfile,"\n");
			}
			else{
				float aecount = (double)(aecount_h[j])/ci;
				float aecountT = (double)(aecountT_h[j])/timeStep;

				for(int f = 0; f < def_Ninformat; ++f){
					if(GSF[st].outformat[f] == 19){
						fwrite(&time, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 13){
						fwrite(&index, sizeof(int), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 4){
						fwrite(&x4_h[j].w, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 8){
						fwrite(&v4Helio_h[j].w, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 1){
						fwrite(&x4_h[j].x, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 2){
						fwrite(&x4_h[j].y, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 3){
						fwrite(&x4_h[j].z, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 5){
						fwrite(&v4Helio_h[j].x, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 6){
						fwrite(&v4Helio_h[j].y, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 7){
						fwrite(&v4Helio_h[j].z, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 10){
						fwrite(&spin_h[j].x, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 11){
						fwrite(&spin_h[j].y, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 12){
						fwrite(&spin_h[j].z, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 15){
						fwrite(&aelimits_h[j].x, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 16){
						fwrite(&aelimits_h[j].y, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 17){
						fwrite(&aelimits_h[j].z, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 18){
						fwrite(&aelimits_h[j].w, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 20){
						fwrite(&love_h[j].x, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 21){
						fwrite(&love_h[j].y, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 22){
						fwrite(&love_h[j].z, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 47){
						fwrite(&aecount, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 48){
						fwrite(&aecountT, sizeof(float), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 46){
						fwrite(&enccountT_h[j], sizeof(unsigned long long), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 42){
						fwrite(&rcrit_h[j], sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 44){
						fwrite(&spin_h[j].w, sizeof(double), 1, outputfile);
					}
					if(GSF[st].outformat[f] == 45){
						fwrite(&test_h[j], sizeof(double), 1, outputfile);
					}
					if(P.UseMigrationForce > 0){
						if(GSF[st].outformat[f] == 49){
							fwrite(&migration_h[j].x, sizeof(double), 1, outputfile);
						}
						if(GSF[st].outformat[f] == 50){
							fwrite(&migration_h[j].y, sizeof(double), 1, outputfile);
						}
						if(GSF[st].outformat[f] == 51){
							fwrite(&migration_h[j].z, sizeof(double), 1, outputfile);
						}
					}
					else{
						double d = 0.0;
						if(GSF[st].outformat[f] == 49){
							fwrite(&d, sizeof(double), 1, outputfile);
						}
						if(GSF[st].outformat[f] == 50){
							fwrite(&d, sizeof(double), 1, outputfile);
						}
						if(GSF[st].outformat[f] == 51){
							fwrite(&d, sizeof(double), 1, outputfile);
						}
					}
				}
			}
		}
		if(P.FormatP == 0) fclose(outputfile);
	}

	if(P.UseTides > 0 || P.UseRotationalDeformation > 0){
		for(int st = 0; st < Nst; ++st){
			//print star file
			FILE *starfile;
			if(irregular == 0 || irregular == 3){ 
				starfile = fopen(GSF[st].starfilename, "a");
			}
			else{
				starfile = fopen(GSF[st].starIrrfilename, "a");
			}
			hipMemcpy(Msun_h + st, Msun_d + st, sizeof(double2), hipMemcpyDeviceToHost);
			hipMemcpy(Spinsun_h + st, Spinsun_d + st, sizeof(double4), hipMemcpyDeviceToHost);
			hipMemcpy(Lovesun_h + st, Lovesun_d + st, sizeof(double3), hipMemcpyDeviceToHost);
			fprintf(starfile, "%.16g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g\n", time, Msun_h[st].x, Msun_h[st].y, Spinsun_h[st].x, Spinsun_h[st].y, Spinsun_h[st].z, Spinsun_h[st].w, Lovesun_h[st].x, Lovesun_h[st].y, Lovesun_h[st].z);

			fclose(starfile);
		}
	}
}

//this function prints the first close encounter information to the info file, partA
__host__ void Data::firstInfo(){
	hipMemcpy(Nencpairs_h, Nencpairs_d, (Nst + 1) * sizeof(int), hipMemcpyDeviceToHost);
	for(int st = 0; st < Nst; ++st){
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		if(Nst == 1) fprintf(GSF[st].logfile, "Initial Precheck pairs: %d\n", Nencpairs_h[0]);
		else fprintf(GSF[st].logfile, "Initial Precheck pairs: %d\n", Nencpairs_h[st + 1]);
		fclose(GSF[st].logfile);
		if(MTFlag == 1) break;
	}
}

//this function prints the first close encounter information to the info file, partB
__host__ void Data::firstInfoB(){
	for(int st = 0; st < Nst; ++st){
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		hipMemcpy(Nencpairs2_h + st + 1, Nencpairs2_d + st + 1, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Nencpairs_h + st + 1, Nencpairs_d + st + 1, sizeof(int), hipMemcpyDeviceToHost);

		if(Nst == 1){
			fprintf(GSF[0].logfile, "    CE:    %d; ", Nencpairs2_h[0]);
			fprintf(GSF[0].logfile, "groups: %d; ", Nenc_m[0]);
			int nn = 2;
			for(int st = 1; st < def_GMax; ++st){
				if(Nenc_m[st] > 0) fprintf(GSF[0].logfile, "%d: %d; ", nn, Nenc_m[st]);
				nn *= 2;
			}
			fprintf(GSF[0].logfile, "\n");

			fprintf(GSF[0].logfile, "    Precheck-pairs:    %d\n", Nencpairs_h[0]);
		}
		else{
			fprintf(GSF[st].logfile, "    CE:    %d\n", Nencpairs2_h[st + 1]);
			fprintf(GSF[st].logfile, "    Precheck-pairs:    %d\n", Nencpairs_h[st + 1]);
		}
		if(interrupt == 1){
			fprintf(GSF[st].logfile, "GENGA is terminated by SIGINT signal at time step %lld\n", timeStep);

		}
		fclose(GSF[st].logfile);
	}
}

__host__ int Data::firstEnergy(){

	for(int st = 0; st < Nst; ++st){
		EnergyCall(st, 0);
	}
	hipDeviceSynchronize();
	error = hipGetLastError();
	fprintf(masterfile,"Energy error = %d = %s\n",error, hipGetErrorString(error));
	if(error != 0){
		printf("Energy error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	return 1;
}

//This function calls the Energy function and prints information
__host__ int Data::EnergyOutput(int irregular){
	FILE *Energyfile;
	for(int hst = 0; hst < 16; ++hst){
		error = hipGetLastError();
		if(error != 0){
			printf("Energy Stream error = %d = %s %lld\n",error, hipGetErrorString(error), timeStep);
			return 0;
		}
	}
	if(P.Usegas == 1){
		if(Nst == 1){
			gasEnergyCall();
		}
		else{
#if def_CPU == 0
			for(int st = 0; st < Nst; ++st){
				gasEnergyMCall(st);
			}
#endif
		}
	}
	for(int st = 0; st < Nst; ++st){
		EnergyCall(st, 1);
	}

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != 0){
		printf("Energy error = %d = %s %lld\n",error, hipGetErrorString(error), timeStep);
		return 0;
	}

	if(Nst > 1) hipMemcpy(time_h, time_d, Nst*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(Energy_h, Energy_d, sizeof(double) * NEnergyT, hipMemcpyDeviceToHost);
	hipMemcpy(Nencpairs2_h, Nencpairs2_d, sizeof(int), hipMemcpyDeviceToHost);
	for(int st = 0; st < Nst; ++st){

//printf("Print Energy | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		if(Nst > 1){
			int s = 0;
		
			if(irregular < 3) s = 1;	
			if(N_h[st] < Nmin[st].x) s = 1;
			if(Nsmall_h[st] < Nmin[st].y) s = 1;
			if(n1_h[st] < 0) s = 1;
			if(timeStep >= delta_h[st]) s = 1;
			//print only simulations which must be stopped by StopAtEncounter
			//or when the simulation reached the end
			if(s == 0){
				continue;
			}			
		}
//printf("Print Energy2 | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);

		if(irregular == 0 || irregular == 3){
			Energyfile = fopen(GSF[st].Energyfilename, "a");
		}
		else{
			Energyfile = fopen(GSF[st].EnergyIrrfilename, "a");
		}
		if(Energyfile == NULL){
			printf("Error, Energyfile not valid %d %s\n", st, GSF[st].Energyfilename);
			return 0;
		}
		int NE = NEnergy[st];
		fprintf(Energyfile,"%.16g %d %.20g %.20g %.20g %.20g %.20g %.20g %.20g %.20g\n", time_h[st]/365.25, N_h[st] + Nsmall_h[st], Energy_h[0 + NE], Energy_h[1 + NE], Energy_h[2 + NE], Energy_h[3 + NE], Energy_h[4 + NE], Energy_h[5 + NE], Energy_h[6 + NE], Energy_h[7 + NE]);
		fclose(Energyfile);

		if(irregular == 0 || interrupt == 1){
			GSF[st].logfile = fopen(GSF[st].logfilename, "a");
			hipMemcpy(Nencpairs2_h + st + 1, Nencpairs2_d + st + 1, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(Nencpairs_h + st + 1, Nencpairs_d + st + 1, sizeof(int), hipMemcpyDeviceToHost);

			if(Nst == 1){
				fprintf(GSF[0].logfile, "    CE:    %d; ", Nencpairs2_h[0]);
				fprintf(GSF[0].logfile, "groups: %d; ", Nenc_m[0]);
				int nn = 2;
				for(int st = 1; st < def_GMax; ++st){
					if(Nenc_m[st] > 0) fprintf(GSF[0].logfile, "%d: %d; ", nn, Nenc_m[st]);
					nn *= 2;
				}
				fprintf(GSF[0].logfile, "\n");

				fprintf(GSF[0].logfile, "    Precheck-pairs:    %d\n", Nencpairs_h[0]);
			}
			else{
				fprintf(GSF[st].logfile, "    CE:    %d\n", Nencpairs2_h[st + 1]);
				fprintf(GSF[st].logfile, "    Precheck-pairs:    %d\n", Nencpairs_h[st + 1]);
			}
			if(interrupt == 1){
				fprintf(GSF[st].logfile, "GENGA is terminated by SIGINT signal at time step %lld\n", timeStep);

			}
			fclose(GSF[st].logfile);
		}

	}
	
	return 1;
}


__global__ void CoordinateToBuffer_kernel(double4 *x4_d, double4 *v4_d, int *index_d, double4 *spin_d, double3 *love_d, double3 *migration_d, double *rcrit_d, float4 *aelimits_d, unsigned int* aecount_d, unsigned long long *aecountT_d, unsigned long long *enccountT_d, double *test_d, double *coordinateBuffer_d, double *time_d, double *idt_d, const int Nst, const int NT, const int NsmallT, const int NconstT, const int bufferCount, const double dTau, const int UseMigrationForce){

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < NT + NsmallT){
		//time
		if(Nst == 1){
			coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id] = time_d[0] + dTau * idt_d[0];
		}
		else{
			int st = index_d[id] / def_MaxIndex;
			coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id] = time_d[st] + dTau * idt_d[st];
		}
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 1] = index_d[id];
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 2] = x4_d[id].w;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 3] = v4_d[id].w;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 4] = x4_d[id].x;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 5] = x4_d[id].y;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 6] = x4_d[id].z;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 7] = v4_d[id].x;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 8] = v4_d[id].y;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 9] = v4_d[id].z;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 10] = spin_d[id].x;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 11] = spin_d[id].y;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 12] = spin_d[id].z;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 13] = aelimits_d[id].x;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 14] = aelimits_d[id].y;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 15] = aelimits_d[id].z;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 16] = aelimits_d[id].w;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 17] = aecount_d[id];
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 18] = aecountT_d[id];
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 19] = enccountT_d[id];
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 20] = test_d[id];
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 21] = spin_d[id].w;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 22] = love_d[id].x;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 23] = love_d[id].y;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 24] = love_d[id].z;
		coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 25] = rcrit_d[id];
		if(UseMigrationForce > 0){
			coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 26] = migration_d[id].x;
			coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 27] = migration_d[id].y;
			coordinateBuffer_d[def_BufferSize * NconstT * bufferCount + def_BufferSize * id + 28] = migration_d[id].z;
		}
	}
}

__host__ void Data::CoordinateToBuffer(int bufferCount, int irregular, double dTau){
	if(NT + NsmallT > 0){
		if(irregular == 0){
			CoordinateToBuffer_kernel <<< (NT + NsmallT + 511) / 512, 512 >>> (x4_d, v4_d, index_d, spin_d, love_d, migration_d, rcrit_d, aelimits_d, aecount_d, aecountT_d, enccountT_d, test_d, coordinateBuffer_d, time_d, idt_d, Nst, NT, NsmallT, NconstT, bufferCount, dTau, P.UseMigrationForce);
		}
		else{
			CoordinateToBuffer_kernel <<< (NT + NsmallT + 511) / 512, 512 >>> (x4_d, v4_d, index_d, spin_d, love_d, migration_d, rcrit_d, aelimits_d, aecount_d, aecountT_d, enccountT_d, test_d, coordinateBufferIrr_d, time_d, idt_d, Nst, NT, NsmallT, NconstT, bufferCount, dTau, P.UseMigrationForce);

		}
	}
}

//This function copies the data from the device to host and calls the printoutput function
//irregular indicates irregular output intervals, which are read from a calendar file
//irregular = 2 means to print Coordinates at Collision time
//irregular = 3 means to print the last time step
//irregular = 4 means Step Error output
__host__ void Data::CoordinateOutput(int irregular){
	hipMemcpy(x4_h, x4_d, sizeof(double4)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(v4_h, v4_d, sizeof(double4)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(index_h, index_d, sizeof(int)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(test_h, test_d, sizeof(double)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(spin_h, spin_d, sizeof(double4)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(love_h, love_d, sizeof(double3)*NconstT, hipMemcpyDeviceToHost);
	if(P.UseMigrationForce > 0){
		hipMemcpy(migration_h, migration_d, sizeof(double3)*NconstT, hipMemcpyDeviceToHost);
	}
	hipMemcpy(aelimits_h, aelimits_d, sizeof(float4)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(aecount_h, aecount_d, sizeof(unsigned int)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(enccount_h, enccount_d, sizeof(unsigned int)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(aecountT_h, aecountT_d, sizeof(unsigned long long)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(enccountT_h, enccountT_d, sizeof(unsigned long long)*NconstT, hipMemcpyDeviceToHost);
	hipMemcpy(rcrit_h, rcrit_d, sizeof(double) * NconstT * P.SLevels, hipMemcpyDeviceToHost);

	if(Nst > 1) hipMemcpy(time_h, time_d, Nst * sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	char dat_bin[16];
	if(P.OutBinary == 0){
		sprintf(dat_bin, "%s", "dat");
	}
	else{
		sprintf(dat_bin, "%s", "bin");
	}

	for(int st = 0; st < Nst; ++st){


		int NBS = NBS_h[st];

//printf("Print Output | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		if(Nst > 1){
			int s = 0;
		
			if(irregular < 3) s = 1;	
			if(N_h[st] < Nmin[st].x) s = 1;
			if(Nsmall_h[st] < Nmin[st].y) s = 1;
			if(n1_h[st] < 0) s = 1;
			if(timeStep >= delta_h[st]) s = 1;
			//print only simulations which must be stopped by StopAtEncounter
			//or when the simulation reached the end
			if(s == 0){
				continue;
			}			
		}
//printf("Print Output2 | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		if(P.FormatP == 1){
			if(irregular == 2){
				sprintf(GSF[st].outputfilename,"OutCollision.%s", dat_bin);
				if(P.OutBinary == 0){
					GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
				}
				else{
					GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
				}
			}
			else if(irregular == 4){
				sprintf(GSF[st].outputfilename,"OutError.%s", dat_bin);
				if(P.OutBinary == 0){
					GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
				}
				else{
					GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
				}
			}
			else if(Nst == 1 || P.FormatS == 0){
				if(P.FormatT == 0){
					if(irregular == 0 || irregular == 3){
						long long scale = 1ll;
						if(P.FormatO == 1){
							scale = (long long)(P.ci);
							if(P.ci == -1) scale = (long long)(delta_h[st]);
						}
						sprintf(GSF[st].outputfilename,"%sOut%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, timeStep / scale, dat_bin);
						if(P.FormatO == 1 && interrupt == 1){
							sprintf(GSF[st].outputfilename,"%sOutbackup%s_%.20lld.%s", GSF[st].path, GSF[st].X, timeStep, dat_bin);
						}
					}
					else if(irregular == 1){
						sprintf(GSF[st].outputfilename,"%sOutIrr%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, irrTimeStep, dat_bin);
					}
#if def_TTV == 0
					if(P.OutBinary == 0){
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
					}
					else{
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
					}
#else
					if(P.OutBinary == 0){
						if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
						else GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
					}
					else{
						if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
						else GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
					}
#endif	
				}
				if(P.FormatT == 1){
					if(irregular == 0 || irregular == 3){
						sprintf(GSF[st].outputfilename,"%sOut%s.%s", GSF[st].path, GSF[st].X, dat_bin);
					}
					else if(irregular == 1){
						sprintf(GSF[st].outputfilename,"%sOutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
					}
					if(P.OutBinary == 0){
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
					}
					else{
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
					}
				}
			}
			else{
				if(P.FormatT == 0){
					if(irregular == 0 || irregular == 3){
						long long scale = 1ll;
						if(P.FormatO == 1){
							scale = (long long)(P.ci);
							if(P.ci == -1) scale = (long long)(delta_h[st]);
						}
						sprintf(GSF[st].outputfilename, "%s../Out%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, timeStep / scale, dat_bin);
						if(P.FormatO == 1 && interrupt == 1){
							sprintf(GSF[st].outputfilename, "%s../Outbackup%s_%.20lld.%s", GSF[st].path, GSF[st].X, timeStep, dat_bin);
						}
					}
					else if(irregular == 1){
						sprintf(GSF[st].outputfilename, "%s../OutIrr%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, irrTimeStep, dat_bin);
					}
					if(P.OutBinary == 0){
						if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
						else GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
					}
					else{
						if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
						else GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
					}
				}
				if(P.FormatT == 1){
					if(irregular == 0 || irregular == 3){
						sprintf(GSF[st].outputfilename, "%s../Out%s.%s", GSF[st].path, GSF[st].X, dat_bin);
					}
					else if(irregular == 1){
						sprintf(GSF[st].outputfilename, "%s../OutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
					}
					if(P.OutBinary == 0){
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
					}
					else{
						GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
					}
				}
			}
		}
		//if(irregular < 3 || timeStep == delta_h[st] || irregular == 4){
			printOutput(x4_h + NBS, v4_h + NBS, v4Helio_h + NBS, index_h + NBS, test_h + NBS, time_h[st]/365.25, timeStep, N_h[st], GSF[st].outputfile, Msun_h[st].x, spin_h + NBS, love_h + NBS, migration_h + NBS, rcrit_h + NBS, Nsmall_h[st], Nst, aelimits_h + NBS, aecount_h + NBS, enccount_h + NBS, aecountT_h + NBS, enccountT_h + NBS, P.ci, irregular);

			if(P.FormatP == 1) fclose(GSF[st].outputfile);
		//}

	}
	hipMemcpy(aecountT_d, aecountT_h, sizeof(unsigned long long)*NconstT, hipMemcpyHostToDevice);
	hipMemcpy(enccountT_d, enccountT_h, sizeof(unsigned long long)*NconstT, hipMemcpyHostToDevice);

	hipMemset(aecount_d, 0, sizeof(unsigned int)*NconstT);
	hipMemset(enccount_d, 0, sizeof(unsigned int)*NconstT);
}

//This function copies the data from the coordinate buffer and calls the printoutput function
__host__ void Data::CoordinateOutputBuffer(int irregular){

	double *buffer_h;

	if(irregular == 0){
		hipMemcpy(coordinateBuffer_h, coordinateBuffer_d, P.Buffer * def_BufferSize * NconstT * sizeof(double), hipMemcpyDeviceToHost);
		buffer_h = coordinateBuffer_h;
	}
	else{
		hipMemcpy(coordinateBufferIrr_h, coordinateBufferIrr_d, P.Buffer * def_BufferSize * NconstT * sizeof(double), hipMemcpyDeviceToHost);
		buffer_h = coordinateBufferIrr_h;
	}
	hipDeviceSynchronize();

	char dat_bin[16];
	if(P.OutBinary == 0){
		sprintf(dat_bin, "%s", "dat");
	}
	else{
		sprintf(dat_bin, "%s", "bin");
	}

	int Nbf = bufferCount;
	if(irregular == 1) Nbf = bufferCountIrr;
	for(int bf = 0; bf < Nbf; ++bf){
		for(int i = 0; i < NT + NsmallT; ++i){
			index_h[i] =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 1];
			x4_h[i].w =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 2];
			v4_h[i].w =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 3];
			x4_h[i].x =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 4];
			x4_h[i].y =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 5];
			x4_h[i].z =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 6];
			v4_h[i].x =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 7];
			v4_h[i].y =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 8];
			v4_h[i].z =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 9];
			spin_h[i].x =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 10];
			spin_h[i].y =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 11];
			spin_h[i].z =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 12];
			aelimits_h[i].x =	buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 13];
			aelimits_h[i].y =	buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 14];
			aelimits_h[i].z =	buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 15];
			aelimits_h[i].w =	buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 16];
			aecount_h[i] =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 17];
			aecountT_h[i] =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 18];
			enccountT_h[i] =	buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 19];
			test_h[i] =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 20];
			spin_h[i].w =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 21];
			love_h[i].x =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 22];
			love_h[i].y =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 23];
			love_h[i].z =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 24];
			rcrit_h[i] =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 25];
			if(P.UseMigrationForce > 0){
				migration_h[i].x =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 26];
				migration_h[i].y =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 27];
				migration_h[i].z =		buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * i + 28];
			}

		}
		for(int st = 0; st < Nst; ++st){
			int NBS = NBS_h[st];

//printf("Print Output Buffer %d %d %g\n", irregular, st, n1_h[st]);
		if(Nst > 1){
			int s = 0;

			if(irregular < 3) s = 1;	
			if(N_h[st] < Nmin[st].x) s = 1;
			if(Nsmall_h[st] < Nmin[st].y) s = 1;
			if(n1_h[st] < 0) s = 1;
			if(timeStep >= delta_h[st]) s = 1;
			//print only simulations which must be stopped by StopAtEncounter
			//or when the simulation reached the end
			if(s == 0){
				continue;
			}			
		}

			if(P.FormatP == 1){
				if(Nst == 1 || P.FormatS == 0){
					if(P.FormatT == 0){
						if(irregular == 0){
							sprintf(GSF[st].outputfilename,"%sOut%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, timestepBuffer[bf], dat_bin);
						}
						else{
							sprintf(GSF[st].outputfilename,"%sOutIrr%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, irrTimeStepOut + bf, dat_bin);
						}
						if(P.OutBinary == 0){
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
						}
						else{
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
						}
					}
					if(P.FormatT == 1){
						if(irregular == 0){
							sprintf(GSF[st].outputfilename,"%sOut%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						}
						else{
							sprintf(GSF[st].outputfilename,"%sOutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						}
						if(P.OutBinary == 0){
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
						}
						else{
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
						}
					}
				}
				else{
					if(P.FormatT == 0){
						if(irregular == 0){
							sprintf(GSF[st].outputfilename, "%s../Out%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, timestepBuffer[bf], dat_bin);
						}
						else{
							sprintf(GSF[st].outputfilename, "%s../OutIrr%s_%.*lld.%s", GSF[st].path, GSF[st].X, def_NFileNameDigits, irrTimeStepOut + bf, dat_bin);
						}
						if(P.OutBinary == 0){
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "w");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
						}
						else{
							if(st == 0) GSF[st].outputfile = fopen(GSF[st].outputfilename, "wb");
							else GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
						}
					}
					if(P.FormatT == 1){
						if(irregular == 0){
							sprintf(GSF[st].outputfilename, "%s../Out%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						}
						else{
							sprintf(GSF[st].outputfilename, "%s../OutIrr%s.%s", GSF[st].path, GSF[st].X, dat_bin);
						}
						if(P.OutBinary == 0){
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "a");
						}
						else{
							GSF[st].outputfile = fopen(GSF[st].outputfilename, "ab");
						}
					}
				}
			}
	
			double time;
			if(irregular == 0){
				time = timestepBuffer[bf] * idt_h[st] + ict_h[st] * 365.25;
				int N = NBuffer[Nst * bf + st].x;		
				int Nsmall = NBuffer[Nst * bf + st].y;
				printOutput(x4_h + NBS, v4_h + NBS, v4Helio_h + NBS, index_h + NBS, test_h + NBS, time/365.25, timestepBuffer[bf], N, GSF[st].outputfile, Msun_h[st].x, spin_h + NBS, love_h + NBS, migration_h + NBS, rcrit_h + NBS, Nsmall, Nst, aelimits_h + NBS, aecount_h + NBS, enccount_h + NBS, aecountT_h + NBS, enccountT_h + NBS, P.ci, irregular);
			}
			else{
				int N = NBufferIrr[Nst * bf + st].x;		
				int Nsmall = NBufferIrr[Nst * bf + st].y;		
				time = buffer_h[def_BufferSize * NconstT * bf + def_BufferSize * NBS];
				printOutput(x4_h + NBS, v4_h + NBS, v4Helio_h + NBS, index_h + NBS, test_h + NBS, time/365.25, timestepBufferIrr[bf], N, GSF[st].outputfile, Msun_h[st].x, spin_h + NBS, love_h + NBS, migration_h + NBS, rcrit_h + NBS, Nsmall, Nst, aelimits_h + NBS, aecount_h + NBS, enccount_h + NBS, aecountT_h + NBS, enccountT_h + NBS, P.ci, irregular);
			}

			if(P.FormatP == 1) fclose(GSF[st].outputfile);

		}
	}
	hipMemcpy(aecountT_d, aecountT_h, sizeof(unsigned long long)*NconstT, hipMemcpyHostToDevice);
	hipMemcpy(enccountT_d, enccountT_h, sizeof(unsigned long long)*NconstT, hipMemcpyHostToDevice);

	hipMemset(aecount_d, 0, sizeof(unsigned int)*NconstT);
	hipMemset(enccount_d, 0, sizeof(unsigned int)*NconstT);
}


__host__ void Data::GridaeOutput(){
	int GridNae = Gridae.Na * Gridae.Ne;
	int GridNai = Gridae.Na * Gridae.Ni;
	sprintf(Gridae.filename, "aeCount%s_%.*lld.dat", Gridae.X, def_NFileNameDigits, timeStep);
	Gridae.file = fopen(Gridae.filename, "w");
	hipMemcpy(Gridaecount_h, Gridaecount_d, sizeof(unsigned int)*GridNae, hipMemcpyDeviceToHost);
	hipMemcpy(Gridaicount_h, Gridaicount_d, sizeof(unsigned int)*GridNai, hipMemcpyDeviceToHost);
	//ae grid
	for(int i = 0; i < Gridae.Ne; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			if(timeStep > Gridae.Start){
				GridaecountS_h[i * Gridae.Na + j] += Gridaecount_h[i * Gridae.Na + j];
				GridaecountT_h[i * Gridae.Na + j] += Gridaecount_h[i * Gridae.Na + j];
			}
			fprintf(Gridae.file, "%llu ", GridaecountT_h[i * Gridae.Na + j]);
		}
		fprintf(Gridae.file, "\n");
	}
	fprintf(Gridae.file, "\n");
	fprintf(Gridae.file, "\n");
	for(int i = 0; i < Gridae.Ne; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			fprintf(Gridae.file, "%llu ", GridaecountS_h[i * Gridae.Na + j]);
			GridaecountS_h[i * Gridae.Na + j] = 0;
		}
		fprintf(Gridae.file, "\n");
	}
	fprintf(Gridae.file, "\n");
	fprintf(Gridae.file, "\n");
	//ai grid
	for(int i = 0; i < Gridae.Ni; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			if(timeStep > Gridae.Start){
				GridaicountS_h[i * Gridae.Na + j] += Gridaicount_h[i * Gridae.Na + j];
				GridaicountT_h[i * Gridae.Na + j] += Gridaicount_h[i * Gridae.Na + j];
			}
			fprintf(Gridae.file, "%llu ", GridaicountT_h[i * Gridae.Na + j]);
		}
		fprintf(Gridae.file, "\n");
	}
	fprintf(Gridae.file, "\n");
	fprintf(Gridae.file, "\n");
	for(int i = 0; i < Gridae.Ni; ++i){
		for(int j = 0; j < Gridae.Na; ++j){
			fprintf(Gridae.file, "%llu ", GridaicountS_h[i * Gridae.Na + j]);
			GridaicountS_h[i * Gridae.Na + j] = 0;
		}
		fprintf(Gridae.file, "\n");
	}

	fclose(Gridae.file);
	hipMemset(Gridaecount_d, 0, sizeof(int)*GridNae);
	hipMemset(Gridaicount_d, 0, sizeof(int)*GridNai);
}


//This function prints information if a too big close encounter group occurs and stops the integrations
__host__ int Data::MaxGroups(){
	for(int nm = def_GMax - 1; nm < def_GMax; ++nm){
	//for(int nm = 12; nm < def_GMax; ++nm){
		if(Nenc_m[nm] > 0){
			GSF[0].logfile = fopen(GSF[0].logfilename, "a");
			hipMemcpy(Nencpairs2_h, Nencpairs2_d, sizeof(int), hipMemcpyDeviceToHost);
			fprintf(GSF[0].logfile, "Number of Close-Encounter-pairs: %d\n", *Nencpairs2_h);
			fprintf(GSF[0].logfile, "Total number of groups: %d; ", Nenc_m[0]);
			int nn = 2;
			for(int st = 1; st < def_GMax; ++st){
				if(Nenc_m[st] > 0) fprintf(GSF[0].logfile, "%d: %d; ", nn, Nenc_m[st]);
				nn *= 2;
			}
			fprintf(GSF[0].logfile, "\n");

			fprintf(GSF[0].logfile, "Number of Precheck-pairs: %d\n", *Nencpairs_h);
			fprintf(GSF[0].logfile,"Output data when Error occured:\n");
			hipMemcpy(index_h, index_d, sizeof(int)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(test_h, test_d, sizeof(double)*NB[0], hipMemcpyDeviceToHost);
			hipMemcpy(spin_h, spin_d, sizeof(double4)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(love_h, love_d, sizeof(double3)*NconstT, hipMemcpyDeviceToHost);
			if(P.UseMigrationForce > 0){
				hipMemcpy(migration_h, migration_d, sizeof(double3)*NconstT, hipMemcpyDeviceToHost);
			}
			hipMemcpy(aelimits_h, aelimits_d, sizeof(float4)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(aecount_h, aecount_d, sizeof(unsigned int)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(enccount_h, enccount_d, sizeof(unsigned int)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(aecountT_h, aecountT_d, sizeof(unsigned long long)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(enccountT_h, enccountT_d, sizeof(unsigned long long)*NconstT, hipMemcpyDeviceToHost);
			hipMemcpy(rcrit_h, rcrit_d, sizeof(double) * NconstT * P.SLevels, hipMemcpyDeviceToHost);


			if(P.OutBinary == 0){
				GSF[0].outputfile = fopen(GSF[0].outputfilename, "w");	
			}
			else{
				GSF[0].outputfile = fopen(GSF[0].outputfilename, "wb");	
			}
			printOutput(x4_h, v4_h, v4Helio_h, index_h, test_h, time_h[0]/365.25, timeStep, N_h[0], GSF[0].outputfile, Msun_h[0].x, spin_h, love_h, migration_h, rcrit_h, Nsmall_h[0], Nst, aelimits_h, aecount_h, enccount_h, aecountT_h, enccountT_h, P.ci, 0);
			fclose(GSF[0].outputfile);

			fprintf(GSF[0].logfile,"Error: Too big group:%g. Integration Stopped at timestep = %lld\n", pow(2.0, nm), timeStep);
			printf("Error: Too big group:%g. Integration Stopped at timestep = %lld\n", pow(2.0, nm), timeStep);
			fclose(GSF[0].logfile);
			return 0;
		}
	}
	return 1;
}


//This functions set the starting rutime of the integrations
__host__ void Data::setStartTime(){
	hipEventCreate(&tt1);
	hipEventCreate(&tt2);
	hipEventCreate(&tt3);
	hipEventCreate(&tt4);

	hipEventRecord(tt1, 0);
	hipEventRecord(tt2, 0);

	times = 0.0f;
}


//This function prints information how long the integration takes
__host__ int Data::printTime(int irregular){
	
	hipEventRecord(tt3, 0);
	hipEventSynchronize(tt3);
	hipEventElapsedTime(&times, tt2, tt3);
	FILE *timefile;
	for(int st = 0; st < Nst; ++st){

//printf("Print time | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		if(Nst > 1){
			int s = 0;
		
			if(irregular < 3) s = 1;	
			if(N_h[st] < Nmin[st].x) s = 1;
			if(Nsmall_h[st] < Nmin[st].y) s = 1;
			if(n1_h[st] < 0) s = 1;
			if(timeStep >= delta_h[st]) s = 1;
			//print only simulations which must be stopped by StopAtEncounter
			//or when the simulation reached the end
			if(s == 0){
				continue;
			}			
		}
//printf("Print time2 | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);

		timefile = fopen(GSF[st].timefilename, "a");
		if(timefile == NULL){
			printf("Error, timefile not valid %d %s\n", st, GSF[st].timefilename);
			return 0;
		}

		fprintf(timefile, "%lld %.20g\n", timeStep, times * 0.001);
		fclose(timefile);
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		if(GSF[st].logfile == NULL){
			printf("Error, infofile not valid %d %s\n", st, GSF[st].logfilename);
			return 0;
		}
		fprintf(GSF[st].logfile,"Reached timestep %lld with %d bodies, %d test particles. Total Energy: %.20g\n", timeStep, N_h[st], Nsmall_h[st], Energy_h[4 + NEnergy[st]]);
		fclose(GSF[st].logfile);

		if(Nst == 1){
			printf("Reached timestep %lld with %d bodies, %d test particles. Total Energy: %.20g\n", timeStep, N_h[0], Nsmall_h[0], Energy_h[4]);
			fprintf(masterfile, "Reached timestep %lld with %d bodies, %d test particles. Total Energy: %.20g\n", timeStep, N_h[0], Nsmall_h[0], Energy_h[4]);
		}
		else if(st == 0) {
			printf("Reached timestep %lld with %d simulations\n", timeStep, Nst);
			fprintf(masterfile, "Reached timestep %lld with %d simulations\n", timeStep, Nst);
		}
	}
	if(irregular == 0){
		hipEventRecord(tt2, 0);
	}
	return 1;
}

//This function prints the total integration runtime
__host__ void Data::printLastTime(int irregular){
	hipEventRecord(tt4, 0);
	hipEventSynchronize(tt4);
	hipEventElapsedTime(&times, tt1, tt4);
	FILE *timefile;
	for(int st = 0; st < Nst; ++st){

//printf("Print last time | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		if(Nst > 1){
			int s = 0;
		
			if(irregular < 3) s = 1;	
			if(N_h[st] < Nmin[st].x) s = 1;
			if(Nsmall_h[st] < Nmin[st].y) s = 1;
			if(n1_h[st] < 0) s = 1;
			if(timeStep >= delta_h[st]) s = 1;
			//print only simulations which must be stopped by StopAtEncounter
			//or when the simulation reached the end
			if(s == 0){
				continue;
			}			
		}
//printf("Print last time2 | irregular: %d st: %d n1: %g\n", irregular, st, n1_h[st]);
		timefile = fopen(GSF[st].timefilename, "a");
		if(irregular == 0){
			fprintf(timefile, "\n\n%lld %.20g\n", timeStep -1, times * 0.001);
		}
		else{
			fprintf(timefile, "\n\n%lld %.20g\n", timeStep, times * 0.001);
		}
		if(st == 0) printf("Execution time: \n\n%g\n", times * 0.001);
		fclose(timefile);
	}
}


//This function prints the last information
__host__ void Data::LastInfo(){
	for(int st = 0; st < Nst; ++st){
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		fprintf(GSF[st].logfile,"Integration finished with %d bodies, %d test particles. Total Energy: %.20g\n", N_h[st], Nsmall_h[st], Energy_h[4 + NEnergy[st]]);
		fclose (GSF[st].logfile);
	}
	if(Nst > 1) printf("Integration finished with %d simulations\n", Nst);
	else printf("Integration finished with %d bodies, %d test particles. Total Energy: %.20g\n", N_h[0], Nsmall_h[0], Energy_h[4]);
}

//This function prints details of the Collisions
//stopAtCollision checks if one of the 2 colliding bodies is large enough to resolve the collision externally.
__host__ int Data::printCollisions(){
  
	hipMemcpy(Coll_h, Coll_d, sizeof(double) * def_NColl * Ncoll_m[0], hipMemcpyDeviceToHost);
	FILE *collisionfile;
	FILE *logfile;
	int stopAtCollision = 0;
	for(int nc = 0; nc < Ncoll_m[0]; ++nc){
		int st;
		if(Nst == 1) st = 0;
		else st = (int)(Coll_h[nc * def_NColl + 1]) / def_MaxIndex;
		collisionfile = fopen(GSF[st].collisionfilename, "a");

		logfile = fopen(GSF[st].logfilename, "a");

		for(int in = 0; in < def_NColl; ++in){
			if(in == 1 || in == 13){
				if(Nst == 1) fprintf(collisionfile, "%d ", (int)(Coll_h[nc * def_NColl + in]));
				else fprintf(collisionfile, "%d ", ((int)(Coll_h[nc * def_NColl + in])) % def_MaxIndex);
			}
			else fprintf(collisionfile, "%.20g ", Coll_h[nc * def_NColl + in]);
		}
		if(Nst == 1){
			fprintf(logfile, "Collision between body %d and %d\n", (int)(Coll_h[nc * def_NColl + 1]), (int)(Coll_h[nc * def_NColl + 13]));
			printf("Collision between body %d and %d\n", (int)(Coll_h[nc * def_NColl + 1]), (int)(Coll_h[nc * def_NColl + 13]));
		}
		else{
			fprintf(logfile, "Collision between body %d and %d\n", (int)(Coll_h[nc * def_NColl + 1]) % def_MaxIndex , (int)(Coll_h[nc * def_NColl + 13]) % def_MaxIndex);
			printf("In Simulation %s: Collision between body %d and %d\n", GSF[st].path, (int)(Coll_h[nc * def_NColl + 1]) % def_MaxIndex , (int)(Coll_h[nc * def_NColl + 13]) % def_MaxIndex);
		}
	
		if(Coll_h[nc * def_NColl + 2] >= P.StopMinMass && Coll_h[nc * def_NColl + 14] >= P.StopMinMass){
			stopAtCollision = 1;
		}

		fprintf(collisionfile, "\n");
		fclose(collisionfile);
		fclose(logfile);
	}
	return stopAtCollision;
}

//This function prints details of the Collisions
__host__ void Data::printCollisionsTshift(){
  
	FILE *collisionfile;
	for(int nc = Ncoll_m[0] / 2; nc < Ncoll_m[0]; ++nc){
		int st;
		if(Nst == 1) st = 0;
		else st = (int)(Coll_h[nc * def_NColl + 1]) / def_MaxIndex;
		collisionfile = fopen(GSF[st].collisionTshiftfilename, "a");

		for(int in = 0; in < def_NColl; ++in){
			if(in == 1 || in == 13){
				if(Nst == 1) fprintf(collisionfile, "%d ", (int)(Coll_h[nc * def_NColl + in]));
				else fprintf(collisionfile, "%d ", ((int)(Coll_h[nc * def_NColl + in])) % def_MaxIndex);
			}
			else fprintf(collisionfile, "%.20g ", Coll_h[nc * def_NColl + in]);
		}
		fprintf(collisionfile, "\n");
		fclose(collisionfile);
	}
}

//This function prints details of the Encounters
__host__ int Data::printEncounters(){
 
	if(NWriteEnc_m[0] >= def_MaxWriteEnc){
		for(int st = 0; st < Nst; ++st){ 
			GSF[st].logfile = fopen(GSF[st].logfilename, "a");
			fprintf(GSF[st].logfile, "Error: Too many Encounters to write %d, allowed are %d\n", NWriteEnc_m[0], def_MaxWriteEnc);
			printf("Error: Too many Encounters to write %d, allowed are %d\n", NWriteEnc_m[0], def_MaxWriteEnc);
			fclose(GSF[st].logfile);
		}
		return 0;
	}
 
	hipMemcpy(writeEnc_h, writeEnc_d, sizeof(double) * def_NColl * NWriteEnc_m[0], hipMemcpyDeviceToHost);

	FILE *encounterfile;
	for(int nc = 0; nc < NWriteEnc_m[0]; ++nc){
		int st;
		if(Nst == 1) st = 0;
		else st = (int)(writeEnc_h[nc * def_NColl + 1]) / def_MaxIndex;
		encounterfile = fopen(GSF[st].encounterfilename, "a");

		for(int in = 0; in < def_NColl; ++in){
			if(in == 1 || in == 13){
				if(Nst == 1) fprintf(encounterfile, "%d ", (int)(writeEnc_h[nc * def_NColl + in]));
				else fprintf(encounterfile, "%d ", ((int)(writeEnc_h[nc * def_NColl + in])) % def_MaxIndex);
			}
			else fprintf(encounterfile, "%.20g ", writeEnc_h[nc * def_NColl + in]);
		}
		fprintf(encounterfile, "\n");
		fclose(encounterfile);
	}
	return 1;
}

//This function prints details of fragmentations
__host__ int Data::printFragments(int nf){

	int st = 0; 
	GSF[st].logfile = fopen(GSF[st].logfilename, "a");
	fprintf(GSF[st].logfile, "Created %d fragments\n", nf);
	printf("Created %d fragments\n", nf);
	fclose(GSF[st].logfile);

	if(nf > P.Nfragments){
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		fprintf(GSF[st].logfile, "Error: More particles created than Nfragments: %d %d\n", nf, P.Nfragments);
		printf("Error: Error: More particles created than Nfragments: %d %d\n", nf, P.Nfragments);
		fclose(GSF[st].logfile);

		return 0;
	}

	if(N_h[0] + Nsmall_h[0] >= NconstT){
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");
		fprintf(GSF[st].logfile, "Error: Too many particles created\n");
		printf("Error: Too many particles created\n");
		fclose(GSF[st].logfile);

		return 0;
	}
 
	hipMemcpy(Fragments_h, Fragments_d, sizeof(double) * 25 * P.Nfragments, hipMemcpyDeviceToHost);

	FILE *fragmentfile;
	for(int nc = 0; nc < nf + 1; ++nc){
		int st;
		if(Nst == 1) st = 0;
		else st = (int)(Fragments_h[nc * 25 + 1]) / def_MaxIndex;
		fragmentfile = fopen(GSF[st].fragmentfilename, "a");

		for(int in = 0; in < 13; ++in){
			if(in == 1 || in == 13){
				if(Nst == 1) fprintf(fragmentfile, "%d ", (int)(Fragments_h[nc * 25 + in]));
				else fprintf(fragmentfile, "%d ", ((int)(Fragments_h[nc * 25 + in])) % def_MaxIndex);
			}
			else fprintf(fragmentfile, "%.20g ", Fragments_h[nc * 25 + in]);
		}
		if(nc == 0) fprintf(fragmentfile, " -1\n");	//particle is destroyed
		else{
			if(Fragments_h[nc * 25 + 3] * def_AU < P.Asteroid_rdel) fprintf(fragmentfile, " 2\n");	//new particle but too small
			else fprintf(fragmentfile, " 1\n");							//new particle
		}
		fclose(fragmentfile);
	}
	return 1;
}
//This function prints details of rotation resets
__host__ int Data::printRotation(){

	int st = 0; 
	GSF[st].logfile = fopen(GSF[st].logfilename, "a");
	fprintf(GSF[st].logfile, "Rotation reset\n");
	printf("Rotation reset\n");
	fclose(GSF[st].logfile);

	hipMemcpy(Fragments_h, Fragments_d, sizeof(double) * 25, hipMemcpyDeviceToHost);

	FILE *fragmentfile;
	if(Nst == 1) st = 0;
	else st = (int)(Fragments_h[1]) / def_MaxIndex;
	fragmentfile = fopen(GSF[st].fragmentfilename, "a");

	for(int in = 0; in < 13; ++in){
		if(in == 1 || in == 13){
			if(Nst == 1) fprintf(fragmentfile, "%d ", (int)(Fragments_h[in]));
			else fprintf(fragmentfile, "%d ", ((int)(Fragments_h[in])) % def_MaxIndex);
		}
		else fprintf(fragmentfile, "%.20g ", Fragments_h[in]);
	}
	fprintf(fragmentfile, " 0\n");
	fclose(fragmentfile);

	return 1;
}

//This function prints details of particle creation events
__host__ int Data::printCreateparticle(int nf){

	int st = 0; 

	hipMemcpy(Fragments_h, Fragments_d, sizeof(double) * 25 * nf, hipMemcpyDeviceToHost);

	FILE *fragmentfile;
	for(int nc = 0; nc < nf; ++nc){
		if(Nst == 1) st = 0;
		else st = (int)(Fragments_h[nc * 25 + 1]) / def_MaxIndex;
		fragmentfile = fopen(GSF[st].fragmentfilename, "a");
		GSF[st].logfile = fopen(GSF[st].logfilename, "a");

		int id = -1;
		for(int in = 0; in < 13; ++in){
			if(in == 1 || in == 13){
				if(Nst == 1){
					id = (int)(Fragments_h[nc * 25 + in]);
					fprintf(fragmentfile, "%d ", id);
				}
				else{
					id = ((int)(Fragments_h[nc * 25 + in])) % def_MaxIndex;
					fprintf(fragmentfile, "%d ", id);
				}
			}
			else fprintf(fragmentfile, "%.20g ", Fragments_h[nc * 25 + in]);
		}
		fprintf(GSF[st].logfile, "Create particle %d\n", id);
		fprintf(fragmentfile, " 10\n");
		fclose(fragmentfile);
		fclose(GSF[st].logfile);
	}

	return 1;
}

//This function prints the transit times
__host__ int Data::printTransits(){
	hipMemcpy(TransitTime_h, TransitTime_d, def_NtransitTimeMax * NconstT * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(elementsC_h, elementsC_d, (Nst + MCMC_NT) * sizeof(int2), hipMemcpyDeviceToHost);

	FILE *Transitfile;
	Transitfile = fopen("Transits.dat", "a");
	for(int i = 0; i < NconstT; ++i){
		int si = i / def_MaxIndex;
		if(elementsC_h[si + MCMC_NT].x >= 0){
			int Epoch = 0;
			int setEpoch = 0;
			for(int EpochObs = 0; EpochObs <= NtransitsT_h[i].x; ++EpochObs){
				double T = TransitTime_h[i * def_NtransitTimeMax + Epoch + 1]; 
				double T1 = TransitTime_h[i * def_NtransitTimeMax + Epoch + 2];
				double2 TObs;
				if(EpochObs <= NtransitsTObs_h[i % N_h[0]]){
					TObs = TransitTimeObs_h[(i % N_h[0]) * def_NtransitTimeMax + EpochObs + 1];
				}
				else{
					TObs.x = 0.0;
					TObs.y = 0.0;
				}

//printf("---- %d %.20g %.20g %d %d\n", i, T, TObs.x, Epoch, EpochObs);

				if(fabs(TObs.x - T) < fabs(TObs.x - T1) && T != 0.0 && TObs.x != 0.0){
					setEpoch = 1;
				}


				if(setEpoch == 0 && T != 0 && TObs.x != 0 && fabs(TObs.x - T) < fabs(TObs.x - T1)){
//printf("***** %d %.20g %.20g %d %d\n", i, T, TObs.x, Epoch, EpochObs);
					++EpochObs;
					TObs = TransitTimeObs_h[(i % N_h[0]) * def_NtransitTimeMax + EpochObs + 1];
				}

				if(fabs(TObs.x - T) < fabs(TObs.x - T1) && T != 0.0 && TObs.x != 0.0){
					setEpoch = 1;
				}

				if(P.PrintTransits == 2){				
					 if(setEpoch == 0) fprintf(Transitfile, "%d %d %25.20g %25.20g %25.20g\n", i, Epoch, T, 0.0, 0.0);
				}

				if(setEpoch == 0 && T != 0 && TObs.x != 0 && fabs(TObs.x - T) >= fabs(TObs.x - T1)){
//printf("#####  %d %.20g %.20g %d %d\n", i, T, TObs.x, Epoch, EpochObs);
					++Epoch;
					--EpochObs;
					continue;
				}

				if(P.PrintTransits == 1){				
					if(setEpoch == 1) fprintf(Transitfile, "%d %d %25.20g %25.20g %25.20g\n", i, EpochObs, T, TObs.x, TObs.y);
				}
				if(P.PrintTransits == 2){				
					 if(setEpoch == 1)fprintf(Transitfile, "%d %d %25.20g %25.20g %25.20g\n", i, Epoch, T, TObs.x, TObs.y);
				}

				++Epoch;
				if(NtransitsTObs_h[i % N_h[0]] >= def_NtransitTimeMax -1){
					printf("Error: more transits than def_NtransitTimeMax for object %d: %d %d\n", i, NtransitsTObs_h[i % N_h[0]], def_NtransitTimeMax);
					return 0;
				}
			}
		}
	}

	fclose(Transitfile);
	return 1;
}

//This function prints the RV data at the obervation times
__host__ int Data::printRV(){
	hipMemcpy(RV_h, RV_d, def_NRVMax * Nst * sizeof(double2), hipMemcpyDeviceToHost);

	FILE *RVfile;
	RVfile = fopen("RVs.dat", "a");
	for(int si = 0; si < Nst; ++si){
//printf("NVRT %d %d %d\n", si, NRVT_h[si].x, NRVTObs_h[si]);
		for(int i = 0; i < NRVT_h[si].x; ++i){
			double2 T = RV_h[si * def_NRVMax + i]; 
			double3 TObs;
			if(i <= NRVTObs_h[si]){
				TObs = RVObs_h[si * def_NRVMax + i];
			}
			else{
				TObs.x = 0.0;
				TObs.y = 0.0;
				TObs.z = 1.0;
			}

			
			fprintf(RVfile, "%d %d %.25g %25.20g %25.20g %25.20g\n", si, i, T.x, T.y, TObs.y, TObs.z);

			if(NRVTObs_h[si] >= def_NRVMax -1){
				printf("Error: more RV data than def_NRVMax: %d %d\n", NRVTObs_h[si], def_NRVMax);
				return 0;
			}
		}
	}

	fclose(RVfile);
	return 1;
}
//This function prints the RV data at all time steps and no observation data
__host__ int Data::printRV2(){
	hipMemcpy(RV_h, RV_d, def_NRVMax * Nst * sizeof(double2), hipMemcpyDeviceToHost);

	FILE *RVfile;
	RVfile = fopen("RVall.dat", "a");
	for(int si = 0; si < Nst; ++si){
//printf("NVRT %d %d %d\n", si, NRVT_h[si].x, NRVTObs_h[si]);
		for(int i = 0; i < NRVT_h[si].x; ++i){
			double2 T = RV_h[si * def_NRVMax + i]; 
			
			fprintf(RVfile, "%d %d %.25g %25.20g\n", si, i, T.x, T.y);

			if(NRVTObs_h[si] >= def_NRVMax -1){
				printf("Error: more RV data than def_NRVMax: %d %d\n", NRVTObs_h[si], def_NRVMax);
				return 0;
			}
		}
	}

	fclose(RVfile);
	return 1;
}

__host__ void Data::printMCMC(int E){
	FILE *MCMCfile;
	MCMCfile = fopen("MCMC.dat", "a");

	if(P.PrintMCMC == 3){
	//print all, reprint old values for not accepted steps
		hipMemcpy(elementsA_h, elementsAOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsB_h, elementsBOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsT_h, elementsTOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsSpin_h, elementsSpinOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
	}
	if(P.PrintMCMC == 2){
	//print all, also not accepted steps
		hipMemcpy(elementsA_h, elementsA_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsB_h, elementsB_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsT_h, elementsT_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsSpin_h, elementsSpin_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
	}
	if(P.PrintMCMC == 1){
	//print only accepted
		hipMemcpy(elementsA_h, elementsAOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsB_h, elementsBOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsT_h, elementsTOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
		hipMemcpy(elementsSpin_h, elementsSpinOld_d, NconstT * sizeof(double4), hipMemcpyDeviceToHost);
	}
	hipMemcpy(elementsL_h, elementsL_d, NconstT * sizeof(elements10), hipMemcpyDeviceToHost);
	hipMemcpy(elementsP_h, elementsP_d, Nst * sizeof(double4), hipMemcpyDeviceToHost);
	hipMemcpy(elementsSA_h, elementsSA_d, Nst * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(elementsC_h, elementsC_d, (Nst + MCMC_NT) * sizeof(int2), hipMemcpyDeviceToHost);

	if(E == 0){

#if MCMC_Q == 0
		for(int id = 0; id < NconstT; ++id){
#elif MCMC_Q == 2
		for(int id = 0; id < NconstT; ++id){
#else
		for(int id = 0; id < NconstT / 3; ++id){

#endif
			int si = 0;
			if(Nst > 1) si = index_h[id] / def_MaxIndex;

			int p = 0;
			double pp = elementsP_h[si].z;
			if(P.PrintMCMC == 1){

				if(elementsC_h[si + MCMC_NT].x >= 0) p = 1;
			}
			if(P.PrintMCMC == 2){
				pp = elementsP_h[si].x;
				if(pp >= 1.0e299) pp = elementsP_h[si].z;
				if(elementsP_h[si].z < 1.0e299){
					p = 1;
				}
			}
			if(P.PrintMCMC == 3){
				if(pp >= 1.0e299) pp = elementsP_h[si].z;
				if(elementsP_h[si].z < 1.0e299){
					p = 1;
				}
			}

			if(p == 1){
				double f = 1.0;
				double time = ict_h[0];
		
				int ii = id;
				fprintf(MCMCfile, "%#15.10g %d %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g\n", time, id % N_h[0], elementsA_h[ii].w, elementsB_h[ii].w, elementsT_h[ii].z, elementsA_h[ii].y, elementsA_h[ii].z, elementsB_h[ii].x, elementsB_h[ii].y, elementsT_h[ii].x, f * elementsL_h[ii].m, f * elementsL_h[ii].r, f * elementsL_h[ii].P, f * elementsL_h[ii].e, f * elementsL_h[ii].inc, f * elementsL_h[ii].O, f * elementsL_h[ii].w, f * elementsL_h[ii].T, pp * 2.0, elementsP_h[si].w, elementsSA_h[si], Msun_h[si].x);
				//fprintf(MCMCfile, "%#15.10g %d %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g\n", time, id % N_h[0], elementsA_h[ii].w, elementsSpin_h[ii].y, elementsT_h[ii].z, elementsA_h[ii].y, elementsA_h[ii].z, elementsB_h[ii].x, elementsB_h[ii].y, elementsT_h[ii].x, f * elementsL_h[ii].m, f * elementsL_h[ii].r, f * elementsL_h[ii].P, f * elementsL_h[ii].e, f * elementsL_h[ii].inc, f * elementsL_h[ii].O, f * elementsL_h[ii].w, f * elementsL_h[ii].T, pp * 2.0, elementsP_h[si].w, elementsSA_h[si], Msun_h[si].x);
			}
		}
		fclose(MCMCfile);
	}
	else{
		MCMCfile = fopen("MCMC_bak.dat", "w");
		for(int id = 0; id < NconstT; ++id){
			int si = 0;
			if(Nst > 1) si = index_h[id] / def_MaxIndex;
			double f = 1.0;
			double time = ict_h[0];
		
			int ii = id;
			fprintf(MCMCfile, "%#15.10g %d %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#25.20g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g %#15.10g\n", time, id % N_h[0], elementsA_h[ii].w, elementsB_h[ii].w, elementsT_h[ii].z, elementsA_h[ii].y, elementsA_h[ii].z, elementsB_h[ii].x, elementsB_h[ii].y, elementsT_h[ii].x, f * elementsL_h[ii].m, f * elementsL_h[ii].r, f * elementsL_h[ii].P, f * elementsL_h[ii].e, f * elementsL_h[ii].inc, f * elementsL_h[ii].O, f * elementsL_h[ii].w, f * elementsL_h[ii].T, elementsP_h[si].x * 2.0, elementsP_h[si].w, elementsSA_h[si], Msun_h[si].x);
		}
		fclose(MCMCfile);
	}

}

