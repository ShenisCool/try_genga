
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// **********************************
// This code prints the number of NVIDIA GPUs and the compute capabilities
// compile with: nvcc -o CheckGPU CheckGPU.cu 

// Date: April 2022
// Author: Simon Grimm
// **********************************

int main(){

	int devCount = 0;
	int runtimeVersion = 0;
	int driverVersion = 0;

	hipError_t error;
	error = hipGetDeviceCount(&devCount);
	if(error > 0){
		printf("device error = %d = %s\n",error, hipGetErrorString(error));
		return 0;
	}
	if(devCount == 0){
		printf("Error: No valid cuda device!\n");
		return 0;
	}

	hipDeviceProp_t devProp;

	hipRuntimeGetVersion(&runtimeVersion);
	hipDriverGetVersion(&driverVersion);

	printf("There are %d CUDA devices.\n", devCount);
	printf("Runtime Version: %d\n", runtimeVersion);
	printf("Driver Version: %d\n", driverVersion);

	for(int i = 0; i < devCount; ++i){
		hipGetDeviceProperties(&devProp, i);

		int computeCapability = devProp.major * 10 + devProp.minor;

		printf("Name:%s, Major:%d, Minor:%d, Compute Capability: %d\n",
		devProp.name, devProp.major, devProp.minor, computeCapability);
	}

	return 0;
}

